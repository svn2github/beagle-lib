/*
 * @author Marc Suchard
 */
#ifndef _Included_PeelingFunctions
#define _Included_PeelingFunctions

/**************INCLUDES***********/
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include "BeagleCUDAImpl.h"
#include "CUDASharedFunctions.h"

#include "PeelingKernels.cu"

/**************CODE***********/
#ifdef __cplusplus
extern "C" {
#endif

REAL* ones = NULL; // TODO: Memory leak, need to free at some point.

void nativeGPUPartialsPartialsPruningDynamicScaling(REAL* partials1,
                                                    REAL* partials2,
                                                    REAL* partials3,
                                                    REAL* matrices1,
                                                    REAL* matrices2,
                                                    REAL* scalingFactors,
                                                    const unsigned int patternCount,
                                                    const unsigned int matrixCount,
                                                    int doRescaling) {
#ifdef DEBUG
    fprintf(stderr, "Entering GPU PP\n");
    hipDeviceSynchronize();
    checkCUDAError("PP kernel pre-invocation");
#endif

#if (PADDED_STATE_COUNT == 4)
    dim3 grid(patternCount / (PATTERN_BLOCK_SIZE * 4), matrixCount);
    if (patternCount % (PATTERN_BLOCK_SIZE * 4) != 0)
        grid.x += 1;
    dim3 block(16, PATTERN_BLOCK_SIZE);
#else
    dim3 grid(patternCount / PATTERN_BLOCK_SIZE, matrixCount);
    if (patternCount % PATTERN_BLOCK_SIZE != 0)
        grid.x += 1;
    dim3 block(PADDED_STATE_COUNT, PATTERN_BLOCK_SIZE);
#endif

    if (doRescaling)    {
        // Compute partials without any rescaling
#if (PADDED_STATE_COUNT == 4)
        kernelPartialsPartialsByPatternBlockCoherentSmall<<<grid, block>>>(partials1, partials2,
                                                                           partials3, matrices1,
                                                                           matrices2, patternCount);
#else
        kernelPartialsPartialsByPatternBlockCoherent<<<grid, block>>>(partials1, partials2,
                                                                      partials3, matrices1,
                                                                      matrices2, patternCount);
#endif

        hipDeviceSynchronize();

        // Rescale partials and save scaling factors
        nativeGPURescalePartials(partials3, scalingFactors, patternCount, matrixCount, 0);

    } else {

    // Compute partials with known rescalings
#if (PADDED_STATE_COUNT == 4)
        kernelPartialsPartialsByPatternBlockSmallFixedScaling<<<grid, block>>>(partials1, partials2,
                                                                               partials3, matrices1,
                                                                               matrices2,
                                                                               scalingFactors,
                                                                               patternCount);
#else
        kernelPartialsPartialsByPatternBlockFixedScaling<<<grid, block>>>(partials1, partials2,
                                                                          partials3, matrices1,
                                                                          matrices2, scalingFactors,
                                                                          patternCount);
#endif

    }

#ifdef DEBUG
    hipDeviceSynchronize();
    checkCUDAError("PP kernel invocation");
    fprintf(stderr, "Completed GPU PP\n");
#endif

}

void nativeGPUPartialsPartialsPruning(REAL* partials1,
                                      REAL* partials2,
                                      REAL* partials3,
                                      REAL* matrices1,
                                      REAL* matrices2,
                                      const unsigned int patternCount,
                                      const unsigned int matrixCount) {
#ifdef DEBUG
    fprintf(stderr, "Entering GPU PP\n");
    hipDeviceSynchronize();
    checkCUDAError("PP kernel pre-invocation");
#endif


#if (PADDED_STATE_COUNT == 4)
    dim3 block(16, PATTERN_BLOCK_SIZE);
    dim3 grid(patternCount / (PATTERN_BLOCK_SIZE * 4), matrixCount);
    if (patternCount % (PATTERN_BLOCK_SIZE * 4) != 0)
        grid.x += 1;

    kernelPartialsPartialsByPatternBlockCoherentSmall<<<grid, block>>>(partials1, partials2,
                                                                       partials3, matrices1,
                                                                       matrices2, patternCount);
#else
    dim3 grid(patternCount / PATTERN_BLOCK_SIZE, matrixCount);
    if (patternCount % PATTERN_BLOCK_SIZE != 0)
        grid.x += 1;
    dim3 block(PADDED_STATE_COUNT, PATTERN_BLOCK_SIZE);

    kernelPartialsPartialsByPatternBlockCoherent<<<grid, block>>>(partials1, partials2, partials3,
                                                                  matrices1, matrices2,
                                                                  patternCount);
#endif

#ifdef DEBUG
    hipDeviceSynchronize();
    checkCUDAError("PP kernel invocation");
    fprintf(stderr, "Completed GPU PP\n");
#endif

}

void nativeGPURescalePartials(REAL* partials3,
                              REAL* scalingFactors,
                              int patternCount,
                              int matrixCount,
                              int fillWithOnes) {
    // Rescale partials and save scaling factors
//#if (PADDED_STATE_COUNT == 4) 
    if (fillWithOnes != 0) {
        if (ones == NULL) {
            ones = (REAL*) malloc(SIZE_REAL * patternCount);
            for(int i = 0; i < patternCount; i++)
                ones[i] = 1.0;
        }
        hipMemcpy(scalingFactors, ones, sizeof(REAL*) * patternCount, hipMemcpyHostToDevice);
        return;
    }
//#endif

#ifndef SLOW_REWEIGHING
    dim3 grid2(patternCount, matrixCount / MATRIX_BLOCK_SIZE);
    if (matrixCount % MATRIX_BLOCK_SIZE != 0)
        grid2.y += 1;
    if (grid2.y > 1) {
        fprintf(stderr, "Not yet implemented! Try slow reweighing.\n");
        exit(0);
    }
    dim3 block2(PADDED_STATE_COUNT, MATRIX_BLOCK_SIZE);
    // TODO: Totally incoherent for PADDED_STATE_COUNT == 4
    kernelPartialsDynamicScaling<<<grid2, block2>>>(partials3, scalingFactors, matrixCount);
#else
    dim3 grid2(patternCount, 1);
    dim3 block2(PADDED_STATE_COUNT);
    kernelPartialsDynamicScalingSlow<<<grid2, block2>>>(partials3, scalingFactors, matrixCount);
#endif
}

void nativeGPUStatesStatesPruningDynamicScaling(INT* states1,
                                                INT* states2,
                                                REAL* partials3,
                                                REAL* matrices1,
                                                REAL* matrices2,
                                                REAL* scalingFactors,
                                                const unsigned int patternCount,
                                                const unsigned int matrixCount,
                                                int doRescaling) {
#if (PADDED_STATE_COUNT == 4)
    dim3 grid(patternCount/(PATTERN_BLOCK_SIZE * 4), matrixCount);
    if (patternCount % (PATTERN_BLOCK_SIZE * 4) != 0)
        grid.x += 1;
    dim3 block(16, PATTERN_BLOCK_SIZE);
#else
    dim3 grid(patternCount / PATTERN_BLOCK_SIZE, matrixCount);
    if (patternCount % PATTERN_BLOCK_SIZE != 0)
        grid.x += 1;
    dim3 block(PADDED_STATE_COUNT, PATTERN_BLOCK_SIZE);
#endif

    if (doRescaling)    {
        // Compute partials without any rescaling
#if (PADDED_STATE_COUNT == 4)
        kernelStatesStatesByPatternBlockCoherentSmall<<<grid, block>>>(states1, states2, partials3,
                                                                       matrices1, matrices2,
                                                                       patternCount);
#else
        kernelStatesStatesByPatternBlockCoherent<<<grid, block>>>(states1, states2, partials3,
                                                                  matrices1, matrices2,
                                                                  patternCount);
#endif
        hipDeviceSynchronize();

        // Rescale partials and save scaling factors
        // If PADDED_STATE_COUNT == 4, just with ones.
        nativeGPURescalePartials(partials3, scalingFactors, patternCount, matrixCount, 1);

    } else {

        // Compute partials with known rescalings
#if (PADDED_STATE_COUNT == 4)
        kernelStatesStatesByPatternBlockCoherentSmall<<<grid, block>>>(states1, states2, partials3,
                                                                       matrices1, matrices2,
                                                                       patternCount);
#else
        kernelStatesStatesByPatternBlockFixedScaling<<<grid, block>>>(states1, states2, partials3,
                                                                      matrices1, matrices2,
                                                                      scalingFactors, patternCount);
#endif
    }

#ifdef DEBUG
    fprintf(stderr, "Completed GPU SP\n");
#endif
}

void nativeGPUStatesPartialsPruningDynamicScaling(INT* states1,
                                                  REAL* partials2,
                                                  REAL* partials3,
                                                  REAL* matrices1,
                                                  REAL* matrices2,
                                                  REAL* scalingFactors,
                                                  const unsigned int patternCount,
                                                  const unsigned int matrixCount,
                                                  int doRescaling) {
#if (PADDED_STATE_COUNT == 4)
    dim3 grid(patternCount / (PATTERN_BLOCK_SIZE * 4), matrixCount);
    if (patternCount % (PATTERN_BLOCK_SIZE * 4) != 0)
        grid.x += 1;
    dim3 block(16, PATTERN_BLOCK_SIZE);
#else
    dim3 grid(patternCount / PATTERN_BLOCK_SIZE, matrixCount);
    if (patternCount % PATTERN_BLOCK_SIZE != 0)
        grid.x += 1;
    dim3 block(PADDED_STATE_COUNT, PATTERN_BLOCK_SIZE);
#endif

    if (doRescaling)    {
        // Compute partials without any rescaling
#if (PADDED_STATE_COUNT == 4)
        kernelStatesPartialsByPatternBlockCoherentSmall<<<grid, block>>>(states1, partials2,
                                                                         partials3, matrices1,
                                                                         matrices2, patternCount);
#else
        kernelStatesPartialsByPatternBlockCoherent<<<grid, block>>>(states1, partials2, partials3,
                                                                    matrices1, matrices2,
                                                                    patternCount);
#endif
        hipDeviceSynchronize();

        // Rescale partials and save scaling factors
        nativeGPURescalePartials(partials3, scalingFactors, patternCount, matrixCount, 1);
    } else {

        // Compute partials with known rescalings
#if (PADDED_STATE_COUNT == 4)
        kernelStatesPartialsByPatternBlockCoherentSmall<<<grid, block>>>(states1, partials2,
                                                                         partials3, matrices1,
                                                                         matrices2, patternCount);
#else
        kernelStatesPartialsByPatternBlockFixedScaling<<<grid, block>>>(states1, partials2,
                                                                        partials3, matrices1,
                                                                        matrices2, scalingFactors,
                                                                        patternCount);
#endif
    }

#ifdef DEBUG
    fprintf(stderr,"Completed GPU SP\n");
#endif

}

void nativeGPUComputeRootDynamicScaling(REAL** dNodePtrQueue,
                                        REAL* dRootScalingFactors,
                                        int nodeCount,
                                        int patternCount) {
    dim3 grid(patternCount / PATTERN_BLOCK_SIZE);
    if (patternCount % PATTERN_BLOCK_SIZE != 0)
        grid.x += 1;
    dim3 block(PATTERN_BLOCK_SIZE);

    kernelGPUComputeRootDynamicScaling<<<grid, block>>>(dNodePtrQueue, dRootScalingFactors,
                                                        nodeCount, patternCount);
}

void nativeGPUIntegrateLikelihoodsDynamicScaling(REAL* dResult,
                                                 REAL* dRootPartials,
                                                 REAL* dCategoryProportions,
                                                 REAL* dFrequencies,
                                                 REAL* dRootScalingFactors,
                                                 int patternCount,
                                                 int matrixCount,
                                                 int nodeCount) {

#ifdef DEBUG
    fprintf(stderr, "Entering IL\n");
#endif

    dim3 grid(patternCount);
    dim3 block(PADDED_STATE_COUNT);

    kernelGPUIntegrateLikelihoodsDynamicScaling<<<grid, block>>>(dResult, dRootPartials,
                                                                 dCategoryProportions, dFrequencies,
                                                                 dRootScalingFactors, matrixCount,
                                                                 nodeCount);

#ifdef DEBUG
    fprintf(stderr, "Exiting IL\n");
#endif
}

void nativeGPUIntegrateLikelihoods(REAL* dResult,
                                   REAL* dRootPartials,
                                   REAL* dCategoryProportions,
                                   REAL* dFrequencies,
                                   int patternCount,
                                   int matrixCount) {
#ifdef DEBUG
    fprintf(stderr,"Entering IL\n");
#endif

    dim3 grid(patternCount);
    dim3 block(PADDED_STATE_COUNT);

    kernelGPUIntegrateLikelihoods<<<grid, block>>>(dResult, dRootPartials, dCategoryProportions,
                                                   dFrequencies, matrixCount);

#ifdef DEBUG
    fprintf(stderr, "Exiting IL\n");
#endif

}

void nativeGPUStatesPartialsPruning(int* states1,
                                    REAL* partials2,
                                    REAL* partials3,
                                    REAL* matrices1,
                                    REAL* matrices2,
                                    const unsigned int patternCount,
                                    const unsigned int matrixCount) {
#ifdef DEBUG
    fprintf(stderr, "Entering GPU PP\n");
    hipDeviceSynchronize();
    checkCUDAError("PP kernel pre-invocation");
#endif


#if (PADDED_STATE_COUNT == 4)
    dim3 block(16, PATTERN_BLOCK_SIZE);
    dim3 grid(patternCount / (PATTERN_BLOCK_SIZE * 4), matrixCount);
    if (patternCount % (PATTERN_BLOCK_SIZE * 4) != 0)
        grid.x += 1;

    kernelStatesPartialsByPatternBlockCoherentSmall<<<grid, block>>>(states1, partials2, partials3,
                                                                     matrices1, matrices2,
                                                                     patternCount);
#else
    dim3 grid(patternCount / PATTERN_BLOCK_SIZE, matrixCount);
    if (patternCount % PATTERN_BLOCK_SIZE != 0)
        grid.x += 1;
    dim3 block(PADDED_STATE_COUNT, PATTERN_BLOCK_SIZE);

    kernelStatesPartialsByPatternBlockCoherent<<<grid, block>>>(states1, partials2, partials3,
                                                                matrices1, matrices2, patternCount);
#endif

#ifdef DEBUG
    hipDeviceSynchronize();
    checkCUDAError("PP kernel invocation");
    fprintf(stderr, "Completed GPU PP\n");
#endif

}

void nativeGPUStatesStatesPruning(int* states1,
                                  int* states2,
                                  REAL* partials3,
                                  REAL* matrices1,
                                  REAL* matrices2,
                                  const unsigned int patternCount,
                                  const unsigned int matrixCount) {
#ifdef DEBUG
    fprintf(stderr, "Entering GPU PP\n");
    hipDeviceSynchronize();
    checkCUDAError("PP kernel pre-invocation");
#endif


#if (PADDED_STATE_COUNT == 4)
    dim3 block(16, PATTERN_BLOCK_SIZE);
    dim3 grid(patternCount/(PATTERN_BLOCK_SIZE * 4), matrixCount);
    if (patternCount % (PATTERN_BLOCK_SIZE * 4) != 0)
        grid.x += 1;

    kernelStatesStatesByPatternBlockCoherentSmall<<<grid, block>>>(states1, states2, partials3,
                                                                   matrices1, matrices2,
                                                                   patternCount);
#else
    dim3 grid(patternCount / PATTERN_BLOCK_SIZE, matrixCount);
    if (patternCount % PATTERN_BLOCK_SIZE != 0)
        grid.x += 1;
    dim3 block(PADDED_STATE_COUNT, PATTERN_BLOCK_SIZE);

    kernelStatesStatesByPatternBlockCoherent<<<grid, block>>>(states1, states2, partials3,
                                                              matrices1, matrices2, patternCount);
#endif

#ifdef DEBUG
    hipDeviceSynchronize();
    checkCUDAError("PP kernel invocation");
    fprintf(stderr, "Completed GPU PP\n");
#endif

}

#ifdef __cplusplus
}
#endif
#endif
//
