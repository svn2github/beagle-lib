#include "hip/hip_runtime.h"
/*
 *
 * Copyright 2009 Phylogenetic Likelihood Working Group
 *
 * This file is part of BEAGLE.
 *
 * BEAGLE is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * BEAGLE is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with BEAGLE.  If not, see
 * <http://www.gnu.org/licenses/>.
 *
 * @author Marc Suchard
 * @author Daniel Ayres
 * @author Andrew Rambaut
 */

#define STATE_COUNT 4

#ifdef CUDA
    #include "libhmsbeagle/GPU/GPUImplDefs.h"
    #include "libhmsbeagle/GPU/kernels/kernelsAll.cu" // This file includes the non-state-count specific kernels
    extern "C" {
#endif 

#define multBy4(x)  (x << 2)
#define multBy16(x) (x << 4)
#define LIKE_PATTERN_BLOCK_SIZE PATTERN_BLOCK_SIZE

///////////////////////////////////////////////////////////////////////////////
// kernel macros

// Do not use | (instead of +) for any term involing PATTERN_BLOCK_SIZE
// as this should be adjustable
#define DETERMINE_INDICES_4()\
    int tx = KW_LOCAL_ID_0;\
    int state = tx & 0x3;\
    int pat = tx >> 2;\
    int patIdx = KW_LOCAL_ID_1;\
    int matrix = KW_GROUP_ID_1;\
    int pattern = __umul24(KW_GROUP_ID_0, PATTERN_BLOCK_SIZE * 4) + multBy4(patIdx) + pat;\
    int deltaPartialsByState = multBy16(KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE + patIdx);\
    int deltaPartialsByMatrix = __umul24(matrix, multBy4(totalPatterns));\
    int x2 = multBy16(matrix);\
    int u = tx + deltaPartialsByState + deltaPartialsByMatrix;
    
#define SUM_PARTIALS_PARTIALS_CPU()\
    REAL sum10, sum11, sum12, sum13;\
    REAL sum20, sum21, sum22, sum23;\
    int patIdx = KW_LOCAL_ID_0;\
    int matrix = KW_GROUP_ID_1;\
    int pattern = KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE + patIdx;\
    int deltaPartialsByState = pattern * PADDED_STATE_COUNT;\
    int deltaPartialsByMatrix = matrix * PADDED_STATE_COUNT * totalPatterns;\
    int deltaMatrix = matrix * PADDED_STATE_COUNT * PADDED_STATE_COUNT;\
    int deltaPartials = deltaPartialsByMatrix + deltaPartialsByState;\
    KW_GLOBAL_VAR REAL* matrix1 = matrices1 + deltaMatrix;\
    KW_GLOBAL_VAR REAL* matrix2 = matrices2 + deltaMatrix;\
    KW_GLOBAL_VAR REAL* sMatrix1 = matrix1;\
    KW_GLOBAL_VAR REAL* sMatrix2 = matrix2;\
    KW_GLOBAL_VAR REAL* sPartials1 = partials1 + deltaPartials;\
    KW_GLOBAL_VAR REAL* sPartials2 = partials2 + deltaPartials;\
    sum10 = sMatrix1[0 * 4 + 0] * sPartials1[0];\
    sum11 = sMatrix1[0 * 4 + 1] * sPartials1[0];\
    sum12 = sMatrix1[0 * 4 + 2] * sPartials1[0];\
    sum13 = sMatrix1[0 * 4 + 3] * sPartials1[0];\
    sum20 = sMatrix2[0 * 4 + 0] * sPartials2[0];\
    sum21 = sMatrix2[0 * 4 + 1] * sPartials2[0];\
    sum22 = sMatrix2[0 * 4 + 2] * sPartials2[0];\
    sum23 = sMatrix2[0 * 4 + 3] * sPartials2[0];\
    for (int i = 1; i < 4; i++) {\
        FMA(sMatrix1[i * 4 + 0],  sPartials1[i], sum10);\
        FMA(sMatrix1[i * 4 + 1],  sPartials1[i], sum11);\
        FMA(sMatrix1[i * 4 + 2],  sPartials1[i], sum12);\
        FMA(sMatrix1[i * 4 + 3],  sPartials1[i], sum13);\
        FMA(sMatrix2[i * 4 + 0],  sPartials2[i], sum20);\
        FMA(sMatrix2[i * 4 + 1],  sPartials2[i], sum21);\
        FMA(sMatrix2[i * 4 + 2],  sPartials2[i], sum22);\
        FMA(sMatrix2[i * 4 + 3],  sPartials2[i], sum23);\
    }

#define INTEGRATE_PARTIALS_CPU()\
    int pat = KW_LOCAL_ID_0;\
    int pattern = KW_GROUP_ID_0 * LIKE_PATTERN_BLOCK_SIZE + pat;\
    int u = pattern * PADDED_STATE_COUNT;\
    int delta = patternCount * PADDED_STATE_COUNT;\
    REAL sum[4];\
    sum[0] = dRootPartials[0 + u] * dWeights[0];\
    sum[1] = dRootPartials[1 + u] * dWeights[0];\
    sum[2] = dRootPartials[2 + u] * dWeights[0];\
    sum[3] = dRootPartials[3 + u] * dWeights[0];\
    for(int r = 1; r < matrixCount; r++) {\
        FMA(dRootPartials[0 + u + delta * r],  dWeights[r], sum[0]);\
        FMA(dRootPartials[1 + u + delta * r],  dWeights[r], sum[1]);\
        FMA(dRootPartials[2 + u + delta * r],  dWeights[r], sum[2]);\
        FMA(dRootPartials[3 + u + delta * r],  dWeights[r], sum[3]);\
    }\
    sum[0] *= dFrequencies[0];\
    sum[1] *= dFrequencies[1];\
    sum[2] *= dFrequencies[2];\
    sum[3] *= dFrequencies[3];

#define LOAD_MATRIX_GPU()\
    KW_GLOBAL_VAR REAL* matrix1 = matrices1 + x2; /*Points to *this* matrix*/\
    KW_GLOBAL_VAR REAL* matrix2 = matrices2 + x2;\
    KW_LOCAL_MEM REAL sMatrix1[16]; /*Load values into shared memory*/\
    KW_LOCAL_MEM REAL sMatrix2[16];\
    if (patIdx == 0 ) {\
        sMatrix1[tx] = matrix1[tx]; /*All coalesced memory reads*/\
        sMatrix2[tx] = matrix2[tx];\
    }

#define LOAD_PARTIALS_PARTIALS_GPU()\
    REAL sum1, sum2;\
    int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);\
    int y = deltaPartialsByState + deltaPartialsByMatrix;\
    KW_LOCAL_MEM REAL sPartials1[PATTERN_BLOCK_SIZE * 4 * 4];\
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];\
    /* copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials*/\
    if (pattern < totalPatterns) {\
        sPartials1[multBy16(patIdx) | tx] = partials1[y | tx]; /*All coalesced memory*/\
        sPartials2[multBy16(patIdx) | tx] = partials2[y | tx];\
    } else {\
        sPartials1[multBy16(patIdx) | tx] = 0;\
        sPartials2[multBy16(patIdx) | tx] = 0;\
    }

#define LOAD_STATES_PARTIALS_GPU()\
    REAL sum1 = 1;\
    REAL sum2;\
    int y = deltaPartialsByState + deltaPartialsByMatrix;\
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];\
    if (pattern < totalPatterns) {\
        sPartials2[patIdx * 16 + tx] = partials2[y + tx];\
    } else {\
        sPartials2[patIdx * 16 + tx] = 0;\
    }

#define LOAD_SCALING_GPU()\
    KW_LOCAL_MEM REAL fixedScalingFactors[PATTERN_BLOCK_SIZE * 4];\
    if (patIdx < 4) { /* need to load 4*PATTERN_BLOCK_SIZE factors for this block*/\
        fixedScalingFactors[patIdx * PATTERN_BLOCK_SIZE + tx] = \
            scalingFactors[KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE * 4 + patIdx * PATTERN_BLOCK_SIZE + tx];\
    }

#define SUM_PARTIALS_PARTIALS_GPU()\
    int i = pat;\
    sum1 = sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];\
    sum2 = sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];\
    i = (++i) & 0x3;\
    FMA(   sMatrix1[multBy4(i) | state],  sPartials1[patIdx16pat4 | i], sum1);\
    FMA(   sMatrix2[multBy4(i) | state],  sPartials2[patIdx16pat4 | i], sum2);\
    i = (++i) & 0x3;\
    FMA(   sMatrix1[multBy4(i) | state],  sPartials1[patIdx16pat4 | i], sum1);\
    FMA(   sMatrix2[multBy4(i) | state],  sPartials2[patIdx16pat4 | i], sum2);\
    i = (++i) & 0x3;\
    FMA(   sMatrix1[multBy4(i) | state],  sPartials1[patIdx16pat4 | i], sum1);\
    FMA(   sMatrix2[multBy4(i) | state],  sPartials2[patIdx16pat4 | i], sum2);

#define SUM_STATES_PARTIALS_GPU()\
    int state1 = states1[pattern];\
    if (state1 < PADDED_STATE_COUNT)\
        sum1 = sMatrix1[state1 * 4 + state];\
    int i = pat;\
    sum2  = sMatrix2[i * 4 + state] * sPartials2[patIdx * 16 + pat * 4 + i];\
    i = (++i) & 0x3;\
    FMA(    sMatrix2[i * 4 + state],  sPartials2[patIdx * 16 + pat * 4 + i], sum2);\
    i = (++i) & 0x3;\
    FMA(    sMatrix2[i * 4 + state],  sPartials2[patIdx * 16 + pat * 4 + i], sum2);\
    i = (++i) & 0x3;\
    FMA(    sMatrix2[i * 4 + state],  sPartials2[patIdx * 16 + pat * 4 + i], sum2);

#define FIND_MAX_PARTIALS_STATE_GPU()\
    int tx = KW_LOCAL_ID_0;\
    int state = tx & 0x3;\
    int pat = tx >> 2;\
    int patIdx = KW_GROUP_ID_0;\
    int pattern = (patIdx << 2) + pat;\
    int matrix = KW_LOCAL_ID_1;\
    /* TODO: Assumes matrixCount < MATRIX_BLOCK_SIZE*/\
    /* Patterns are always padded, so no reading/writing past end possible*/\
    /* Find start of patternBlock for thread-block*/\
    int partialsOffset = (matrix * KW_NUM_GROUPS_0 + patIdx) << 4; /* 16;*/\
    KW_LOCAL_MEM REAL partials[MATRIX_BLOCK_SIZE][16]; /* 4 patterns at a time*/\
    KW_LOCAL_MEM REAL storedPartials[MATRIX_BLOCK_SIZE][16];\
    KW_LOCAL_MEM REAL matrixMax[4];\
    if (matrix < matrixCount)\
        partials[matrix][tx] = allPartials[partialsOffset + tx];          \
    storedPartials[matrix][tx] = partials[matrix][tx];\
    KW_LOCAL_FENCE;\
    /* Unrolled parallel max-reduction*/\
    if (state < 2) {\
        REAL compare1 = partials[matrix][tx];\
        REAL compare2 = partials[matrix][tx + 2];\
        if (compare2 > compare1)\
            partials[matrix][tx] = compare2;\
    }\
    KW_LOCAL_FENCE;\
    if (state < 1) {\
        REAL compare1 = partials[matrix][tx];\
        REAL compare2 = partials[matrix][tx + 1];\
        if (compare2 > compare1)\
            partials[matrix][tx] = compare2;\
    }\
    KW_LOCAL_FENCE;

#define FIND_MAX_PARTIALS_MATRIX_GPU()\
    matrixMax[pat] = 0;\
    int m;\
    for(m = 0; m < matrixCount; m++) {\
        if (partials[m][tx] > matrixMax[pat])\
            matrixMax[pat] = partials[m][tx];\
    }

#define INTEGRATE_PARTIALS_GPU()\
    int state   = KW_LOCAL_ID_0;\
    int pat = KW_LOCAL_ID_1;\
    int pattern = KW_GROUP_ID_0 * LIKE_PATTERN_BLOCK_SIZE + KW_LOCAL_ID_1;\
    int u = state + pattern * PADDED_STATE_COUNT;\
    int delta = patternCount * PADDED_STATE_COUNT;\
    KW_LOCAL_MEM REAL stateFreq[4];\
    /* TODO: Currently assumes MATRIX_BLOCK_SIZE >= matrixCount */\
    KW_LOCAL_MEM REAL matrixProp[MATRIX_BLOCK_SIZE];\
    KW_LOCAL_MEM REAL sum[LIKE_PATTERN_BLOCK_SIZE][4];\
    /* Load shared memory */\
    if (pat == 0) {\
        stateFreq[state] = dFrequencies[state];\
    }\
    sum[pat][state] = 0;\
    /* TODO: Assumes matrixCount < LIKE_PATTERN_BLOCK_SIZE * 4 */\
    if (pat * 4 + state < matrixCount) {\
        matrixProp[pat * 4 + state] = dWeights[pat * 4 + state];\
    }\
    KW_LOCAL_FENCE;\
    for(int r = 0; r < matrixCount; r++) {\
        FMA(dRootPartials[u + delta * r], matrixProp[r], sum[pat][state]);\
    }\
    sum[pat][state] *= stateFreq[state];\
    KW_LOCAL_FENCE;\
    if (state < 2)\
        sum[pat][state] += sum[pat][state + 2];\
    KW_LOCAL_FENCE;\
    if (state < 1) {\
        sum[pat][state] += sum[pat][state + 1];\
    }

///////////////////////////////////////////////////////////////////////////////

KW_GLOBAL_KERNEL void kernelPartialsPartialsNoScale(KW_GLOBAL_VAR REAL* partials1,
                                                    KW_GLOBAL_VAR REAL* partials2,
                                                    KW_GLOBAL_VAR REAL* partials3,
                                                    KW_GLOBAL_VAR REAL* matrices1,
                                                    KW_GLOBAL_VAR REAL* matrices2,
                                                    int totalPatterns) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    SUM_PARTIALS_PARTIALS_CPU();
    partials3[deltaPartials + 0] = sum10 * sum20;
    partials3[deltaPartials + 1] = sum11 * sum21;
    partials3[deltaPartials + 2] = sum12 * sum22;
    partials3[deltaPartials + 3] = sum13 * sum23;
#else // GPU implementation
    DETERMINE_INDICES_4();
    LOAD_MATRIX_GPU();
    LOAD_PARTIALS_PARTIALS_GPU();
    KW_LOCAL_FENCE;
    if (pattern < totalPatterns) { // Remove padded threads!
        SUM_PARTIALS_PARTIALS_GPU();
        partials3[u] = sum1 * sum2;
    }
#endif

#ifdef KERNEL_PRINT_ENABLED
    printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n",
           matrix, pattern, tx, state, u);
#endif
}

KW_GLOBAL_KERNEL void kernelPartialsPartialsFixedScale(KW_GLOBAL_VAR REAL* partials1,
                                                       KW_GLOBAL_VAR REAL* partials2,
                                                       KW_GLOBAL_VAR REAL* partials3,
                                                       KW_GLOBAL_VAR REAL* matrices1,
                                                       KW_GLOBAL_VAR REAL* matrices2,
                                                       KW_GLOBAL_VAR REAL* scalingFactors,
                                                       int totalPatterns) {
    DETERMINE_INDICES_4();
    LOAD_MATRIX_GPU();
    LOAD_PARTIALS_PARTIALS_GPU();
    LOAD_SCALING_GPU();
    KW_LOCAL_FENCE;
    if (pattern < totalPatterns) { // Remove padded threads!
        SUM_PARTIALS_PARTIALS_GPU();
        partials3[u] = sum1 * sum2 / fixedScalingFactors[patIdx * 4 + pat];
    }
}
    
KW_GLOBAL_KERNEL void kernelStatesPartialsNoScale(KW_GLOBAL_VAR int* states1,
                                                  KW_GLOBAL_VAR REAL* partials2,
                                                  KW_GLOBAL_VAR REAL* partials3,
                                                  KW_GLOBAL_VAR REAL* matrices1,
                                                  KW_GLOBAL_VAR REAL* matrices2,
                                                  int totalPatterns) {
    DETERMINE_INDICES_4();
    LOAD_MATRIX_GPU();
    LOAD_STATES_PARTIALS_GPU();
    KW_LOCAL_FENCE;
    if (pattern < totalPatterns) { // Remove padded threads!
        SUM_STATES_PARTIALS_GPU();
        partials3[u] = sum1 * sum2;
    }
}

KW_GLOBAL_KERNEL void kernelStatesPartialsFixedScale(KW_GLOBAL_VAR int* states1,
                                                     KW_GLOBAL_VAR REAL* partials2,
                                                     KW_GLOBAL_VAR REAL* partials3,
                                                     KW_GLOBAL_VAR REAL* matrices1,
                                                     KW_GLOBAL_VAR REAL* matrices2,
                                                     KW_GLOBAL_VAR REAL* scalingFactors,
                                                     int totalPatterns) {

    DETERMINE_INDICES_4();
    LOAD_MATRIX_GPU();
    LOAD_STATES_PARTIALS_GPU();
    LOAD_SCALING_GPU();
    KW_LOCAL_FENCE;
    if (pattern < totalPatterns) { // Remove padded threads!
        SUM_STATES_PARTIALS_GPU();
        partials3[u] = sum1 * sum2 / fixedScalingFactors[patIdx * 4 + pat];
    }
}

KW_GLOBAL_KERNEL void kernelStatesStatesNoScale(KW_GLOBAL_VAR int* states1,
                                                KW_GLOBAL_VAR int* states2,
                                                KW_GLOBAL_VAR REAL* partials3,
                                                KW_GLOBAL_VAR REAL* matrices1,
                                                KW_GLOBAL_VAR REAL* matrices2,
                                                int totalPatterns) {

	DETERMINE_INDICES_4();
    LOAD_MATRIX_GPU();
    KW_LOCAL_FENCE;
    if (pattern < totalPatterns) {
        int state1 = states1[pattern];
        int state2 = states2[pattern];
        if (state1 < PADDED_STATE_COUNT && state2 < PADDED_STATE_COUNT) {
            partials3[u] = sMatrix1[state1 * 4 + state] * sMatrix2[state2 * 4 + state];
        } else if (state1 < PADDED_STATE_COUNT) {
            partials3[u] = sMatrix1[state1 * 4 + state];
        } else if (state2 < PADDED_STATE_COUNT) {
            partials3[u] = sMatrix2[state2 * 4 + state];
        } else {
            partials3[u] = 1.0;
        }
    }
}

KW_GLOBAL_KERNEL void kernelStatesStatesFixedScale(KW_GLOBAL_VAR int* states1,
                                                   KW_GLOBAL_VAR int* states2,
                                                   KW_GLOBAL_VAR REAL* partials3,
                                                   KW_GLOBAL_VAR REAL* matrices1,
                                                   KW_GLOBAL_VAR REAL* matrices2,
                                                   KW_GLOBAL_VAR REAL* scalingFactors,
                                                   int totalPatterns) {
	DETERMINE_INDICES_4();
    LOAD_MATRIX_GPU();
    LOAD_SCALING_GPU();
    KW_LOCAL_FENCE;
    if (pattern < totalPatterns) {
        int state1 = states1[pattern];
        int state2 = states2[pattern];
        if (state1 < PADDED_STATE_COUNT && state2 < PADDED_STATE_COUNT) {
            partials3[u] = sMatrix1[state1 * 4 + state] * sMatrix2[state2 * 4 + state]
                           / fixedScalingFactors[patIdx * 4 + pat];
        } else if (state1 < PADDED_STATE_COUNT) {
            partials3[u] = sMatrix1[state1 * 4 + state] / fixedScalingFactors[patIdx * 4 + pat];
        } else if (state2 < PADDED_STATE_COUNT) {
            partials3[u] = sMatrix2[state2 * 4 + state] / fixedScalingFactors[patIdx * 4 + pat];
        } else {
            partials3[u] = 1.0 / fixedScalingFactors[patIdx * 4 + pat];
        }
    }
}

// Find a scaling factor for each pattern
KW_GLOBAL_KERNEL void kernelPartialsDynamicScaling(KW_GLOBAL_VAR REAL* allPartials,
                                                   KW_GLOBAL_VAR REAL* scalingFactors,
                                                   int matrixCount) {
    FIND_MAX_PARTIALS_STATE_GPU();
    // Could also parallel-reduce here.
    if (state == 0 && matrix == 0) {
        FIND_MAX_PARTIALS_MATRIX_GPU();
        if (matrixMax[pat] == 0)
        	matrixMax[pat] = 1.0;
        scalingFactors[pattern] = matrixMax[pat]; // TODO: Are these incoherent writes?
    }
    KW_LOCAL_FENCE;
    if (matrix < matrixCount)
        allPartials[partialsOffset + tx] = storedPartials[matrix][tx] / matrixMax[pat];
}

KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingScalersLog(KW_GLOBAL_VAR REAL* allPartials,
                                                             KW_GLOBAL_VAR REAL* scalingFactors,
                                                             int matrixCount) {
    FIND_MAX_PARTIALS_STATE_GPU();
    if (state == 0 && matrix == 0) {
        FIND_MAX_PARTIALS_MATRIX_GPU();
        if (matrixMax[pat] == 0) {
        	matrixMax[pat] = 1.0;
            scalingFactors[pattern] = 0.0;
        } else {
            scalingFactors[pattern] = log(matrixMax[pat]);
        }
    }
    KW_LOCAL_FENCE;
    if (matrix < matrixCount)
        allPartials[partialsOffset + tx] = storedPartials[matrix][tx] / matrixMax[pat];
}

// Find a scaling factor for each pattern and accumulate into buffer
KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingAccumulate(KW_GLOBAL_VAR REAL* allPartials,
                                                             KW_GLOBAL_VAR REAL* scalingFactors,
                                                             KW_GLOBAL_VAR REAL* cumulativeScaling,
                                                             int matrixCount) {
    FIND_MAX_PARTIALS_STATE_GPU();
    if (state == 0 && matrix == 0) {
        FIND_MAX_PARTIALS_MATRIX_GPU();        
        if (matrixMax[pat] == 0)
        	matrixMax[pat] = 1.0;
        scalingFactors[pattern] = matrixMax[pat]; 
        cumulativeScaling[pattern] += log(matrixMax[pat]);
    }
    KW_LOCAL_FENCE;
    if (matrix < matrixCount)
        allPartials[partialsOffset + tx] = storedPartials[matrix][tx] / matrixMax[pat];
}

KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingAccumulateScalersLog(KW_GLOBAL_VAR REAL* allPartials,
                                                                       KW_GLOBAL_VAR REAL* scalingFactors,
                                                                       KW_GLOBAL_VAR REAL* cumulativeScaling,
                                                                       int matrixCount) {
    FIND_MAX_PARTIALS_STATE_GPU();
    if (state == 0 && matrix == 0) {
        FIND_MAX_PARTIALS_MATRIX_GPU();
        if (matrixMax[pat] == 0) {
        	matrixMax[pat] = 1.0;
            scalingFactors[pattern] = 0.0;
        } else {
            REAL logMax = log(matrixMax[pat]);
            scalingFactors[pattern] = logMax;
            cumulativeScaling[pattern] += logMax; // TODO: Fix, this is both a read and write
        }
    }
    KW_LOCAL_FENCE;
    if (matrix < matrixCount)
        allPartials[partialsOffset + tx] = storedPartials[matrix][tx] / matrixMax[pat];        
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoods(KW_GLOBAL_VAR REAL* dResult,
                                                 KW_GLOBAL_VAR REAL* dRootPartials,
                                                 KW_GLOBAL_VAR REAL* dWeights,
                                                 KW_GLOBAL_VAR REAL* dFrequencies,
                                                 int matrixCount,
                                                 int patternCount) {
#ifdef FW_OPENCL_CPU
    INTEGRATE_PARTIALS_CPU();
    dResult[pattern] = log(sum[0] + sum[1] + sum[2] + sum[3]);
#else
    INTEGRATE_PARTIALS_GPU();
    if (state == 0)
        dResult[pattern] = log(sum[pat][state]);
#endif
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsFixedScale(KW_GLOBAL_VAR REAL* dResult,
                                                           KW_GLOBAL_VAR REAL* dRootPartials,
                                                           KW_GLOBAL_VAR REAL* dWeights,
                                                           KW_GLOBAL_VAR REAL* dFrequencies,
                                                           KW_GLOBAL_VAR REAL* dRootScalingFactors,
                                                           int matrixCount,
                                                           int patternCount) {
    INTEGRATE_PARTIALS_GPU();    
    if (state == 0)
        dResult[pattern] = (log(sum[pat][state]) + dRootScalingFactors[pattern]);
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsMulti(KW_GLOBAL_VAR REAL* dResult,
                                                      KW_GLOBAL_VAR REAL* dRootPartials,
                                                      KW_GLOBAL_VAR REAL* dWeights,
                                                      KW_GLOBAL_VAR REAL* dFrequencies,
                                                      int matrixCount,
                                                      int patternCount,
											          int takeLog) {
    INTEGRATE_PARTIALS_GPU();    
    if (state == 0) {
		if (takeLog == 0)
			dResult[pattern] = sum[pat][state];
		else if (takeLog == 1)
			dResult[pattern] = log(dResult[pattern] + sum[pat][state]);
		else 
			dResult[pattern] += sum[pat][state];
	}
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsFixedScaleMulti(KW_GLOBAL_VAR REAL* dResult,
											                    KW_GLOBAL_VAR REAL* dRootPartials,
                                                                KW_GLOBAL_VAR REAL* dWeights,
                                                                KW_GLOBAL_VAR REAL* dFrequencies,
                                                                KW_GLOBAL_VAR REAL* dScalingFactors,
											                    KW_GLOBAL_VAR unsigned int* dPtrQueue,
											                    KW_GLOBAL_VAR REAL* dMaxScalingFactors,
											                    KW_GLOBAL_VAR unsigned int* dIndexMaxScalingFactors,
                                                                int matrixCount,
                                                                int patternCount,
											                    int subsetCount,
											                    int subsetIndex) {
    INTEGRATE_PARTIALS_GPU();
	REAL cumulativeScalingFactor = (dScalingFactors + dPtrQueue[subsetIndex])[pattern];
	if (subsetIndex == 0) {
		int indexMaxScalingFactor = 0;
		REAL maxScalingFactor = cumulativeScalingFactor;
		for (int j = 1; j < subsetCount; j++) {
			REAL tmpScalingFactor = (dScalingFactors + dPtrQueue[j])[pattern];
			if (tmpScalingFactor > maxScalingFactor) {
				indexMaxScalingFactor = j;
				maxScalingFactor = tmpScalingFactor;
			}
		}
		dIndexMaxScalingFactors[pattern] = indexMaxScalingFactor;
		dMaxScalingFactors[pattern] = maxScalingFactor;	
		if (indexMaxScalingFactor != 0)
			sum[pat][state] *= exp((REAL)(cumulativeScalingFactor - maxScalingFactor));
		if (state == 0)
			dResult[pattern] = sum[pat][state];
#ifdef FW_OPENCL
        KW_LOCAL_FENCE;
#endif
	} else {
		if (subsetIndex != dIndexMaxScalingFactors[pattern])
			sum[pat][state] *= exp((REAL)(cumulativeScalingFactor - dMaxScalingFactors[pattern]));
		if (state == 0) {
			if (subsetIndex == subsetCount - 1)
				dResult[pattern] = (log(dResult[pattern] + sum[pat][state]) + dMaxScalingFactors[pattern]);
			else
				dResult[pattern] += sum[pat][state];
		}
	}        
}

////////////////////////////////////////////////////////////////////////////////////////////////
// max likelihood kernels

KW_GLOBAL_KERNEL void kernelPartialsPartialsEdgeLikelihoods(KW_GLOBAL_VAR REAL* dPartialsTmp,
                                                          KW_GLOBAL_VAR REAL* dParentPartials,
                                                          KW_GLOBAL_VAR REAL* dChildParials,
                                                          KW_GLOBAL_VAR REAL* dTransMatrix,
                                                          int totalPatterns) {
   REAL sum1 = 0;

    int i;

    DETERMINE_INDICES_4();
    int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);
    int y = deltaPartialsByState + deltaPartialsByMatrix;
    KW_GLOBAL_VAR REAL* matrix1 = dTransMatrix + x2; // Points to *this* matrix

#ifdef KERNEL_PRINT_ENABLED
    printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
           state, u);
#endif

    // Load values into shared memory
    KW_LOCAL_MEM REAL sMatrix1[16];

    KW_LOCAL_MEM REAL sPartials1[PATTERN_BLOCK_SIZE * 4 * 4];
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

    // copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        sPartials1[multBy16(patIdx) | tx] = dParentPartials[y | tx]; // All coalesced memory reads
        sPartials2[multBy16(patIdx) | tx] = dChildParials  [y | tx];
    } else {
        sPartials1[multBy16(patIdx) | tx] = 0;
        sPartials2[multBy16(patIdx) | tx] = 0;
    }

    if (patIdx == 0 ) {
        sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
    }

    KW_LOCAL_FENCE;

    if (pattern < totalPatterns) { // Remove padded threads!

        i = pat;
        sum1  = sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
        i = (++i) & 0x3;
        FMA(sMatrix1[multBy4(i) | state], sPartials1[patIdx16pat4 | i], sum1);
        i = (++i) & 0x3;
        FMA(sMatrix1[multBy4(i) | state], sPartials1[patIdx16pat4 | i], sum1);
        i = (++i) & 0x3;
        FMA(sMatrix1[multBy4(i) | state], sPartials1[patIdx16pat4 | i], sum1);
        
        dPartialsTmp[u] = sum1 * sPartials2[patIdx16pat4 | state];
    }    

}



KW_GLOBAL_KERNEL void kernelPartialsPartialsEdgeLikelihoodsSecondDeriv(KW_GLOBAL_VAR REAL* dPartialsTmp,
                                                              KW_GLOBAL_VAR REAL* dFirstDerivTmp,
                                                              KW_GLOBAL_VAR REAL* dSecondDerivTmp,
                                                              KW_GLOBAL_VAR REAL* dParentPartials,
                                                              KW_GLOBAL_VAR REAL* dChildParials,
                                                              KW_GLOBAL_VAR REAL* dTransMatrix,
                                                              KW_GLOBAL_VAR REAL* dFirstDerivMatrix,
                                                              KW_GLOBAL_VAR REAL* dSecondDerivMatrix,
                                                              int totalPatterns) {
       REAL sum1 = 0;
       REAL sumFirstDeriv = 0;
       REAL sumSecondDeriv = 0;

        int i;

        DETERMINE_INDICES_4();
        int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);
        int y = deltaPartialsByState + deltaPartialsByMatrix;
        KW_GLOBAL_VAR REAL* matrix1 = dTransMatrix + x2; // Points to *this* matrix
        KW_GLOBAL_VAR REAL* matrixFirstDeriv = dFirstDerivMatrix + x2;
        KW_GLOBAL_VAR REAL* matrixSecondDeriv = dSecondDerivMatrix + x2;

    #ifdef KERNEL_PRINT_ENABLED
        printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
               state, u);
    #endif

        // Load values into shared memory
        KW_LOCAL_MEM REAL sMatrix1[16];
        KW_LOCAL_MEM REAL sMatrixFirstDeriv[16];
        KW_LOCAL_MEM REAL sMatrixSecondDeriv[16];

        KW_LOCAL_MEM REAL sPartials1[PATTERN_BLOCK_SIZE * 4 * 4];
        KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

        // copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials
        if (pattern < totalPatterns) {
            sPartials1[multBy16(patIdx) | tx] = dParentPartials[y | tx]; // All coalesced memory reads
            sPartials2[multBy16(patIdx) | tx] = dChildParials  [y | tx];
        } else {
            sPartials1[multBy16(patIdx) | tx] = 0;
            sPartials2[multBy16(patIdx) | tx] = 0;
        }

        if (patIdx == 0 ) {
            sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
            sMatrixFirstDeriv[tx] = matrixFirstDeriv[tx];
            sMatrixSecondDeriv[tx] = matrixSecondDeriv[tx];
        }

        KW_LOCAL_FENCE;

        if (pattern < totalPatterns) { // Remove padded threads!

            i = pat;
            sum1  = sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
            sumFirstDeriv  = sMatrixFirstDeriv[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
            sumSecondDeriv  = sMatrixSecondDeriv[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
            i = (++i) & 0x3;
            FMA(sMatrix1[multBy4(i) | state], sPartials1[patIdx16pat4 | i], sum1);
            FMA(sMatrixFirstDeriv[multBy4(i) | state], sPartials1[patIdx16pat4 | i], sumFirstDeriv);
            FMA(sMatrixSecondDeriv[multBy4(i) | state], sPartials1[patIdx16pat4 | i], sumSecondDeriv);
            i = (++i) & 0x3;
            FMA(sMatrix1[multBy4(i) | state], sPartials1[patIdx16pat4 | i], sum1);
            FMA(sMatrixFirstDeriv[multBy4(i) | state], sPartials1[patIdx16pat4 | i], sumFirstDeriv);
            FMA(sMatrixSecondDeriv[multBy4(i) | state], sPartials1[patIdx16pat4 | i], sumSecondDeriv);
            i = (++i) & 0x3;
            FMA(sMatrix1[multBy4(i) | state], sPartials1[patIdx16pat4 | i], sum1);
            FMA(sMatrixFirstDeriv[multBy4(i) | state], sPartials1[patIdx16pat4 | i], sumFirstDeriv);
            FMA(sMatrixSecondDeriv[multBy4(i) | state], sPartials1[patIdx16pat4 | i], sumSecondDeriv);
            
            dPartialsTmp[u] = sum1 * sPartials2[patIdx16pat4 | state];
            dFirstDerivTmp[u] = sumFirstDeriv * sPartials2[patIdx16pat4 | state];
            dSecondDerivTmp[u] = sumSecondDeriv * sPartials2[patIdx16pat4 | state];
        }    

    }


KW_GLOBAL_KERNEL void kernelStatesPartialsEdgeLikelihoods(KW_GLOBAL_VAR REAL* dPartialsTmp,
                                                         KW_GLOBAL_VAR REAL* dParentPartials,
                                                         KW_GLOBAL_VAR int* dChildStates,
                                                         KW_GLOBAL_VAR REAL* dTransMatrix,
                                                         int totalPatterns) {
    REAL sum1 = 0;

    DETERMINE_INDICES_4();
    int y = deltaPartialsByState + deltaPartialsByMatrix;
    KW_GLOBAL_VAR REAL* matrix1 = dTransMatrix + x2; // Points to *this* matrix
    
#ifdef KERNEL_PRINT_ENABLED
    printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
           state, u);
#endif

    // Load values into shared memory
    KW_LOCAL_MEM REAL sMatrix1[16];

    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

    // copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        sPartials2[patIdx * 16 + tx] = dParentPartials[y + tx];
    } else {
        sPartials2[patIdx * 16 + tx] = 0;
    }

    if (patIdx == 0) {
        sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
    }

    KW_LOCAL_FENCE;

    if (pattern < totalPatterns) { // Remove padded threads!
        int state1 = dChildStates[pattern];

        if (state1 < PADDED_STATE_COUNT)
            sum1 = sMatrix1[state1 * 4 + state];
        else
            sum1 = 1.0;

        dPartialsTmp[u] = sum1 * sPartials2[patIdx * 16 + pat * 4 + state];
    }
}

KW_GLOBAL_KERNEL void kernelStatesPartialsEdgeLikelihoodsSecondDeriv(KW_GLOBAL_VAR REAL* dPartialsTmp,
                                                              KW_GLOBAL_VAR REAL* dFirstDerivTmp,
                                                              KW_GLOBAL_VAR REAL* dSecondDerivTmp,
                                                              KW_GLOBAL_VAR REAL* dParentPartials,
                                                              KW_GLOBAL_VAR int* dChildStates,
                                                              KW_GLOBAL_VAR REAL* dTransMatrix,
                                                              KW_GLOBAL_VAR REAL* dFirstDerivMatrix,
                                                              KW_GLOBAL_VAR REAL* dSecondDerivMatrix,
                                                              int totalPatterns) {
    REAL sum1 = 0;
    REAL sumFirstDeriv = 0;
    REAL sumSecondDeriv = 0;


    DETERMINE_INDICES_4();
    int y = deltaPartialsByState + deltaPartialsByMatrix;
    KW_GLOBAL_VAR REAL* matrix1 = dTransMatrix + x2; // Points to *this* matrix
    KW_GLOBAL_VAR REAL* matrixFirstDeriv = dFirstDerivMatrix + x2;
    KW_GLOBAL_VAR REAL* matrixSecondDeriv = dSecondDerivMatrix + x2;

    
#ifdef KERNEL_PRINT_ENABLED
    printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
           state, u);
#endif

    // Load values into shared memory
    KW_LOCAL_MEM REAL sMatrix1[16];
    KW_LOCAL_MEM REAL sMatrixFirstDeriv[16];
    KW_LOCAL_MEM REAL sMatrixSecondDeriv[16];

    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

    // copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        sPartials2[patIdx * 16 + tx] = dParentPartials[y + tx];
    } else {
        sPartials2[patIdx * 16 + tx] = 0;
    }

    if (patIdx == 0) {
        sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
        sMatrixFirstDeriv[tx] = matrixFirstDeriv[tx]; // All coalesced memory reads
        sMatrixSecondDeriv[tx] = matrixSecondDeriv[tx]; // All coalesced memory reads
    }

    KW_LOCAL_FENCE;

    if (pattern < totalPatterns) { // Remove padded threads!
        int state1 = dChildStates[pattern];

        if (state1 < PADDED_STATE_COUNT) {
            sum1 = sMatrix1[state1 * 4 + state];
            sumFirstDeriv = sMatrixFirstDeriv[state1 * 4 + state];
            sumSecondDeriv = sMatrixSecondDeriv[state1 * 4 + state];
        } else {
            sum1 = 1.0;
            sumFirstDeriv = 0.0;
            sumSecondDeriv = 0.0;
        }

        dPartialsTmp[u] = sum1 * sPartials2[patIdx * 16 + pat * 4 + state];
        dFirstDerivTmp[u] = sumFirstDeriv * sPartials2[patIdx * 16 + pat * 4 + state];
        dSecondDerivTmp[u] = sumSecondDeriv * sPartials2[patIdx * 16 + pat * 4 + state];
    }
}


KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsFixedScaleSecondDeriv(KW_GLOBAL_VAR REAL* dResult,
                                              KW_GLOBAL_VAR REAL* dFirstDerivResult,
                                              KW_GLOBAL_VAR REAL* dSecondDerivResult,
                                              KW_GLOBAL_VAR REAL* dRootPartials,
                                              KW_GLOBAL_VAR REAL* dRootFirstDeriv,
                                              KW_GLOBAL_VAR REAL* dRootSecondDeriv,
                                              KW_GLOBAL_VAR REAL* dWeights,
                                              KW_GLOBAL_VAR REAL* dFrequencies,
                                              KW_GLOBAL_VAR REAL* dRootScalingFactors,
                                              int matrixCount,
                                              int patternCount) {
    int state   = KW_LOCAL_ID_0;
    int pat = KW_LOCAL_ID_1;
    int pattern = KW_GROUP_ID_0 * LIKE_PATTERN_BLOCK_SIZE + KW_LOCAL_ID_1;
    
    REAL tmpLogLike = 0.0;
    REAL tmpFirstDeriv = 0.0;
    
    KW_LOCAL_MEM REAL stateFreq[4];
    
    // TODO: Currently assumes MATRIX_BLOCK_SIZE >= matrixCount
    KW_LOCAL_MEM REAL matrixProp[MATRIX_BLOCK_SIZE];
    KW_LOCAL_MEM REAL sum[LIKE_PATTERN_BLOCK_SIZE][4];
    KW_LOCAL_MEM REAL sumD1[LIKE_PATTERN_BLOCK_SIZE][4];
    KW_LOCAL_MEM REAL sumD2[LIKE_PATTERN_BLOCK_SIZE][4];

    // Load shared memory

    if (pat == 0) {
        stateFreq[state] = dFrequencies[state];
    }
    
    sum[pat][state] = 0;
    sumD1[pat][state] = 0;
    sumD2[pat][state] = 0;
    
    // TODO: Assumes matrixCount < LIKE_PATTERN_BLOCK_SIZE * 4
    if (pat * 4 + state < matrixCount) {
        matrixProp[pat * 4 + state] = dWeights[pat * 4 + state];
    }

    KW_LOCAL_FENCE;

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;;

    for(int r = 0; r < matrixCount; r++) {
        FMA(dRootPartials[u + delta * r],    matrixProp[r], sum[pat][state]);
        FMA(dRootFirstDeriv[u + delta * r] , matrixProp[r], sumD1[pat][state]);
        FMA(dRootSecondDeriv[u + delta * r], matrixProp[r], sumD2[pat][state]);
    }

    sum[pat][state] *= stateFreq[state];
    sumD1[pat][state] *= stateFreq[state];
    sumD2[pat][state] *= stateFreq[state];
    KW_LOCAL_FENCE;
    if (state < 2) {
        sum[pat][state] += sum[pat][state + 2];
        sumD1[pat][state] += sumD1[pat][state + 2];
        sumD2[pat][state] += sumD2[pat][state + 2];
    }
    KW_LOCAL_FENCE;
    if (state < 1) {
        sum[pat][state] += sum[pat][state + 1];
        sumD1[pat][state] += sumD1[pat][state + 1];
        sumD2[pat][state] += sumD2[pat][state + 1];
    }
    
    if (state == 0) {
        tmpLogLike = sum[pat][state];
        dResult[pattern] = (log(tmpLogLike) + dRootScalingFactors[pattern]);
        
        tmpFirstDeriv = sumD1[pat][state] / tmpLogLike;
        dFirstDerivResult[pattern] = tmpFirstDeriv;
        
        dSecondDerivResult[pattern] = (sumD2[pat][state] / tmpLogLike - tmpFirstDeriv * tmpFirstDeriv);
    }
}


KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsSecondDeriv(KW_GLOBAL_VAR REAL* dResult,
                                              KW_GLOBAL_VAR REAL* dFirstDerivResult,
                                              KW_GLOBAL_VAR REAL* dSecondDerivResult,
                                              KW_GLOBAL_VAR REAL* dRootPartials,
                                              KW_GLOBAL_VAR REAL* dRootFirstDeriv,
                                              KW_GLOBAL_VAR REAL* dRootSecondDeriv,
                                              KW_GLOBAL_VAR REAL* dWeights,
                                              KW_GLOBAL_VAR REAL* dFrequencies,
                                              int matrixCount,
                                              int patternCount) {
    int state   = KW_LOCAL_ID_0;
    int pat = KW_LOCAL_ID_1;
    int pattern = KW_GROUP_ID_0 * LIKE_PATTERN_BLOCK_SIZE + KW_LOCAL_ID_1;
    
    REAL tmpLogLike = 0.0;
    REAL tmpFirstDeriv = 0.0;
    
    KW_LOCAL_MEM REAL stateFreq[4];
    
    // TODO: Currently assumes MATRIX_BLOCK_SIZE >= matrixCount
    KW_LOCAL_MEM REAL matrixProp[MATRIX_BLOCK_SIZE];
    KW_LOCAL_MEM REAL sum[LIKE_PATTERN_BLOCK_SIZE][4];
    KW_LOCAL_MEM REAL sumD1[LIKE_PATTERN_BLOCK_SIZE][4];
    KW_LOCAL_MEM REAL sumD2[LIKE_PATTERN_BLOCK_SIZE][4];

    // Load shared memory

    if (pat == 0) {
        stateFreq[state] = dFrequencies[state];
    }
    
    sum[pat][state] = 0;
    sumD1[pat][state] = 0;
    sumD2[pat][state] = 0;
    
    // TODO: Assumes matrixCount < LIKE_PATTERN_BLOCK_SIZE * 4
    if (pat * 4 + state < matrixCount) {
        matrixProp[pat * 4 + state] = dWeights[pat * 4 + state];
    }

    KW_LOCAL_FENCE;

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;;

    for(int r = 0; r < matrixCount; r++) {
        FMA(dRootPartials[u + delta * r],    matrixProp[r], sum[pat][state]);
        FMA(dRootFirstDeriv[u + delta * r] , matrixProp[r], sumD1[pat][state]);
        FMA(dRootSecondDeriv[u + delta * r], matrixProp[r], sumD2[pat][state]);
    }

    sum[pat][state] *= stateFreq[state];
    sumD1[pat][state] *= stateFreq[state];
    sumD2[pat][state] *= stateFreq[state];
    KW_LOCAL_FENCE;
    if (state < 2) {
        sum[pat][state] += sum[pat][state + 2];
        sumD1[pat][state] += sumD1[pat][state + 2];
        sumD2[pat][state] += sumD2[pat][state + 2];
    }
    KW_LOCAL_FENCE;
    if (state < 1) {
        sum[pat][state] += sum[pat][state + 1];
        sumD1[pat][state] += sumD1[pat][state + 1];
        sumD2[pat][state] += sumD2[pat][state + 1];
    }
    
    if (state == 0) {
        tmpLogLike = sum[pat][state];
        dResult[pattern] = log(tmpLogLike);
        
        tmpFirstDeriv = sumD1[pat][state] / tmpLogLike;
        dFirstDerivResult[pattern] = tmpFirstDeriv;
        
        dSecondDerivResult[pattern] = (sumD2[pat][state] / tmpLogLike - tmpFirstDeriv * tmpFirstDeriv);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////
// scaling experiments kernels

KW_GLOBAL_KERNEL void kernelPartialsPartialsCheckScale(KW_GLOBAL_VAR REAL* partials1,
                                                                  KW_GLOBAL_VAR REAL* partials2,
                                                                  KW_GLOBAL_VAR REAL* partials3,
                                                                  KW_GLOBAL_VAR REAL* matrices1,
                                                                  KW_GLOBAL_VAR REAL* matrices2,
                                                                  KW_GLOBAL_VAR int* dRescalingTrigger,
                                                                  int totalPatterns) {
        REAL sum1;
        REAL sum2;
        int i;

        DETERMINE_INDICES_4();

        int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);
        int y = deltaPartialsByState + deltaPartialsByMatrix;
        
        KW_GLOBAL_VAR REAL* matrix1 = matrices1 + x2; // Points to *this* matrix
        KW_GLOBAL_VAR REAL* matrix2 = matrices2 + x2;

    #ifdef KERNEL_PRINT_ENABLED
        printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
               state, u);
    #endif

        // Load values into shared memory
        KW_LOCAL_MEM REAL sMatrix1[16];
        KW_LOCAL_MEM REAL sMatrix2[16];

        KW_LOCAL_MEM REAL sPartials1[PATTERN_BLOCK_SIZE * 4 * 4];
        KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

        // copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials
        if (pattern < totalPatterns) {
            sPartials1[multBy16(patIdx) | tx] = partials1[y | tx]; // All coalesced memory reads
            sPartials2[multBy16(patIdx) | tx] = partials2[y | tx];
        } else {
            sPartials1[multBy16(patIdx) | tx] = 0;
            sPartials2[multBy16(patIdx) | tx] = 0;
        }

        if (patIdx == 0 ) {
            sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
            sMatrix2[tx] = matrix2[tx];
        }

        KW_LOCAL_FENCE;

        if (pattern < totalPatterns) { // Remove padded threads!

            i = pat;
            sum1  = sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
            sum2  = sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

            i = (++i) & 0x3;
            sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
            sum2 += sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

            i = (++i) & 0x3;
            sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
            sum2 += sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

            i = (++i) & 0x3;
            sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
            sum2 += sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];
            
            REAL tmpPartial = sum1 * sum2;
            
            partials3[u] = tmpPartial;

            if (tmpPartial < SCALING_THRESHOLD_LOWER || tmpPartial > SCALING_THRESHOLD_UPPER)
                *dRescalingTrigger = 1;
            
//            union {float f; long l;} fl;
//            fl.f = sum1 * sum2;;
//
//          partials3[u] = fl.f;
//            
//            int expTmp  = ((fl.l >> 23) & 0x000000ff) - 0x7e;
//            
//            if (abs(expTmp) > SCALING_EXPONENT_THRESHOLD)
//                *dRescalingTrigger = 1;
        }

    }

KW_GLOBAL_KERNEL void kernelPartialsPartialsFixedCheckScale(KW_GLOBAL_VAR REAL* partials1,
                                                      KW_GLOBAL_VAR REAL* partials2,
                                                      KW_GLOBAL_VAR REAL* partials3,
                                                      KW_GLOBAL_VAR REAL* matrices1,
                                                      KW_GLOBAL_VAR REAL* matrices2,
                                                      KW_GLOBAL_VAR REAL* scalingFactors,
                                                      KW_GLOBAL_VAR int* dRescalingTrigger,
                                                      int totalPatterns) {
    REAL sum1;
    REAL sum2;
    int i;

    DETERMINE_INDICES_4();
    int y = deltaPartialsByState + deltaPartialsByMatrix;
    KW_GLOBAL_VAR REAL* matrix1 = matrices1 + x2; // Points to *this* matrix
    KW_GLOBAL_VAR REAL* matrix2 = matrices2 + x2;

#ifdef KERNEL_PRINT_ENABLED
    printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
           state, u);
#endif

    // Load values into shared memory
    KW_LOCAL_MEM REAL sMatrix1[16];
    KW_LOCAL_MEM REAL sMatrix2[16];

    KW_LOCAL_MEM REAL sPartials1[PATTERN_BLOCK_SIZE * 4 * 4];
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

    KW_LOCAL_MEM REAL fixedScalingFactors[PATTERN_BLOCK_SIZE * 4];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        sPartials1[patIdx * 16 + tx] = partials1[y + tx]; // All coalesced memory reads
        sPartials2[patIdx * 16 + tx] = partials2[y + tx];
    } else {
        sPartials1[patIdx * 16 + tx] = 0;
        sPartials2[patIdx * 16 + tx] = 0;
    }

    if (patIdx < 4) // need to load 4*PATTERN_BLOCK_SIZE factors for this block
        fixedScalingFactors[patIdx * PATTERN_BLOCK_SIZE + tx] =
            scalingFactors[KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE * 4 + patIdx * PATTERN_BLOCK_SIZE + tx];

    if (patIdx == 0 ) {
        sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
        sMatrix2[tx] = matrix2[tx];
    }

    KW_LOCAL_FENCE;

    if (pattern < totalPatterns) { // Remove padded threads!

        i = pat;
        sum1  = sMatrix1[i * 4 + state] * sPartials1[patIdx * 16 + pat * 4 + i];
        sum2  = sMatrix2[i * 4 + state] * sPartials2[patIdx * 16 + pat * 4 + i];

        i = (++i) & 0x3;
        sum1 += sMatrix1[i * 4 + state] * sPartials1[patIdx * 16 + pat * 4 + i];
        sum2 += sMatrix2[i * 4 + state] * sPartials2[patIdx * 16 + pat * 4 + i];

        i = (++i) & 0x3;
        sum1 += sMatrix1[i * 4 + state] * sPartials1[patIdx * 16 + pat * 4 + i];
        sum2 += sMatrix2[i * 4 + state] * sPartials2[patIdx * 16 + pat * 4 + i];

        i = (++i) & 0x3;
        sum1 += sMatrix1[i * 4 + state] * sPartials1[patIdx * 16 + pat * 4 + i];
        sum2 += sMatrix2[i * 4 + state] * sPartials2[patIdx * 16 + pat * 4 + i];
        
        REAL tmpPartial = sum1 * sum2 * fixedScalingFactors[patIdx * 4 + pat];
        
        partials3[u] = tmpPartial;

        if (tmpPartial < SCALING_THRESHOLD_LOWER || tmpPartial > SCALING_THRESHOLD_UPPER)
            *dRescalingTrigger = 1;

    }

}

KW_GLOBAL_KERNEL void kernelPartialsPartialsAutoScale(KW_GLOBAL_VAR REAL* partials1,
                                                KW_GLOBAL_VAR REAL* partials2,
                                                KW_GLOBAL_VAR REAL* partials3,
                                                KW_GLOBAL_VAR REAL* matrices1,
                                                KW_GLOBAL_VAR REAL* matrices2,
                                                KW_GLOBAL_VAR signed char* scalingFactors,
                                                int totalPatterns) {
    REAL sum1;
    REAL sum2;
    int i;

    DETERMINE_INDICES_4();

    int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);
    int y = deltaPartialsByState + deltaPartialsByMatrix;
    int myIdx = multBy16(patIdx) + tx; // threadId in block
    
    KW_GLOBAL_VAR REAL* matrix1 = matrices1 + x2; // Points to *this* matrix
    KW_GLOBAL_VAR REAL* matrix2 = matrices2 + x2;

#ifdef KERNEL_PRINT_ENABLED
    printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
           state, u);
#endif

    // Load values into shared memory
    KW_LOCAL_MEM REAL sMatrix1[16];
    KW_LOCAL_MEM REAL sMatrix2[16];

    KW_LOCAL_MEM REAL sPartials1[PATTERN_BLOCK_SIZE * 4 * 4];
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

    // copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        sPartials1[multBy16(patIdx) | tx] = partials1[y | tx]; // All coalesced memory reads
        sPartials2[multBy16(patIdx) | tx] = partials2[y | tx];
    } else {
        sPartials1[multBy16(patIdx) | tx] = 0;
        sPartials2[multBy16(patIdx) | tx] = 0;
    }

    if (patIdx == 0 ) {
        sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
        sMatrix2[tx] = matrix2[tx];
    }

    KW_LOCAL_FENCE;

    i = pat;
    sum1  = sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
    sum2  = sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

    i = (++i) & 0x3;
    sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
    sum2 += sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

    i = (++i) & 0x3;
    sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
    sum2 += sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

    i = (++i) & 0x3;
    sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
    sum2 += sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];
    
    REAL tmpPartial = sum1 * sum2;
    int expTmp;
    REAL sigTmp = frexp(tmpPartial, &expTmp);        

    KW_LOCAL_FENCE;
    
    if (pattern < totalPatterns) {
        if (abs(expTmp) > SCALING_EXPONENT_THRESHOLD) {
            // now using sPartials2 to hold scaling trigger boolean
            sPartials2[patIdx16pat4] = 1;
        } else {
            partials3[u] = tmpPartial;
            sPartials2[patIdx16pat4] = 0;
            sPartials1[myIdx] = 0;
        }
    } 
    
    KW_LOCAL_FENCE;
    
    int scalingActive = sPartials2[patIdx16pat4];
        
    if (scalingActive) {
        // now using sPartials1 to store max unscaled partials3
        sPartials1[myIdx] = tmpPartial;
    }
        
    KW_LOCAL_FENCE;
        
    // Unrolled parallel max-reduction
    if (scalingActive && state < 2) {
        REAL compare = sPartials1[myIdx + 2];
        if (compare >  sPartials1[myIdx])
            sPartials1[myIdx] = compare;
    }
     
    KW_LOCAL_FENCE;
            
    if (scalingActive && state < 1) {
        REAL maxPartial = sPartials1[myIdx + 1];
        if (maxPartial < sPartials1[myIdx])
            maxPartial = sPartials1[myIdx];
        int expMax;
        frexp(maxPartial, &expMax);
        sPartials1[myIdx] = expMax;
    }

    KW_LOCAL_FENCE;
            
    if (scalingActive) 
        partials3[u] = ldexp(sigTmp, expTmp - sPartials1[patIdx16pat4]);
        
    if ((myIdx < PATTERN_BLOCK_SIZE * 4) && (myIdx + (KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE * 4) < totalPatterns))
        scalingFactors[(KW_GROUP_ID_0 * PATTERN_BLOCK_SIZE * 4) + (matrix * totalPatterns) + myIdx] = sPartials1[multBy4(myIdx)];
}


KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingAccumulateReciprocal(KW_GLOBAL_VAR REAL* allPartials,
                                                       KW_GLOBAL_VAR REAL* scalingFactors,
                                                       KW_GLOBAL_VAR REAL* cumulativeScaling,
                                                       int matrixCount) {
    int tx = KW_LOCAL_ID_0;
    
    int state = tx & 0x3;
    int pat = tx >> 2;
                             
    int patIdx = KW_GROUP_ID_0;
    
    int pattern = (patIdx << 2) + pat;
    int matrix = KW_LOCAL_ID_1;
    // TODO: Assumes matrixCount < MATRIX_BLOCK_SIZ
    
    // Patterns are always padded, so no reading/writing past end possible
    // Find start of patternBlock for thread-block
    int partialsOffset = (matrix * KW_NUM_GROUPS_0 + patIdx) << 4; //* 16;

    KW_LOCAL_MEM REAL partials[MATRIX_BLOCK_SIZE][16]; // 4 patterns at a time
    KW_LOCAL_MEM REAL storedPartials[MATRIX_BLOCK_SIZE][16];

    KW_LOCAL_MEM REAL matrixMax[4];
    
    if (matrix < matrixCount)
        partials[matrix][tx] = allPartials[partialsOffset + tx];          

    storedPartials[matrix][tx] = partials[matrix][tx];
           
    KW_LOCAL_FENCE;
    
    // Unrolled parallel max-reduction
    if (state < 2) {
        REAL compare1 = partials[matrix][tx];
        REAL compare2 = partials[matrix][tx + 2];
        if (compare2 > compare1)
            partials[matrix][tx] = compare2;
    }
    KW_LOCAL_FENCE;
    
    if (state < 1) {
        REAL compare1 = partials[matrix][tx];
        REAL compare2 = partials[matrix][tx + 1];
        if (compare2 > compare1)
            partials[matrix][tx] = compare2;
    }
    KW_LOCAL_FENCE;
 
    // Could also parallel-reduce here.
    if (state == 0 && matrix == 0) {
        matrixMax[pat] = 0;
        int m;
        for(m = 0; m < matrixCount; m++) {
            if (partials[m][tx] > matrixMax[pat])
                matrixMax[pat] = partials[m][tx];
        }
        
        if (matrixMax[pat] == 0)
            matrixMax[pat] = 1.0;
   
        scalingFactors[pattern] = 1/matrixMax[pat]; 
        cumulativeScaling[pattern] += log(matrixMax[pat]);
    }

    KW_LOCAL_FENCE;

    if (matrix < matrixCount)
        allPartials[partialsOffset + tx] = storedPartials[matrix][tx] / matrixMax[pat];
        
}

KW_GLOBAL_KERNEL void kernelPartialsDynamicScalingAccumulateDifference(KW_GLOBAL_VAR REAL* allPartials,
                                                                 KW_GLOBAL_VAR REAL* scalingFactors,
                                                                 KW_GLOBAL_VAR REAL* existingScalingFactors,
                                                                 KW_GLOBAL_VAR REAL* cumulativeScaling,
                                                                 int matrixCount) {
    int tx = KW_LOCAL_ID_0;
    
    int state = tx & 0x3;
    int pat = tx >> 2;
                             
    int patIdx = KW_GROUP_ID_0;
    
    int pattern = (patIdx << 2) + pat;
    int matrix = KW_LOCAL_ID_1;
    // TODO: Assumes matrixCount < MATRIX_BLOCK_SIZ
    
    // Patterns are always padded, so no reading/writing past end possible
    // Find start of patternBlock for thread-block
    int partialsOffset = (matrix * KW_NUM_GROUPS_0 + patIdx) << 4; //* 16;

    KW_LOCAL_MEM REAL partials[MATRIX_BLOCK_SIZE][16]; // 4 patterns at a time
    KW_LOCAL_MEM REAL storedPartials[MATRIX_BLOCK_SIZE][16];

    KW_LOCAL_MEM REAL matrixMax[4];
    
    if (matrix < matrixCount)
        partials[matrix][tx] = allPartials[partialsOffset + tx];          

    storedPartials[matrix][tx] = partials[matrix][tx];
           
    KW_LOCAL_FENCE;
    
    // Unrolled parallel max-reduction
    if (state < 2) {
        REAL compare1 = partials[matrix][tx];
        REAL compare2 = partials[matrix][tx + 2];
        if (compare2 > compare1)
            partials[matrix][tx] = compare2;
    }
    KW_LOCAL_FENCE;
    
    if (state < 1) {
        REAL compare1 = partials[matrix][tx];
        REAL compare2 = partials[matrix][tx + 1];
        if (compare2 > compare1)
            partials[matrix][tx] = compare2;
    }
    KW_LOCAL_FENCE;
 
    // Could also parallel-reduce here.
    if (state == 0 && matrix == 0) {
        matrixMax[pat] = 0;
        int m;
        for(m = 0; m < matrixCount; m++) {
            if (partials[m][tx] > matrixMax[pat])
                matrixMax[pat] = partials[m][tx];
        }
        
        if (matrixMax[pat] == 0)
            matrixMax[pat] = 1.0;
   
        REAL currentFactors = existingScalingFactors[pattern];
        scalingFactors[pattern] = 1/matrixMax[pat] * currentFactors; 
        cumulativeScaling[pattern] += (log(matrixMax[pat]));
    }

    KW_LOCAL_FENCE;

    if (matrix < matrixCount)
        allPartials[partialsOffset + tx] = storedPartials[matrix][tx] / matrixMax[pat];
        
}

KW_GLOBAL_KERNEL void kernelIntegrateLikelihoodsAutoScaling(KW_GLOBAL_VAR REAL* dResult,
                                                     KW_GLOBAL_VAR REAL* dRootPartials,
                                                     KW_GLOBAL_VAR REAL* dWeights,
                                                     KW_GLOBAL_VAR REAL* dFrequencies,
                                                     KW_GLOBAL_VAR int* dRootScalingFactors,
                                                     int matrixCount,
                                                     int patternCount) {
     int state   = KW_LOCAL_ID_0;
    int pat = KW_LOCAL_ID_1;
    int pattern = KW_GROUP_ID_0 * LIKE_PATTERN_BLOCK_SIZE + KW_LOCAL_ID_1;
    
    KW_LOCAL_MEM REAL stateFreq[4];
    
    // TODO: Currently assumes MATRIX_BLOCK_SIZE >= matrixCount
    KW_LOCAL_MEM REAL matrixProp[MATRIX_BLOCK_SIZE];
    KW_LOCAL_MEM REAL sum[LIKE_PATTERN_BLOCK_SIZE][4];

    // Load shared memory

    if (pat == 0) {
        stateFreq[state] = dFrequencies[state];
    }
    
    sum[pat][state] = 0;
    
    // TODO: Assumes matrixCount < LIKE_PATTERN_BLOCK_SIZE * 4
    if (pat * 4 + state < matrixCount) {
        matrixProp[pat * 4 + state] = dWeights[pat * 4 + state];
    }

    KW_LOCAL_FENCE;

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;

    short maxScaleFactor = dRootScalingFactors[pattern];
    for(int r = 1; r < matrixCount; r++) {
        int tmpFactor = dRootScalingFactors[pattern + (r * patternCount)];
        if (tmpFactor > maxScaleFactor)
            maxScaleFactor = tmpFactor;
    }

    for(int r = 0; r < matrixCount; r++) {
        int tmpFactor = dRootScalingFactors[pattern + (r * patternCount)];
        if (tmpFactor != maxScaleFactor) {
            // TODO: verify which of the two methods below is faster
            int expTmp;
            sum[pat][state] += ldexp(frexp(dRootPartials[u + delta * r], &expTmp), expTmp + (tmpFactor - maxScaleFactor)) * matrixProp[r];
//            sum[pat][state] += dRootPartials[u + delta * r] * pow(2.0, tmpFactor - maxScaleFactor) * matrixProp[r];
        } else {
            sum[pat][state] += dRootPartials[u + delta * r] * matrixProp[r];
        }
    }

    sum[pat][state] *= stateFreq[state];
    KW_LOCAL_FENCE;
    if (state < 2)
        sum[pat][state] += sum[pat][state + 2];
    KW_LOCAL_FENCE;
    if (state < 1) {
        sum[pat][state] += sum[pat][state + 1];
    }

    if (state == 0)
        dResult[pattern] = (log(sum[pat][state]) + (M_LN2 * maxScaleFactor));
}

#ifdef CUDA
} // extern "C"
#endif //CUDA
