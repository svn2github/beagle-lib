#include "hip/hip_runtime.h"
#define STATE_COUNT 4

/*
 *
 * Copyright 2009 Phylogenetic Likelihood Working Group
 *
 * This file is part of BEAGLE.
 *
 * BEAGLE is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * BEAGLE is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with BEAGLE.  If not, see
 * <http://www.gnu.org/licenses/>.
 *
 * @author Marc Suchard
 * @author Daniel Ayres
 * @author Andrew Rambaut
 */

#include "libhmsbeagle/GPU/GPUImplDefs.h"
#include "libhmsbeagle/GPU/kernels/kernelsAll.cu" // This file includes the non-state-count specific kernels

#define multBy4(x)	(x << 2)
#define multBy16(x)	(x << 4)

// Do not use | (instead of +) for any term involing PATTERN_BLOCK_SIZE
// as this should be adjustable
#define DETERMINE_INDICES_4() \
    int tx = threadIdx.x; \
    int state = tx & 0x3; \
    int pat = tx >> 2; \
    int patIdx = threadIdx.y; \
    int matrix = blockIdx.y; \
    int pattern = __umul24(blockIdx.x, PATTERN_BLOCK_SIZE * 4) + multBy4(patIdx) + pat; \
    int deltaPartialsByState = multBy16(blockIdx.x * PATTERN_BLOCK_SIZE + patIdx); \
    int deltaPartialsByMatrix = __umul24(matrix, multBy4(totalPatterns)); \
    int x2 = multBy16(matrix); \
    int u = tx + deltaPartialsByState + deltaPartialsByMatrix;

extern "C" {

__global__ void kernelPartialsPartialsNoScale(REAL* partials1,
                                                                  REAL* partials2,
                                                                  REAL* partials3,
                                                                  REAL* matrices1,
                                                                  REAL* matrices2,
                                                                  int totalPatterns) {
		REAL sum1;
	    REAL sum2;
	    int i;

	    DETERMINE_INDICES_4();

	    int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);
	    int y = deltaPartialsByState + deltaPartialsByMatrix;
	    
	    REAL* matrix1 = matrices1 + x2; // Points to *this* matrix
	    REAL* matrix2 = matrices2 + x2;

	#ifdef KERNEL_PRINT_ENABLED
	    printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
	           state, u);
	#endif

	    // Load values into shared memory
	    __shared__ REAL sMatrix1[16];
	    __shared__ REAL sMatrix2[16];

	    __shared__ REAL sPartials1[PATTERN_BLOCK_SIZE * 4 * 4];
	    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

	    // copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials
	    if (pattern < totalPatterns) {
	        sPartials1[multBy16(patIdx) | tx] = partials1[y | tx]; // All coalesced memory reads
	        sPartials2[multBy16(patIdx) | tx] = partials2[y | tx];
	    } else {
	        sPartials1[multBy16(patIdx) | tx] = 0;
	        sPartials2[multBy16(patIdx) | tx] = 0;
	    }

	    if (patIdx == 0 ) {
	        sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
	        sMatrix2[tx] = matrix2[tx];
	    }

	    __syncthreads();

	    if (pattern < totalPatterns) { // Remove padded threads!

 	        i = pat;
	        sum1  = sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        sum2  = sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

	        i = (++i) & 0x3;
	        sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        sum2 += sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

	        i = (++i) & 0x3;
	        sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        sum2 += sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

	        i = (++i) & 0x3;
	        sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        sum2 += sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

	        partials3[u] = sum1 * sum2;
	    }

	}
    
__global__ void kernelPartialsPartialsAutoScale(REAL* partials1,
                                                REAL* partials2,
                                                REAL* partials3,
                                                REAL* matrices1,
                                                REAL* matrices2,
                                                signed char* scalingFactors,
                                                unsigned short* activeScalingFactors,
                                                int totalPatterns) {
    REAL sum1;
    REAL sum2;
    int i;

    DETERMINE_INDICES_4();

    int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);
    int y = deltaPartialsByState + deltaPartialsByMatrix;
    int myIdx = multBy16(patIdx) + tx; // threadId in block
    
    REAL* matrix1 = matrices1 + x2; // Points to *this* matrix
    REAL* matrix2 = matrices2 + x2;

#ifdef KERNEL_PRINT_ENABLED
    printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
           state, u);
#endif

    // Load values into shared memory
    __shared__ REAL sMatrix1[16];
    __shared__ REAL sMatrix2[16];

    __shared__ REAL sPartials1[PATTERN_BLOCK_SIZE * 4 * 4];
    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

    // copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        sPartials1[multBy16(patIdx) | tx] = partials1[y | tx]; // All coalesced memory reads
        sPartials2[multBy16(patIdx) | tx] = partials2[y | tx];
    } else {
        sPartials1[multBy16(patIdx) | tx] = 0;
        sPartials2[multBy16(patIdx) | tx] = 0;
    }

    if (patIdx == 0 ) {
        sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
        sMatrix2[tx] = matrix2[tx];
    }

    __syncthreads();

    i = pat;
    sum1  = sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
    sum2  = sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

    i = (++i) & 0x3;
    sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
    sum2 += sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

    i = (++i) & 0x3;
    sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
    sum2 += sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];

    i = (++i) & 0x3;
    sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
    sum2 += sMatrix2[multBy4(i) | state] * sPartials2[patIdx16pat4 | i];
    
    REAL tmpPartial = sum1 * sum2;
    int expTmp;
    REAL sigTmp = frexp(tmpPartial, &expTmp);        

    __syncthreads();
    
    if (pattern < totalPatterns) {
        if (abs(expTmp) > SCALING_EXPONENT_THRESHOLD) {
            // now using sPartials2 to hold scaling trigger boolean
            sPartials2[patIdx16pat4] = 1;
        } else {
            partials3[u] = tmpPartial;
            sPartials2[patIdx16pat4] = 0;
            sPartials1[myIdx] = 0;
        }
    } 
    
    __syncthreads();
    
    int scalingActive = sPartials2[patIdx16pat4];
        
    if (scalingActive) {
        // now using sPartials1 to store max unscaled partials3
        sPartials1[myIdx] = tmpPartial;
    }
        
    __syncthreads();
        
    // Unrolled parallel max-reduction
    if (scalingActive && state < 2) {
        REAL compare = sPartials1[myIdx + 2];
        if (compare >  sPartials1[myIdx])
            sPartials1[myIdx] = compare;
    }
     
    __syncthreads();
            
    if (scalingActive && state < 1) {
        REAL maxPartial = sPartials1[myIdx + 1];
        if (maxPartial < sPartials1[myIdx])
            maxPartial = sPartials1[myIdx];
        int expMax;
        frexp(maxPartial, &expMax);
        sPartials1[myIdx] = expMax;
        *activeScalingFactors = 1;
    }

    __syncthreads();
            
    if (scalingActive) 
        partials3[u] = ldexp(sigTmp, expTmp - sPartials1[patIdx16pat4]);
        
    if ((myIdx < PATTERN_BLOCK_SIZE * 4) && (myIdx + __umul24(blockIdx.x, PATTERN_BLOCK_SIZE * 4) < totalPatterns))
        scalingFactors[(blockIdx.x * PATTERN_BLOCK_SIZE * 4) + (matrix * totalPatterns) + myIdx] = sPartials1[multBy4(myIdx)];
}

__global__ void kernelPartialsPartialsFixedScale(REAL* partials1,
                                                                      REAL* partials2,
                                                                      REAL* partials3,
                                                                      REAL* matrices1,
                                                                      REAL* matrices2,
                                                                      REAL* scalingFactors,
                                                                      int totalPatterns) {
    REAL sum1;
    REAL sum2;
    int i;

    DETERMINE_INDICES_4();
    int y = deltaPartialsByState + deltaPartialsByMatrix;
    REAL* matrix1 = matrices1 + x2; // Points to *this* matrix
    REAL* matrix2 = matrices2 + x2;

#ifdef KERNEL_PRINT_ENABLED
    printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
           state, u);
#endif

    // Load values into shared memory
    __shared__ REAL sMatrix1[16];
    __shared__ REAL sMatrix2[16];

    __shared__ REAL sPartials1[PATTERN_BLOCK_SIZE * 4 * 4];
    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

    __shared__ REAL fixedScalingFactors[PATTERN_BLOCK_SIZE * 4];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        sPartials1[patIdx * 16 + tx] = partials1[y + tx]; // All coalesced memory reads
        sPartials2[patIdx * 16 + tx] = partials2[y + tx];
    } else {
        sPartials1[patIdx * 16 + tx] = 0;
        sPartials2[patIdx * 16 + tx] = 0;
    }

    if (patIdx < 4) // need to load 4*PATTERN_BLOCK_SIZE factors for this block
        fixedScalingFactors[patIdx * PATTERN_BLOCK_SIZE + tx] =
            scalingFactors[blockIdx.x * PATTERN_BLOCK_SIZE * 4 + patIdx * PATTERN_BLOCK_SIZE + tx];

    if (patIdx == 0 ) {
        sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
        sMatrix2[tx] = matrix2[tx];
    }

    __syncthreads();

    if (pattern < totalPatterns) { // Remove padded threads!

        i = pat;
        sum1  = sMatrix1[i * 4 + state] * sPartials1[patIdx * 16 + pat * 4 + i];
        sum2  = sMatrix2[i * 4 + state] * sPartials2[patIdx * 16 + pat * 4 + i];

        i = (++i) & 0x3;
        sum1 += sMatrix1[i * 4 + state] * sPartials1[patIdx * 16 + pat * 4 + i];
        sum2 += sMatrix2[i * 4 + state] * sPartials2[patIdx * 16 + pat * 4 + i];

        i = (++i) & 0x3;
        sum1 += sMatrix1[i * 4 + state] * sPartials1[patIdx * 16 + pat * 4 + i];
        sum2 += sMatrix2[i * 4 + state] * sPartials2[patIdx * 16 + pat * 4 + i];

        i = (++i) & 0x3;
        sum1 += sMatrix1[i * 4 + state] * sPartials1[patIdx * 16 + pat * 4 + i];
        sum2 += sMatrix2[i * 4 + state] * sPartials2[patIdx * 16 + pat * 4 + i];
        
        partials3[u] = sum1 * sum2 / fixedScalingFactors[patIdx * 4 + pat];
    }

}

__global__ void kernelStatesPartialsNoScale(int* states1,
                                                                REAL* partials2,
                                                                REAL* partials3,
                                                                REAL* matrices1,
                                                                REAL* matrices2,
                                                                int totalPatterns) {
    REAL sum1 = 1;
    REAL sum2;
    int i;

    DETERMINE_INDICES_4();
    int y = deltaPartialsByState + deltaPartialsByMatrix;
    REAL* matrix1 = matrices1 + x2; // Points to *this* matrix
    REAL* matrix2 = matrices2 + x2;


#ifdef KERNEL_PRINT_ENABLED
    printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
           state, u);
#endif

    // Load values into shared memory
    __shared__ REAL sMatrix1[16];
    __shared__ REAL sMatrix2[16];

    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

    // copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        sPartials2[patIdx * 16 + tx] = partials2[y + tx];
    } else {
        sPartials2[patIdx * 16 + tx] = 0;
    }

    if (patIdx == 0) {
        sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
        sMatrix2[tx] = matrix2[tx];
    }

    __syncthreads();

    if (pattern < totalPatterns) { // Remove padded threads!

        int state1 = states1[pattern];

        if (state1 < PADDED_STATE_COUNT)
            sum1 = sMatrix1[state1 * 4 + state];

        i = pat;
        sum2  = sMatrix2[i * 4 + state] * sPartials2[patIdx * 16 + pat * 4 + i];
        i = (++i) & 0x3;
        sum2 += sMatrix2[i * 4 + state] * sPartials2[patIdx * 16 + pat * 4 + i];
        i = (++i) & 0x3;
        sum2 += sMatrix2[i * 4 + state] * sPartials2[patIdx * 16 + pat * 4 + i];
        i = (++i) & 0x3;
        sum2 += sMatrix2[i * 4 + state] * sPartials2[patIdx * 16 + pat * 4 + i];
        partials3[u] = sum1 * sum2;
    }

}


__global__ void kernelStatesStatesNoScale(int* states1,
                                                              int* states2,
                                                              REAL* partials3,
                                                              REAL* matrices1,
                                                              REAL* matrices2,
                                                              int totalPatterns) {

	DETERMINE_INDICES_4();
    REAL* matrix1 = matrices1 + x2; // Points to *this* matrix
    REAL* matrix2 = matrices2 + x2;
    
#ifdef KERNEL_PRINT_ENABLED
    printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
           state, u);
#endif

    // Load values into shared memory
    __shared__ REAL sMatrix1[16];
    __shared__ REAL sMatrix2[16];

    if (patIdx == 0 ) {
        sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
        sMatrix2[tx] = matrix2[tx];
    }

    __syncthreads();

    if (pattern < totalPatterns) {
        int state1 = states1[pattern];
        int state2 = states2[pattern];

        if ( state1 < PADDED_STATE_COUNT && state2 < PADDED_STATE_COUNT) {
            partials3[u] = sMatrix1[state1 * 4 + state] * sMatrix2[state2 * 4 + state];
        } else if (state1 < PADDED_STATE_COUNT) {
            partials3[u] = sMatrix1[state1 * 4 + state];
        } else if (state2 < PADDED_STATE_COUNT) {
            partials3[u] = sMatrix2[state2 * 4 + state];
        } else {
            partials3[u] = 1.0;
        }
    }
}

__global__ void kernelPartialsPartialsEdgeLikelihoods(REAL* dPartialsTmp,
                                                              REAL* dParentPartials,
                                                              REAL* dChildParials,
                                                              REAL* dTransMatrix,
                                                              int totalPatterns) {
	   REAL sum1 = 0;

	    int i;

	    DETERMINE_INDICES_4();
	    int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);
	    int y = deltaPartialsByState + deltaPartialsByMatrix;
	    REAL* matrix1 = dTransMatrix + x2; // Points to *this* matrix

	#ifdef KERNEL_PRINT_ENABLED
	    printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
	           state, u);
	#endif

	    // Load values into shared memory
	    __shared__ REAL sMatrix1[16];

	    __shared__ REAL sPartials1[PATTERN_BLOCK_SIZE * 4 * 4];
	    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

	    // copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials
	    if (pattern < totalPatterns) {
	        sPartials1[multBy16(patIdx) | tx] = dParentPartials[y | tx]; // All coalesced memory reads
	        sPartials2[multBy16(patIdx) | tx] = dChildParials  [y | tx];
	    } else {
	        sPartials1[multBy16(patIdx) | tx] = 0;
	        sPartials2[multBy16(patIdx) | tx] = 0;
	    }

	    if (patIdx == 0 ) {
	        sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
	    }

	    __syncthreads();

	    if (pattern < totalPatterns) { // Remove padded threads!

	        i = pat;
	        sum1  = sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        i = (++i) & 0x3;
	        sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        i = (++i) & 0x3;
	        sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        i = (++i) & 0x3;
	        sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        
	        dPartialsTmp[u] = sum1 * sPartials2[patIdx16pat4 | state];
	    }    

	}
    
__global__ void kernelPartialsPartialsEdgeLikelihoodsSecondDeriv(REAL* dPartialsTmp,
                                                              REAL* dFirstDerivTmp,
                                                              REAL* dSecondDerivTmp,
                                                              REAL* dParentPartials,
                                                              REAL* dChildParials,
                                                              REAL* dTransMatrix,
                                                              REAL* dFirstDerivMatrix,
                                                              REAL* dSecondDerivMatrix,
                                                              int totalPatterns) {
	   REAL sum1 = 0;
	   REAL sumFirstDeriv = 0;
	   REAL sumSecondDeriv = 0;

	    int i;

	    DETERMINE_INDICES_4();
	    int patIdx16pat4 = multBy16(patIdx) | (tx & 0xC);
	    int y = deltaPartialsByState + deltaPartialsByMatrix;
	    REAL* matrix1 = dTransMatrix + x2; // Points to *this* matrix
	    REAL* matrixFirstDeriv = dFirstDerivMatrix + x2;
	    REAL* matrixSecondDeriv = dSecondDerivMatrix + x2;

	#ifdef KERNEL_PRINT_ENABLED
	    printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
	           state, u);
	#endif

	    // Load values into shared memory
	    __shared__ REAL sMatrix1[16];
	    __shared__ REAL sMatrixFirstDeriv[16];
	    __shared__ REAL sMatrixSecondDeriv[16];

	    __shared__ REAL sPartials1[PATTERN_BLOCK_SIZE * 4 * 4];
	    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

	    // copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials
	    if (pattern < totalPatterns) {
	        sPartials1[multBy16(patIdx) | tx] = dParentPartials[y | tx]; // All coalesced memory reads
	        sPartials2[multBy16(patIdx) | tx] = dChildParials  [y | tx];
	    } else {
	        sPartials1[multBy16(patIdx) | tx] = 0;
	        sPartials2[multBy16(patIdx) | tx] = 0;
	    }

	    if (patIdx == 0 ) {
	        sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
	        sMatrixFirstDeriv[tx] = matrixFirstDeriv[tx];
	        sMatrixSecondDeriv[tx] = matrixSecondDeriv[tx];
	    }

	    __syncthreads();

	    if (pattern < totalPatterns) { // Remove padded threads!

	        i = pat;
	        sum1  = sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        sumFirstDeriv  = sMatrixFirstDeriv[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        sumSecondDeriv  = sMatrixSecondDeriv[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        i = (++i) & 0x3;
	        sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        sumFirstDeriv  += sMatrixFirstDeriv[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        sumSecondDeriv  += sMatrixSecondDeriv[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        i = (++i) & 0x3;
	        sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        sumFirstDeriv  += sMatrixFirstDeriv[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        sumSecondDeriv  += sMatrixSecondDeriv[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        i = (++i) & 0x3;
	        sum1 += sMatrix1[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        sumFirstDeriv  += sMatrixFirstDeriv[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        sumSecondDeriv  += sMatrixSecondDeriv[multBy4(i) | state] * sPartials1[patIdx16pat4 | i];
	        
	        dPartialsTmp[u] = sum1 * sPartials2[patIdx16pat4 | state];
            dFirstDerivTmp[u] = sumFirstDeriv * sPartials2[patIdx16pat4 | state];
            dSecondDerivTmp[u] = sumSecondDeriv * sPartials2[patIdx16pat4 | state];
	    }    

	}


__global__ void kernelStatesPartialsEdgeLikelihoods(REAL* dPartialsTmp,
                                                         REAL* dParentPartials,
                                                         int* dChildStates,
                                                         REAL* dTransMatrix,
                                                         int totalPatterns) {
    REAL sum1 = 0;

    DETERMINE_INDICES_4();
    int y = deltaPartialsByState + deltaPartialsByMatrix;
    REAL* matrix1 = dTransMatrix + x2; // Points to *this* matrix
    
#ifdef KERNEL_PRINT_ENABLED
    printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
           state, u);
#endif

    // Load values into shared memory
    __shared__ REAL sMatrix1[16];

    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

    // copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        sPartials2[patIdx * 16 + tx] = dParentPartials[y + tx];
    } else {
        sPartials2[patIdx * 16 + tx] = 0;
    }

    if (patIdx == 0) {
        sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
    }

    __syncthreads();

    if (pattern < totalPatterns) { // Remove padded threads!
        int state1 = dChildStates[pattern];

        if (state1 < PADDED_STATE_COUNT)
            sum1 = sMatrix1[state1 * 4 + state];
        else
            sum1 = 1.0;

        dPartialsTmp[u] = sum1 * sPartials2[patIdx * 16 + pat * 4 + state];
    }
}

__global__ void kernelStatesPartialsEdgeLikelihoodsSecondDeriv(REAL* dPartialsTmp,
                                                              REAL* dFirstDerivTmp,
                                                              REAL* dSecondDerivTmp,
                                                              REAL* dParentPartials,
                                                              int* dChildStates,
                                                              REAL* dTransMatrix,
                                                              REAL* dFirstDerivMatrix,
                                                              REAL* dSecondDerivMatrix,
                                                              int totalPatterns) {
    REAL sum1 = 0;
    REAL sumFirstDeriv = 0;
    REAL sumSecondDeriv = 0;


    DETERMINE_INDICES_4();
    int y = deltaPartialsByState + deltaPartialsByMatrix;
    REAL* matrix1 = dTransMatrix + x2; // Points to *this* matrix
    REAL* matrixFirstDeriv = dFirstDerivMatrix + x2;
    REAL* matrixSecondDeriv = dSecondDerivMatrix + x2;

    
#ifdef KERNEL_PRINT_ENABLED
    printf("matrix = %d, pat = %d for tx = %d and state = %d :  u = %d\n", matrix, pattern, tx,
           state, u);
#endif

    // Load values into shared memory
    __shared__ REAL sMatrix1[16];
    __shared__ REAL sMatrixFirstDeriv[16];
    __shared__ REAL sMatrixSecondDeriv[16];

    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

    // copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        sPartials2[patIdx * 16 + tx] = dParentPartials[y + tx];
    } else {
        sPartials2[patIdx * 16 + tx] = 0;
    }

    if (patIdx == 0) {
        sMatrix1[tx] = matrix1[tx]; // All coalesced memory reads
        sMatrixFirstDeriv[tx] = matrixFirstDeriv[tx]; // All coalesced memory reads
        sMatrixSecondDeriv[tx] = matrixSecondDeriv[tx]; // All coalesced memory reads
    }

    __syncthreads();

    if (pattern < totalPatterns) { // Remove padded threads!
        int state1 = dChildStates[pattern];

        if (state1 < PADDED_STATE_COUNT) {
            sum1 = sMatrix1[state1 * 4 + state];
            sumFirstDeriv = sMatrixFirstDeriv[state1 * 4 + state];
            sumSecondDeriv = sMatrixSecondDeriv[state1 * 4 + state];
        } else {
            sum1 = 1.0;
            sumFirstDeriv = 0.0;
            sumSecondDeriv = 0.0;
        }

        dPartialsTmp[u] = sum1 * sPartials2[patIdx * 16 + pat * 4 + state];
        dFirstDerivTmp[u] = sumFirstDeriv * sPartials2[patIdx * 16 + pat * 4 + state];
        dSecondDerivTmp[u] = sumSecondDeriv * sPartials2[patIdx * 16 + pat * 4 + state];
    }
}


/*
 * Find a scaling factor for each pattern
 */
__global__ void kernelPartialsDynamicScaling(REAL* allPartials,
                                             REAL* scalingFactors,
                                             int matrixCount) {
                                             
    int tx = threadIdx.x;
    
    int state = tx & 0x3;
    int pat = tx >> 2;
                             
    int patIdx = blockIdx.x;
    
    int pattern = (patIdx << 2) + pat;
    int matrix = threadIdx.y;
    // TODO: Assumes matrixCount < MATRIX_BLOCK_SIZ
    
    // Patterns are always padded, so no reading/writing past end possible
    // Find start of patternBlock for thread-block
    int partialsOffset = (matrix * gridDim.x + patIdx) << 4; //* 16;

    __shared__ REAL partials[MATRIX_BLOCK_SIZE][16]; // 4 patterns at a time
    __shared__ REAL storedPartials[MATRIX_BLOCK_SIZE][16];

    __shared__ REAL matrixMax[4];
    
    if (matrix < matrixCount)
        partials[matrix][tx] = allPartials[partialsOffset + tx];          

    storedPartials[matrix][tx] = partials[matrix][tx];
           
    __syncthreads();
    
    // Unrolled parallel max-reduction
    if (state < 2) {
        REAL compare1 = partials[matrix][tx];
        REAL compare2 = partials[matrix][tx + 2];
        if (compare2 > compare1)
            partials[matrix][tx] = compare2;
    }
    __syncthreads();
    
    if (state < 1) {
        REAL compare1 = partials[matrix][tx];
        REAL compare2 = partials[matrix][tx + 1];
        if (compare2 > compare1)
            partials[matrix][tx] = compare2;
    }
    __syncthreads();
 
    // Could also parallel-reduce here.
    if (state == 0 && matrix == 0) {
        matrixMax[pat] = 0;
        int m;
        for(m = 0; m < matrixCount; m++) {
            if (partials[m][tx] > matrixMax[pat])
                matrixMax[pat] = partials[m][tx];
        }
        
        if (matrixMax[pat] == 0)
        	matrixMax[pat] = 1.0;
   
        scalingFactors[pattern] = matrixMax[pat]; // TODO: Are these incoherent writes?
    }

    // Attempt at a parallel reduction that (1) does not work and (2) is slower
//    if (state == 0) {    
//        for (int i = MATRIX_BLOCK_SIZE / 2; i > 0; i >>= 1) {
//            if (matrix < i) {
//                REAL compare1 = partials[matrix][tx];
//                REAL compare2 = partials[matrix+i][tx];
//                if (compare2 > compare1)
//                    partials[matrix][tx] = compare2;              
//            }
//            __syncthreads();
//        }         
//        
//        if (matrix == 0) {
//            matrixMax[pat] = partials[matrix][tx];
//            if (matrixMax[pat] == 0)
//                matrixMax[pat] = 1.0;
//                
//            scalingFactors[pattern] = matrixMax[pat];
//        }
//    }

    __syncthreads();

    if (matrix < matrixCount)
        allPartials[partialsOffset + tx] = storedPartials[matrix][tx] / matrixMax[pat];
}


/*
 * Find a scaling factor for each pattern
 */
__global__ void kernelPartialsDynamicScalingScalersLog(REAL* allPartials,
                                                      REAL* scalingFactors,
                                                      int matrixCount) {
                                             
    int tx = threadIdx.x;
    
    int state = tx & 0x3;
    int pat = tx >> 2;
                             
    int patIdx = blockIdx.x;
    
    int pattern = (patIdx << 2) + pat;
    int matrix = threadIdx.y;
    // TODO: Assumes matrixCount < MATRIX_BLOCK_SIZ
    
    // Patterns are always padded, so no reading/writing past end possible
    // Find start of patternBlock for thread-block
    int partialsOffset = (matrix * gridDim.x + patIdx) << 4; //* 16;

    __shared__ REAL partials[MATRIX_BLOCK_SIZE][16]; // 4 patterns at a time
    __shared__ REAL storedPartials[MATRIX_BLOCK_SIZE][16];

    __shared__ REAL matrixMax[4];
    
    if (matrix < matrixCount)
        partials[matrix][tx] = allPartials[partialsOffset + tx];          

    storedPartials[matrix][tx] = partials[matrix][tx];
           
    __syncthreads();
    
    // Unrolled parallel max-reduction
    if (state < 2) {
        REAL compare1 = partials[matrix][tx];
        REAL compare2 = partials[matrix][tx + 2];
        if (compare2 > compare1)
            partials[matrix][tx] = compare2;
    }
    __syncthreads();
    
    if (state < 1) {
        REAL compare1 = partials[matrix][tx];
        REAL compare2 = partials[matrix][tx + 1];
        if (compare2 > compare1)
            partials[matrix][tx] = compare2;
    }
    __syncthreads();
 
    // Could also parallel-reduce here.
    if (state == 0 && matrix == 0) {
        matrixMax[pat] = 0;
        int m;
        for(m = 0; m < matrixCount; m++) {
            if (partials[m][tx] > matrixMax[pat])
                matrixMax[pat] = partials[m][tx];
        }
        
        if (matrixMax[pat] == 0) {
        	matrixMax[pat] = 1.0;
            scalingFactors[pattern] = 0.0;
        } else {
            scalingFactors[pattern] = log(matrixMax[pat]);
        }
    }

    // Attempt at a parallel reduction that (1) does not work and (2) is slower
//    if (state == 0) {    
//        for (int i = MATRIX_BLOCK_SIZE / 2; i > 0; i >>= 1) {
//            if (matrix < i) {
//                REAL compare1 = partials[matrix][tx];
//                REAL compare2 = partials[matrix+i][tx];
//                if (compare2 > compare1)
//                    partials[matrix][tx] = compare2;              
//            }
//            __syncthreads();
//        }         
//        
//        if (matrix == 0) {
//            matrixMax[pat] = partials[matrix][tx];
//            if (matrixMax[pat] == 0)
//                matrixMax[pat] = 1.0;
//                
//            scalingFactors[pattern] = matrixMax[pat];
//        }
//    }

    __syncthreads();

    if (matrix < matrixCount)
        allPartials[partialsOffset + tx] = storedPartials[matrix][tx] / matrixMax[pat];
}

/*
 * Find a scaling factor for each pattern and accumulate into buffer
 */
__global__ void kernelPartialsDynamicScalingAccumulate(REAL* allPartials,
                                                       REAL* scalingFactors,
                                                       REAL* cumulativeScaling,
                                                       int matrixCount) {
    int tx = threadIdx.x;
    
    int state = tx & 0x3;
    int pat = tx >> 2;
                             
    int patIdx = blockIdx.x;
    
    int pattern = (patIdx << 2) + pat;
    int matrix = threadIdx.y;
    // TODO: Assumes matrixCount < MATRIX_BLOCK_SIZ
    
    // Patterns are always padded, so no reading/writing past end possible
    // Find start of patternBlock for thread-block
    int partialsOffset = (matrix * gridDim.x + patIdx) << 4; //* 16;

    __shared__ REAL partials[MATRIX_BLOCK_SIZE][16]; // 4 patterns at a time
    __shared__ REAL storedPartials[MATRIX_BLOCK_SIZE][16];

    __shared__ REAL matrixMax[4];
    
    if (matrix < matrixCount)
        partials[matrix][tx] = allPartials[partialsOffset + tx];          

    storedPartials[matrix][tx] = partials[matrix][tx];
           
    __syncthreads();
    
    // Unrolled parallel max-reduction
    if (state < 2) {
        REAL compare1 = partials[matrix][tx];
        REAL compare2 = partials[matrix][tx + 2];
        if (compare2 > compare1)
            partials[matrix][tx] = compare2;
    }
    __syncthreads();
    
    if (state < 1) {
        REAL compare1 = partials[matrix][tx];
        REAL compare2 = partials[matrix][tx + 1];
        if (compare2 > compare1)
            partials[matrix][tx] = compare2;
    }
    __syncthreads();
 
    // Could also parallel-reduce here.
    if (state == 0 && matrix == 0) {
        matrixMax[pat] = 0;
        int m;
        for(m = 0; m < matrixCount; m++) {
            if (partials[m][tx] > matrixMax[pat])
                matrixMax[pat] = partials[m][tx];
        }
        
        if (matrixMax[pat] == 0)
        	matrixMax[pat] = 1.0;
   
        scalingFactors[pattern] = matrixMax[pat]; 
        cumulativeScaling[pattern] += log(matrixMax[pat]);
    }

    __syncthreads();

    if (matrix < matrixCount)
        allPartials[partialsOffset + tx] = storedPartials[matrix][tx] / matrixMax[pat];
        
}

/*
 * Find a scaling factor for each pattern and accumulate into buffer
 */
__global__ void kernelPartialsDynamicScalingAccumulateScalersLog(REAL* allPartials,
                                                                REAL* scalingFactors,
                                                                REAL* cumulativeScaling,
                                                                int matrixCount) {
    int tx = threadIdx.x;
    
    int state = tx & 0x3;
    int pat = tx >> 2;
                             
    int patIdx = blockIdx.x;
    
    int pattern = (patIdx << 2) + pat;
    int matrix = threadIdx.y;
    // TODO: Assumes matrixCount < MATRIX_BLOCK_SIZ
    
    // Patterns are always padded, so no reading/writing past end possible
    // Find start of patternBlock for thread-block
    int partialsOffset = (matrix * gridDim.x + patIdx) << 4; //* 16;

    __shared__ REAL partials[MATRIX_BLOCK_SIZE][16]; // 4 patterns at a time
    __shared__ REAL storedPartials[MATRIX_BLOCK_SIZE][16];

    __shared__ REAL matrixMax[4];
    
    if (matrix < matrixCount)
        partials[matrix][tx] = allPartials[partialsOffset + tx];          

    storedPartials[matrix][tx] = partials[matrix][tx];
           
    __syncthreads();
    
    // Unrolled parallel max-reduction
    if (state < 2) {
        REAL compare1 = partials[matrix][tx];
        REAL compare2 = partials[matrix][tx + 2];
        if (compare2 > compare1)
            partials[matrix][tx] = compare2;
    }
    __syncthreads();
    
    if (state < 1) {
        REAL compare1 = partials[matrix][tx];
        REAL compare2 = partials[matrix][tx + 1];
        if (compare2 > compare1)
            partials[matrix][tx] = compare2;
    }
    __syncthreads();
 
    // Could also parallel-reduce here.
    if (state == 0 && matrix == 0) {
        matrixMax[pat] = 0;
        int m;
        for(m = 0; m < matrixCount; m++) {
            if (partials[m][tx] > matrixMax[pat])
                matrixMax[pat] = partials[m][tx];
        }
        
        if (matrixMax[pat] == 0) {
        	matrixMax[pat] = 1.0;
            scalingFactors[pattern] = 0.0;
        } else {
            REAL logMax = log(matrixMax[pat]);
            scalingFactors[pattern] = logMax;
            cumulativeScaling[pattern] += logMax; // TODO: Fix, this is both a read and write
        }
    }

    __syncthreads();

    if (matrix < matrixCount)
        allPartials[partialsOffset + tx] = storedPartials[matrix][tx] / matrixMax[pat];
        
}


#define LIKE_PATTERN_BLOCK_SIZE PATTERN_BLOCK_SIZE

__global__ void kernelIntegrateLikelihoodsFixedScale(REAL* dResult,
                                                     REAL* dRootPartials,
                                                     REAL *dWeights,
                                                     REAL *dFrequencies,
                                                     REAL *dRootScalingFactors,
                                                     int matrixCount,
                                                     int patternCount) {
    int state   = threadIdx.x;
    int pat = threadIdx.y;
    int pattern = blockIdx.x * LIKE_PATTERN_BLOCK_SIZE + threadIdx.y;
    
    __shared__ REAL stateFreq[4];
    
    // TODO: Currently assumes MATRIX_BLOCK_SIZE >= matrixCount
    __shared__ REAL matrixProp[MATRIX_BLOCK_SIZE];
    __shared__ REAL sum[LIKE_PATTERN_BLOCK_SIZE][4];

    // Load shared memory

    if (pat == 0) {
        stateFreq[state] = dFrequencies[state];
    }
    
    sum[pat][state] = 0;
    
    // TODO: Assumes matrixCount < LIKE_PATTERN_BLOCK_SIZE * 4
    if (pat * 4 + state < matrixCount) {
        matrixProp[pat * 4 + state] = dWeights[pat * 4 + state];
    }

    __syncthreads();

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;;

    for(int r = 0; r < matrixCount; r++) {
        sum[pat][state] += dRootPartials[u + delta * r] * matrixProp[r];
    }

    sum[pat][state] *= stateFreq[state];
        
    if (state < 2)
        sum[pat][state] += sum[pat][state + 2];
    __syncthreads();
    if (state < 1) {
        sum[pat][state] += sum[pat][state + 1];
    }
    __syncthreads();
    
    if (state == 0)
        dResult[pattern] = (log(sum[pat][state]) + dRootScalingFactors[pattern]);
}

__global__ void kernelIntegrateLikelihoodsAutoScaling(REAL* dResult,
                                                     REAL* dRootPartials,
                                                     REAL* dWeights,
                                                     REAL* dFrequencies,
                                                     int* dRootScalingFactors,
                                                     int matrixCount,
                                                     int patternCount) {
     int state   = threadIdx.x;
    int pat = threadIdx.y;
    int pattern = blockIdx.x * LIKE_PATTERN_BLOCK_SIZE + threadIdx.y;
    
    __shared__ REAL stateFreq[4];
    
    // TODO: Currently assumes MATRIX_BLOCK_SIZE >= matrixCount
    __shared__ REAL matrixProp[MATRIX_BLOCK_SIZE];
    __shared__ REAL sum[LIKE_PATTERN_BLOCK_SIZE][4];

    // Load shared memory

    if (pat == 0) {
        stateFreq[state] = dFrequencies[state];
    }
    
    sum[pat][state] = 0;
    
    // TODO: Assumes matrixCount < LIKE_PATTERN_BLOCK_SIZE * 4
    if (pat * 4 + state < matrixCount) {
        matrixProp[pat * 4 + state] = dWeights[pat * 4 + state];
    }

    __syncthreads();

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;

    short maxScaleFactor = dRootScalingFactors[pattern];
    for(int r = 1; r < matrixCount; r++) {
        int tmpFactor = dRootScalingFactors[pattern + (r * patternCount)];
        if (tmpFactor > maxScaleFactor)
            maxScaleFactor = tmpFactor;
    }

    for(int r = 0; r < matrixCount; r++) {
        int tmpFactor = dRootScalingFactors[pattern + (r * patternCount)];
        if (tmpFactor != maxScaleFactor) {
            // TODO: verify which of the two methods below is faster
            int expTmp;
            sum[pat][state] += ldexp(frexp(dRootPartials[u + delta * r], &expTmp), expTmp + (tmpFactor - maxScaleFactor)) * matrixProp[r];
//            sum[pat][state] += dRootPartials[u + delta * r] * pow(2.0, tmpFactor - maxScaleFactor) * matrixProp[r];
        } else {
            sum[pat][state] += dRootPartials[u + delta * r] * matrixProp[r];
        }
    }

    sum[pat][state] *= stateFreq[state];
        
    if (state < 2)
        sum[pat][state] += sum[pat][state + 2];
    __syncthreads();
    if (state < 1) {
        sum[pat][state] += sum[pat][state + 1];
    }
    __syncthreads();
    
    if (state == 0)
        dResult[pattern] = (log(sum[pat][state]) + (M_LN2 * maxScaleFactor));
}


__global__ void kernelIntegrateLikelihoodsFixedScaleSecondDeriv(REAL* dResult,
                                              REAL* dFirstDerivResult,
                                              REAL* dSecondDerivResult,
                                              REAL* dRootPartials,
                                              REAL* dRootFirstDeriv,
                                              REAL* dRootSecondDeriv,
                                              REAL* dWeights,
                                              REAL* dFrequencies,
                                              REAL *dRootScalingFactors,
                                              int matrixCount,
                                              int patternCount) {
    int state   = threadIdx.x;
    int pat = threadIdx.y;
    int pattern = blockIdx.x * LIKE_PATTERN_BLOCK_SIZE + threadIdx.y;
    
    REAL tmpLogLike = 0.0;
    REAL tmpFirstDeriv = 0.0;
    
    __shared__ REAL stateFreq[4];
    
    // TODO: Currently assumes MATRIX_BLOCK_SIZE >= matrixCount
    __shared__ REAL matrixProp[MATRIX_BLOCK_SIZE];
    __shared__ REAL sum[LIKE_PATTERN_BLOCK_SIZE][4];
    __shared__ REAL sumD1[LIKE_PATTERN_BLOCK_SIZE][4];
    __shared__ REAL sumD2[LIKE_PATTERN_BLOCK_SIZE][4];

    // Load shared memory

    if (pat == 0) {
        stateFreq[state] = dFrequencies[state];
    }
    
    sum[pat][state] = 0;
    sumD1[pat][state] = 0;
    sumD2[pat][state] = 0;
    
    // TODO: Assumes matrixCount < LIKE_PATTERN_BLOCK_SIZE * 4
    if (pat * 4 + state < matrixCount) {
        matrixProp[pat * 4 + state] = dWeights[pat * 4 + state];
    }

    __syncthreads();

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;;

    for(int r = 0; r < matrixCount; r++) {
        sum[pat][state] += dRootPartials[u + delta * r] * matrixProp[r];
        sumD1[pat][state] += dRootFirstDeriv[u + delta * r] * matrixProp[r];
        sumD2[pat][state] += dRootSecondDeriv[u + delta * r] * matrixProp[r];
    }

    sum[pat][state] *= stateFreq[state];
    sumD1[pat][state] *= stateFreq[state];
    sumD2[pat][state] *= stateFreq[state];
        
    if (state < 2) {
        sum[pat][state] += sum[pat][state + 2];
        sumD1[pat][state] += sumD1[pat][state + 2];
        sumD2[pat][state] += sumD2[pat][state + 2];
    }
    __syncthreads();
    if (state < 1) {
        sum[pat][state] += sum[pat][state + 1];
        sumD1[pat][state] += sumD1[pat][state + 1];
        sumD2[pat][state] += sumD2[pat][state + 1];
    }
    __syncthreads();
    
    if (state == 0) {
        tmpLogLike = sum[pat][state];
        dResult[pattern] = (log(tmpLogLike) + dRootScalingFactors[pattern]);
        
        tmpFirstDeriv = sumD1[pat][state] / tmpLogLike;
        dFirstDerivResult[pattern] = tmpFirstDeriv;
        
        dSecondDerivResult[pattern] = (sumD2[pat][state] / tmpLogLike - tmpFirstDeriv * tmpFirstDeriv);
    }
}


__global__ void kernelIntegrateLikelihoods(REAL* dResult,
                                              REAL* dRootPartials,
                                              REAL* dWeights,
                                              REAL* dFrequencies,
                                              int matrixCount,
                                              int patternCount) {
    int state   = threadIdx.x;
    int pat = threadIdx.y;
    int pattern = blockIdx.x * LIKE_PATTERN_BLOCK_SIZE + threadIdx.y;
    
    __shared__ REAL stateFreq[4];
    
    // TODO: Currently assumes MATRIX_BLOCK_SIZE >= matrixCount
    __shared__ REAL matrixProp[MATRIX_BLOCK_SIZE];
    __shared__ REAL sum[LIKE_PATTERN_BLOCK_SIZE][4];

    // Load shared memory

    if (pat == 0) {
        stateFreq[state] = dFrequencies[state];
    }
    
    sum[pat][state] = 0;
    
    // TODO: Assumes matrixCount < LIKE_PATTERN_BLOCK_SIZE * 4
    if (pat * 4 + state < matrixCount) {
        matrixProp[pat * 4 + state] = dWeights[pat * 4 + state];
    }

    __syncthreads();

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;;

    for(int r = 0; r < matrixCount; r++) {
        sum[pat][state] += dRootPartials[u + delta * r] * matrixProp[r];
    }

    sum[pat][state] *= stateFreq[state];
        
    if (state < 2)
        sum[pat][state] += sum[pat][state + 2];
    __syncthreads();
    if (state < 1) {
        sum[pat][state] += sum[pat][state + 1];
    }
    
    // TODO: remove this extra syncthreads for all integrate kernels
    __syncthreads();
    
    if (state == 0)
        dResult[pattern] = log(sum[pat][state]);
        
}

__global__ void kernelIntegrateLikelihoodsSecondDeriv(REAL* dResult,
                                              REAL* dFirstDerivResult,
                                              REAL* dSecondDerivResult,
                                              REAL* dRootPartials,
                                              REAL* dRootFirstDeriv,
                                              REAL* dRootSecondDeriv,
                                              REAL* dWeights,
                                              REAL* dFrequencies,
                                              int matrixCount,
                                              int patternCount) {
    int state   = threadIdx.x;
    int pat = threadIdx.y;
    int pattern = blockIdx.x * LIKE_PATTERN_BLOCK_SIZE + threadIdx.y;
    
    REAL tmpLogLike = 0.0;
    REAL tmpFirstDeriv = 0.0;
    
    __shared__ REAL stateFreq[4];
    
    // TODO: Currently assumes MATRIX_BLOCK_SIZE >= matrixCount
    __shared__ REAL matrixProp[MATRIX_BLOCK_SIZE];
    __shared__ REAL sum[LIKE_PATTERN_BLOCK_SIZE][4];
    __shared__ REAL sumD1[LIKE_PATTERN_BLOCK_SIZE][4];
    __shared__ REAL sumD2[LIKE_PATTERN_BLOCK_SIZE][4];

    // Load shared memory

    if (pat == 0) {
        stateFreq[state] = dFrequencies[state];
    }
    
    sum[pat][state] = 0;
    sumD1[pat][state] = 0;
    sumD2[pat][state] = 0;
    
    // TODO: Assumes matrixCount < LIKE_PATTERN_BLOCK_SIZE * 4
    if (pat * 4 + state < matrixCount) {
        matrixProp[pat * 4 + state] = dWeights[pat * 4 + state];
    }

    __syncthreads();

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;;

    for(int r = 0; r < matrixCount; r++) {
        sum[pat][state] += dRootPartials[u + delta * r] * matrixProp[r];
        sumD1[pat][state] += dRootFirstDeriv[u + delta * r] * matrixProp[r];
        sumD2[pat][state] += dRootSecondDeriv[u + delta * r] * matrixProp[r];
    }

    sum[pat][state] *= stateFreq[state];
    sumD1[pat][state] *= stateFreq[state];
    sumD2[pat][state] *= stateFreq[state];
        
    if (state < 2) {
        sum[pat][state] += sum[pat][state + 2];
        sumD1[pat][state] += sumD1[pat][state + 2];
        sumD2[pat][state] += sumD2[pat][state + 2];
    }
    __syncthreads();
    if (state < 1) {
        sum[pat][state] += sum[pat][state + 1];
        sumD1[pat][state] += sumD1[pat][state + 1];
        sumD2[pat][state] += sumD2[pat][state + 1];
    }
    __syncthreads();
    
    if (state == 0) {
        tmpLogLike = sum[pat][state];
        dResult[pattern] = log(tmpLogLike);
        
        tmpFirstDeriv = sumD1[pat][state] / tmpLogLike;
        dFirstDerivResult[pattern] = tmpFirstDeriv;
        
        dSecondDerivResult[pattern] = (sumD2[pat][state] / tmpLogLike - tmpFirstDeriv * tmpFirstDeriv);
    }
}


__global__ void kernelIntegrateLikelihoodsMulti(REAL* dResult,
                                              REAL* dRootPartials,
                                              REAL* dWeights,
                                              REAL* dFrequencies,
                                              int matrixCount,
                                              int patternCount,
											  int takeLog) {
    int state   = threadIdx.x;
    int pat = threadIdx.y;
    int pattern = blockIdx.x * LIKE_PATTERN_BLOCK_SIZE + threadIdx.y;
    
    __shared__ REAL stateFreq[4];
    
    // TODO: Currently assumes MATRIX_BLOCK_SIZE >= matrixCount
    __shared__ REAL matrixProp[MATRIX_BLOCK_SIZE];
    __shared__ REAL sum[LIKE_PATTERN_BLOCK_SIZE][4];

    // Load shared memory

    if (pat == 0) {
        stateFreq[state] = dFrequencies[state];
    }
    
    sum[pat][state] = 0;
    
    // TODO: Assumes matrixCount < LIKE_PATTERN_BLOCK_SIZE * 4
    if (pat * 4 + state < matrixCount) {
        matrixProp[pat * 4 + state] = dWeights[pat * 4 + state];
    }

    __syncthreads();

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;;

    for(int r = 0; r < matrixCount; r++) {
        sum[pat][state] += dRootPartials[u + delta * r] * matrixProp[r];
    }

    sum[pat][state] *= stateFreq[state];
        
    if (state < 2)
        sum[pat][state] += sum[pat][state + 2];
    __syncthreads();
    if (state < 1) {
        sum[pat][state] += sum[pat][state + 1];
    }
    __syncthreads();
    
    if (state == 0) {
		if (takeLog == 0)
			dResult[pattern] = sum[pat][state];
		else if (takeLog == 1)
			dResult[pattern] = log(dResult[pattern] + sum[pat][state]);
		else 
			dResult[pattern] += sum[pat][state];
	}
        
}

__global__ void kernelIntegrateLikelihoodsFixedScaleMulti(REAL* dResult,
											  REAL* dRootPartials,
                                              REAL* dWeights,
                                              REAL* dFrequencies,
											  REAL** dPtrQueue,
											  REAL* dMaxScalingFactors,
											  REAL* dIndexMaxScalingFactors,
                                              int matrixCount,
                                              int patternCount,
											  int subsetCount,
											  int subsetIndex) {
    int state   = threadIdx.x;
    int pat = threadIdx.y;
    int pattern = blockIdx.x * LIKE_PATTERN_BLOCK_SIZE + threadIdx.y;
    
    __shared__ REAL stateFreq[4];
    
    // TODO: Currently assumes MATRIX_BLOCK_SIZE >= matrixCount
    __shared__ REAL matrixProp[MATRIX_BLOCK_SIZE];
    __shared__ REAL sum[LIKE_PATTERN_BLOCK_SIZE][4];

    // Load shared memory

    if (pat == 0) {
        stateFreq[state] = dFrequencies[state];
    }
    
    sum[pat][state] = 0;
    
    // TODO: Assumes matrixCount < LIKE_PATTERN_BLOCK_SIZE * 4
    if (pat * 4 + state < matrixCount) {
        matrixProp[pat * 4 + state] = dWeights[pat * 4 + state];
    }

    __syncthreads();

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;;

    for(int r = 0; r < matrixCount; r++) {
        sum[pat][state] += dRootPartials[u + delta * r] * matrixProp[r];
    }

    sum[pat][state] *= stateFreq[state];
        
    if (state < 2)
        sum[pat][state] += sum[pat][state + 2];
    __syncthreads();
    if (state < 1) {
        sum[pat][state] += sum[pat][state + 1];
    }
    __syncthreads();
    
	REAL cumulativeScalingFactor = ((REAL*) *((int*)dPtrQueue + subsetIndex))[pattern];
	
	if (subsetIndex == 0) {
		int indexMaxScalingFactor = 0;
		REAL maxScalingFactor = cumulativeScalingFactor;
		for (int j = 1; j < subsetCount; j++) {
			REAL tmpScalingFactor = ((REAL*) *((int*)dPtrQueue + j))[pattern];
			if (tmpScalingFactor > maxScalingFactor) {
				indexMaxScalingFactor = j;
				maxScalingFactor = tmpScalingFactor;
			}
		}
		
		dIndexMaxScalingFactors[pattern] = indexMaxScalingFactor;
		dMaxScalingFactors[pattern] = maxScalingFactor;	
		
		if (indexMaxScalingFactor != 0)
			sum[pat][state] *= exp((REAL)(cumulativeScalingFactor - maxScalingFactor));
			
		if (state == 0)
			dResult[pattern] = sum[pat][state];
	} else {
		if (subsetIndex != dIndexMaxScalingFactors[pattern])
			sum[pat][state] *= exp((REAL)(cumulativeScalingFactor - dMaxScalingFactors[pattern]));
	
		if (state == 0) {
			if (subsetIndex == subsetCount - 1)
				dResult[pattern] = (log(dResult[pattern] + sum[pat][state]) + dMaxScalingFactors[pattern]);
			else
				dResult[pattern] += sum[pat][state];
		}
	}        
}

} // extern "C"

