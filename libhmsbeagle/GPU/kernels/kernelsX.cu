#include "hip/hip_runtime.h"
/*
 *
 * Copyright 2009 Phylogenetic Likelihood Working Group
 *
 * This file is part of BEAGLE.
 *
 * BEAGLE is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * BEAGLE is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with BEAGLE.  If not, see
 * <http://www.gnu.org/licenses/>.
 *
 * @author Marc Suchard
 * @author Daniel Ayres
 */

#include "libhmsbeagle/GPU/GPUImplDefs.h"
#include "libhmsbeagle/GPU/kernels/kernelsAll.cu" // This file includes the non-state-count specific kernels

#define DETERMINE_INDICES() \
    int state = threadIdx.x; \
    int patIdx = threadIdx.y; \
    int pattern = __umul24(blockIdx.x,PATTERN_BLOCK_SIZE) + patIdx; \
    int matrix = blockIdx.y; \
    int patternCount = totalPatterns; \
    int deltaPartialsByState = pattern * PADDED_STATE_COUNT; \
    int deltaPartialsByMatrix = matrix * PADDED_STATE_COUNT * patternCount; \
    int deltaMatrix = matrix * PADDED_STATE_COUNT * PADDED_STATE_COUNT; \
    int u = state + deltaPartialsByState + deltaPartialsByMatrix;

extern "C" {

__global__ void kernelPartialsPartialsNoScale(REAL* partials1,
                                                             REAL* partials2,
                                                             REAL* partials3,
                                                             REAL* matrices1,
                                                             REAL* matrices2,
                                                             int totalPatterns) {
    REAL sum1 = 0;
    REAL sum2 = 0;
    int i;

    DETERMINE_INDICES();

    REAL* matrix1 = matrices1 + deltaMatrix; // Points to *this* matrix
    REAL* matrix2 = matrices2 + deltaMatrix;

    int y = deltaPartialsByState + deltaPartialsByMatrix;

    // Load values into shared memory
    __shared__ REAL sMatrix1[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];
    __shared__ REAL sMatrix2[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL sPartials1[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];
    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        // These are all coherent global memory reads; checked in Profiler
        sPartials1[patIdx][state] = partials1[y + state];
        sPartials2[patIdx][state] = partials2[y + state];
    } else {
        sPartials1[patIdx][state] = 0;
        sPartials2[patIdx][state] = 0;
    }

    for (i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE) {
        // load one row of matrices
        if (patIdx < BLOCK_PEELING_SIZE) {
            // These are all coherent global memory reads.
            sMatrix1[patIdx][state] = matrix1[patIdx * PADDED_STATE_COUNT + state];
            sMatrix2[patIdx][state] = matrix2[patIdx * PADDED_STATE_COUNT + state];

            // sMatrix now filled with starting in state and ending in i
            matrix1 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
            matrix2 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
        }
        __syncthreads();

        int j;
        for(j = 0; j < BLOCK_PEELING_SIZE; j++) {
            sum1 += sMatrix1[j][state] * sPartials1[patIdx][i + j];
            sum2 += sMatrix2[j][state] * sPartials2[patIdx][i + j];
        }

        __syncthreads(); // GTX280 FIX HERE

    }

    if (pattern < totalPatterns)
        partials3[u] = sum1 * sum2;
}

__global__ void kernelPartialsPartialsAutoScale(REAL* partials1,
                                                             REAL* partials2,
                                                             REAL* partials3,
                                                             REAL* matrices1,
                                                             REAL* matrices2,
                                                             signed char* scalingFactors,
                                                             unsigned short* activeScalingFactors,
                                                             int totalPatterns) {
    REAL sum1 = 0;
    REAL sum2 = 0;
    int i;

    DETERMINE_INDICES();

    REAL* matrix1 = matrices1 + deltaMatrix; // Points to *this* matrix
    REAL* matrix2 = matrices2 + deltaMatrix;

    int y = deltaPartialsByState + deltaPartialsByMatrix;

    // Load values into shared memory
    __shared__ REAL sMatrix1[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];
    __shared__ REAL sMatrix2[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL sPartials1[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];
    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        // These are all coherent global memory reads; checked in Profiler
        sPartials1[patIdx][state] = partials1[y + state];
        sPartials2[patIdx][state] = partials2[y + state];
    } else {
        sPartials1[patIdx][state] = 0;
        sPartials2[patIdx][state] = 0;
    }

    for (i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE) {
        // load one row of matrices
        if (patIdx < BLOCK_PEELING_SIZE) {
            // These are all coherent global memory reads.
            sMatrix1[patIdx][state] = matrix1[patIdx * PADDED_STATE_COUNT + state];
            sMatrix2[patIdx][state] = matrix2[patIdx * PADDED_STATE_COUNT + state];

            // sMatrix now filled with starting in state and ending in i
            matrix1 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
            matrix2 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
        }
        __syncthreads();

        int j;
        for(j = 0; j < BLOCK_PEELING_SIZE; j++) {
            sum1 += sMatrix1[j][state] * sPartials1[patIdx][i + j];
            sum2 += sMatrix2[j][state] * sPartials2[patIdx][i + j];
        }

        __syncthreads(); // GTX280 FIX HERE

    }

    REAL tmpPartial = sum1 * sum2;
    int expTmp;
    REAL sigTmp = frexp(tmpPartial, &expTmp);

    if (pattern < totalPatterns) {
        if (abs(expTmp) > SCALING_EXPONENT_THRESHOLD) {
            // now using sPartials2 to hold scaling trigger boolean
            sPartials2[patIdx][0] = 1;
        } else {
            partials3[u] = tmpPartial;
            sPartials2[patIdx][0] = 0;
            sPartials1[patIdx][0] = 0;
        }
    }
        
    __syncthreads();
    
    int scalingActive = sPartials2[patIdx][0];
        
    if (scalingActive) {
        // now using sPartials1 to store max unscaled partials3
        sPartials1[patIdx][state] = tmpPartial;
    }
            
    __syncthreads();
            
    // Unrolled parallel max-reduction
    if (scalingActive && state < 2) {
        REAL compare = sPartials1[patIdx][state + 2];
        if (compare >  sPartials1[patIdx][state])
            sPartials1[patIdx][state] = compare;
    }
    
    __syncthreads();
            
    if (scalingActive && state < 1) {
        REAL maxPartial = sPartials1[patIdx][1];
        if (maxPartial < sPartials1[patIdx][0])
            maxPartial = sPartials1[patIdx][0];
        int expMax;
        frexp(maxPartial, &expMax);
        sPartials1[patIdx][0] = expMax;
        *activeScalingFactors = 1;
    }

    __syncthreads();
    
    if (scalingActive)
        partials3[u] = ldexp(sigTmp, expTmp - sPartials1[patIdx][0]);

    int myIdx = (patIdx * PADDED_STATE_COUNT) + state; // threadId in block
    if (myIdx < PATTERN_BLOCK_SIZE)
        scalingFactors[(blockIdx.x * PATTERN_BLOCK_SIZE) + (matrix * totalPatterns) + myIdx] = sPartials1[myIdx][0];

}

__global__ void kernelPartialsPartialsFixedScale(REAL* partials1,
                                                                 REAL* partials2,
                                                                 REAL* partials3,
                                                                 REAL* matrices1,
                                                                 REAL* matrices2,
                                                                 REAL* scalingFactors,
                                                                 int totalPatterns) {
    REAL sum1 = 0;
    REAL sum2 = 0;
    int i;

    DETERMINE_INDICES();

    REAL* matrix1 = matrices1 + deltaMatrix; // Points to *this* matrix
    REAL* matrix2 = matrices2 + deltaMatrix;

    int y = deltaPartialsByState + deltaPartialsByMatrix;

    // Load values into shared memory
    __shared__ REAL sMatrix1[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];
    __shared__ REAL sMatrix2[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL sPartials1[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];
    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL fixedScalingFactors[PATTERN_BLOCK_SIZE];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        // These are all coherent global memory reads; checked in Profiler
        sPartials1[patIdx][state] = partials1[y + state];
        sPartials2[patIdx][state] = partials2[y + state];
    } else {
        sPartials1[patIdx][state] = 0;
        sPartials2[patIdx][state] = 0;
    }

    if (patIdx == 0 && state < PATTERN_BLOCK_SIZE )
        // TODO: If PATTERN_BLOCK_SIZE > PADDED_STATE_COUNT, there is a bug here
        fixedScalingFactors[state] = scalingFactors[blockIdx.x * PATTERN_BLOCK_SIZE + state];

    for (i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE) {
        // load one row of matrices
        if (patIdx < BLOCK_PEELING_SIZE) {
            // These are all coherent global memory reads.
            sMatrix1[patIdx][state] = matrix1[patIdx * PADDED_STATE_COUNT + state];
            sMatrix2[patIdx][state] = matrix2[patIdx * PADDED_STATE_COUNT + state];

            // sMatrix now filled with starting in state and ending in i
            matrix1 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
            matrix2 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
        }
        __syncthreads();

        int j;
        for(j = 0; j < BLOCK_PEELING_SIZE; j++) {
            sum1 += sMatrix1[j][state] * sPartials1[patIdx][i + j];
            sum2 += sMatrix2[j][state] * sPartials2[patIdx][i + j];
        }

        __syncthreads(); // GTX280 FIX HERE

    }

    if (pattern < totalPatterns)
        partials3[u] = sum1 * sum2 / fixedScalingFactors[patIdx];

}

__global__ void kernelStatesPartialsNoScale(int* states1,
                                                           REAL* partials2,
                                                           REAL* partials3,
                                                           REAL* matrices1,
                                                           REAL* matrices2,
                                                           int totalPatterns) {
    REAL sum1 = 0;
    REAL sum2 = 0;
    int i;

    DETERMINE_INDICES();

    int y = deltaPartialsByState + deltaPartialsByMatrix;

    // Load values into shared memory
    __shared__ REAL sMatrix2[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        sPartials2[patIdx][state] = partials2[y + state];
    } else {
        sPartials2[patIdx][state] = 0;
    }

    REAL* matrix2 = matrices2 + deltaMatrix;

    if (pattern < totalPatterns) {
        int state1 = states1[pattern]; // Coalesced; no need to share

        REAL* matrix1 = matrices1 + deltaMatrix + state1 * PADDED_STATE_COUNT;

        if (state1 < PADDED_STATE_COUNT)
            sum1 = matrix1[state];
        else
            sum1 = 1.0;
    }

    for (i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE) {
        // load one row of matrices
        if (patIdx < BLOCK_PEELING_SIZE) {
            sMatrix2[patIdx][state] = matrix2[patIdx * PADDED_STATE_COUNT + state];

            // sMatrix now filled with starting in state and ending in i
            matrix2 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
        }
        __syncthreads();

        int j;
        for(j = 0; j < BLOCK_PEELING_SIZE; j++) {
            sum2 += sMatrix2[j][state] * sPartials2[patIdx][i + j];
        }

        __syncthreads(); // GTX280 FIX HERE

    }

    if (pattern < totalPatterns)
        partials3[u] = sum1 * sum2;
}

__global__ void kernelStatesPartialsFixedScale(int* states1,
                                                               REAL* partials2,
                                                               REAL* partials3,
                                                               REAL* matrices1,
                                                               REAL* matrices2,
                                                               REAL* scalingFactors,
                                                               int totalPatterns) {
    REAL sum1 = 0;
    REAL sum2 = 0;
    int i;

    DETERMINE_INDICES();

    int y = deltaPartialsByState + deltaPartialsByMatrix;

    // Load values into shared memory
    __shared__ REAL sMatrix2[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL fixedScalingFactors[PATTERN_BLOCK_SIZE];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        sPartials2[patIdx][state] = partials2[y + state];
    } else {
        sPartials2[patIdx][state] = 0;
    }

    REAL* matrix2 = matrices2 + deltaMatrix;

    if (pattern < totalPatterns) {
        int state1 = states1[pattern]; // Coalesced; no need to share

        REAL* matrix1 = matrices1 + deltaMatrix + state1 * PADDED_STATE_COUNT;

        if (state1 < PADDED_STATE_COUNT)
            sum1 = matrix1[state];
        else
            sum1 = 1.0;
    }

    if (patIdx == 0 && state < PATTERN_BLOCK_SIZE )
        // TODO: If PATTERN_BLOCK_SIZE > PADDED_STATE_COUNT, there is a bug here
        fixedScalingFactors[state] = scalingFactors[blockIdx.x * PATTERN_BLOCK_SIZE + state];

    for (i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE) {
        // load one row of matrices
        if (patIdx < BLOCK_PEELING_SIZE) {
            sMatrix2[patIdx][state] = matrix2[patIdx * PADDED_STATE_COUNT + state];

            // sMatrix now filled with starting in state and ending in i
            matrix2 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
        }
        __syncthreads();

        int j;
        for(j = 0; j < BLOCK_PEELING_SIZE; j++) {
            sum2 += sMatrix2[j][state] * sPartials2[patIdx][i + j];
        }

        __syncthreads(); // GTX280 FIX HERE

    }

    if (pattern < totalPatterns)
        partials3[u] = sum1 * sum2 / fixedScalingFactors[patIdx];
}

__global__ void kernelStatesStatesNoScale(int* states1,
                                                         int* states2,
                                                         REAL* partials3,
                                                         REAL* matrices1,
                                                         REAL* matrices2,
                                                         int totalPatterns) {
    DETERMINE_INDICES();

    // Load values into shared memory
//  __shared__ REAL sMatrix1[PADDED_STATE_COUNT];
//  __shared__ REAL sMatrix2[PADDED_STATE_COUNT];

    int state1 = states1[pattern];
    int state2 = states2[pattern];

    // Points to *this* matrix
    REAL* matrix1 = matrices1 + deltaMatrix + state1 * PADDED_STATE_COUNT;
    REAL* matrix2 = matrices2 + deltaMatrix + state2 * PADDED_STATE_COUNT;

//  if (patIdx == 0) {
//      sMatrix1[state] = matrix1[state];
//      sMatrix2[state] = matrix2[state];
//  }

    __syncthreads();

    if (pattern < totalPatterns) {

        if ( state1 < PADDED_STATE_COUNT && state2 < PADDED_STATE_COUNT) {
//          partials3[u] = sMatrix1[state] * sMatrix2[state];
            partials3[u] = matrix1[state] * matrix2[state];
        } else if (state1 < PADDED_STATE_COUNT) {
//          partials3[u] = sMatrix1[state];
            partials3[u] = matrix1[state];
        } else if (state2 < PADDED_STATE_COUNT) {
//          partials3[u] = sMatrix2[state];
            partials3[u] = matrix2[state];
        } else {
            partials3[u] = 1.0;
        }
    }
}

__global__ void kernelStatesStatesFixedScale(int* states1,
                                                             int* states2,
                                                             REAL* partials3,
                                                             REAL* matrices1,
                                                             REAL* matrices2,
                                                             REAL* scalingFactors,
                                                             int totalPatterns) {
    DETERMINE_INDICES();

    // Load values into shared memory
    // Prefetching into shared memory gives no performance gain
    // TODO: Double-check.
//  __shared__ REAL sMatrix1[PADDED_STATE_COUNT];
//  __shared__ REAL sMatrix2[PADDED_STATE_COUNT];

    __shared__ REAL fixedScalingFactors[PATTERN_BLOCK_SIZE];

    int state1 = states1[pattern];
    int state2 = states2[pattern];

    // Points to *this* matrix
    REAL* matrix1 = matrices1 + deltaMatrix + state1 * PADDED_STATE_COUNT;
    REAL* matrix2 = matrices2 + deltaMatrix + state2 * PADDED_STATE_COUNT;

//  if (patIdx == 0) {
//      sMatrix1[state] = matrix1[state];
//      sMatrix2[state] = matrix2[state];
//  }

    // TODO: If PATTERN_BLOCK_SIZE > PADDED_STATE_COUNT, there is a bug here
    if (patIdx == 0 && state < PATTERN_BLOCK_SIZE )
        fixedScalingFactors[state] = scalingFactors[blockIdx.x * PATTERN_BLOCK_SIZE + state];

    __syncthreads();

    if (pattern < totalPatterns) {
        if (state1 < PADDED_STATE_COUNT && state2 < PADDED_STATE_COUNT) {
//          partials3[u] = sMatrix1[state] * sMatrix2[state];
            partials3[u] = matrix1[state] * matrix2[state] / fixedScalingFactors[patIdx];
        } else if (state1 < PADDED_STATE_COUNT) {
//          partials3[u] = sMatrix1[state];
            partials3[u] = matrix1[state] / fixedScalingFactors[patIdx];
        } else if (state2 < PADDED_STATE_COUNT) {
//          partials3[u] = sMatrix2[state];
            partials3[u] = matrix2[state] / fixedScalingFactors[patIdx];
        } else {
            partials3[u] = 1.0 / fixedScalingFactors[patIdx];
        }
    }
}

__global__ void kernelPartialsPartialsEdgeLikelihoods(REAL* dPartialsTmp,
                                                         REAL* dParentPartials,
                                                         REAL* dChildParials,
                                                         REAL* dTransMatrix,
                                                         int patternCount) {
    REAL sum1 = 0;

    int i;

    int state = threadIdx.x;
    int patIdx = threadIdx.y;
    int pattern = __umul24(blockIdx.x,PATTERN_BLOCK_SIZE) + patIdx;
    int matrix = blockIdx.y;
    int totalPatterns = patternCount;
    int deltaPartialsByState = pattern * PADDED_STATE_COUNT;
    int deltaPartialsByMatrix = matrix * PADDED_STATE_COUNT * totalPatterns;
    int deltaMatrix = matrix * PADDED_STATE_COUNT * PADDED_STATE_COUNT;
    int u = state + deltaPartialsByState + deltaPartialsByMatrix;

    REAL* matrix1 = dTransMatrix + deltaMatrix; // Points to *this* matrix

    int y = deltaPartialsByState + deltaPartialsByMatrix;

    // Load values into shared memory
    __shared__ REAL sMatrix1[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL sPartials1[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];
    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        // These are all coherent global memory reads; checked in Profiler
        sPartials1[patIdx][state] = dParentPartials[y + state];
        sPartials2[patIdx][state] = dChildParials[y + state];
    } else {
        sPartials1[patIdx][state] = 0;
        sPartials2[patIdx][state] = 0;
    }

    for (i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE) {
        // load one row of matrices
        if (patIdx < BLOCK_PEELING_SIZE) {
            // These are all coherent global memory reads.
            sMatrix1[patIdx][state] = matrix1[patIdx * PADDED_STATE_COUNT + state];

            // sMatrix now filled with starting in state and ending in i
            matrix1 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
        }
        __syncthreads();

        int j;
        for(j = 0; j < BLOCK_PEELING_SIZE; j++) {
            sum1 += sMatrix1[j][state] * sPartials1[patIdx][i + j];
        }

        __syncthreads(); // GTX280 FIX HERE

    }

    if (pattern < totalPatterns)
        dPartialsTmp[u] = sum1 * sPartials2[patIdx][state];
}

__global__ void kernelPartialsPartialsEdgeLikelihoodsSecondDeriv(REAL* dPartialsTmp,
                                                              REAL* dFirstDerivTmp,
                                                              REAL* dSecondDerivTmp,
                                                              REAL* dParentPartials,
                                                              REAL* dChildParials,
                                                              REAL* dTransMatrix,
                                                              REAL* dFirstDerivMatrix,
                                                              REAL* dSecondDerivMatrix,
                                                              int patternCount) {
    REAL sum1 = 0;
    REAL sumFirstDeriv = 0;
    REAL sumSecondDeriv = 0;

    int i;

    int state = threadIdx.x;
    int patIdx = threadIdx.y;
    int pattern = __umul24(blockIdx.x,PATTERN_BLOCK_SIZE) + patIdx;
    int matrix = blockIdx.y;
    int totalPatterns = patternCount;
    int deltaPartialsByState = pattern * PADDED_STATE_COUNT;
    int deltaPartialsByMatrix = matrix * PADDED_STATE_COUNT * totalPatterns;
    int deltaMatrix = matrix * PADDED_STATE_COUNT * PADDED_STATE_COUNT;
    int u = state + deltaPartialsByState + deltaPartialsByMatrix;

    REAL* matrix1 = dTransMatrix + deltaMatrix; // Points to *this* matrix
    REAL* matrixFirstDeriv = dFirstDerivMatrix + deltaMatrix;
    REAL* matrixSecondDeriv = dSecondDerivMatrix + deltaMatrix;

    int y = deltaPartialsByState + deltaPartialsByMatrix;

    // Load values into shared memory
    __shared__ REAL sMatrix1[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];
    __shared__ REAL sMatrixFirstDeriv[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];
    __shared__ REAL sMatrixSecondDeriv[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL sPartials1[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];
    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        // These are all coherent global memory reads; checked in Profiler
        sPartials1[patIdx][state] = dParentPartials[y + state];
        sPartials2[patIdx][state] = dChildParials[y + state];
    } else {
        sPartials1[patIdx][state] = 0;
        sPartials2[patIdx][state] = 0;
    }

    for (i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE) {
        // load one row of matrices
        if (patIdx < BLOCK_PEELING_SIZE) {
            // These are all coherent global memory reads.
            sMatrix1[patIdx][state] = matrix1[patIdx * PADDED_STATE_COUNT + state];
	        sMatrixFirstDeriv[patIdx][state] = matrixFirstDeriv[patIdx * PADDED_STATE_COUNT + state];
	        sMatrixSecondDeriv[patIdx][state] = matrixSecondDeriv[patIdx * PADDED_STATE_COUNT + state];

            // sMatrix now filled with starting in state and ending in i
            matrix1 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
            matrixFirstDeriv += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
            matrixSecondDeriv += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
        }
        __syncthreads();

        int j;
        for(j = 0; j < BLOCK_PEELING_SIZE; j++) {
            sum1 += sMatrix1[j][state] * sPartials1[patIdx][i + j];
            sumFirstDeriv += sMatrixFirstDeriv[j][state] * sPartials1[patIdx][i + j];
            sumSecondDeriv += sMatrixSecondDeriv[j][state] * sPartials1[patIdx][i + j];
        }

        __syncthreads(); // GTX280 FIX HERE

    }

    if (pattern < totalPatterns) {
        dPartialsTmp[u] = sum1 * sPartials2[patIdx][state];
        dFirstDerivTmp[u] = sumFirstDeriv * sPartials2[patIdx][state];
        dSecondDerivTmp[u] = sumSecondDeriv * sPartials2[patIdx][state];
    }
}

__global__ void kernelStatesPartialsEdgeLikelihoods(REAL* dPartialsTmp,
                                                    REAL* dParentPartials,
                                                    int* dChildStates,
                                                    REAL* dTransMatrix,
                                                    int patternCount) {
    REAL sum1 = 0;

    int state = threadIdx.x;
    int patIdx = threadIdx.y;
    int pattern = __umul24(blockIdx.x,PATTERN_BLOCK_SIZE) + patIdx;
    int matrix = blockIdx.y;
    int totalPatterns = patternCount;
    int deltaPartialsByState = pattern * PADDED_STATE_COUNT;
    int deltaPartialsByMatrix = matrix * PADDED_STATE_COUNT * patternCount;
    int deltaMatrix = matrix * PADDED_STATE_COUNT * PADDED_STATE_COUNT;
    int u = state + deltaPartialsByState + deltaPartialsByMatrix;

    int y = deltaPartialsByState + deltaPartialsByMatrix;

    // Load values into shared memory
    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        sPartials2[patIdx][state] = dParentPartials[y + state];
    } else {
        sPartials2[patIdx][state] = 0;
    }

    if (pattern < totalPatterns) {
        int state1 = dChildStates[pattern]; // Coalesced; no need to share

        REAL* matrix1 = dTransMatrix + deltaMatrix + state1 * PADDED_STATE_COUNT;

        if (state1 < PADDED_STATE_COUNT)
            sum1 = matrix1[state];
        else
            sum1 = 1.0;
    }

    if (pattern < totalPatterns)
        dPartialsTmp[u] = sum1 * sPartials2[patIdx][state];                         
}

__global__ void kernelStatesPartialsEdgeLikelihoodsSecondDeriv(REAL* dPartialsTmp,
                                                              REAL* dFirstDerivTmp,
                                                              REAL* dSecondDerivTmp,
                                                              REAL* dParentPartials,
                                                              int* dChildStates,
                                                              REAL* dTransMatrix,
                                                              REAL* dFirstDerivMatrix,
                                                              REAL* dSecondDerivMatrix,
                                                              int patternCount) {
    REAL sum1 = 0;
    REAL sumFirstDeriv = 0;
    REAL sumSecondDeriv = 0;

    int state = threadIdx.x;
    int patIdx = threadIdx.y;
    int pattern = __umul24(blockIdx.x,PATTERN_BLOCK_SIZE) + patIdx;
    int matrix = blockIdx.y;
    int totalPatterns = patternCount;
    int deltaPartialsByState = pattern * PADDED_STATE_COUNT;
    int deltaPartialsByMatrix = matrix * PADDED_STATE_COUNT * patternCount;
    int deltaMatrix = matrix * PADDED_STATE_COUNT * PADDED_STATE_COUNT;
    int u = state + deltaPartialsByState + deltaPartialsByMatrix;

    int y = deltaPartialsByState + deltaPartialsByMatrix;

    // Load values into shared memory
    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        sPartials2[patIdx][state] = dParentPartials[y + state];
    } else {
        sPartials2[patIdx][state] = 0;
    }

    if (pattern < totalPatterns) {
        int state1 = dChildStates[pattern]; // Coalesced; no need to share

        REAL* matrix1 = dTransMatrix + deltaMatrix + state1 * PADDED_STATE_COUNT;
        REAL* matrixFirstDeriv = dFirstDerivMatrix + deltaMatrix + state1 * PADDED_STATE_COUNT;
        REAL* matrixSecondDeriv = dSecondDerivMatrix + deltaMatrix + state1 * PADDED_STATE_COUNT;

        if (state1 < PADDED_STATE_COUNT) {
            sum1 = matrix1[state];
            sumFirstDeriv = matrixFirstDeriv[state];
            sumSecondDeriv = matrixSecondDeriv[state];
        } else {
            sum1 = 1.0;
            sumFirstDeriv = 0.0;
            sumSecondDeriv = 0.0;
        }
    }

    if (pattern < totalPatterns) {
        dPartialsTmp[u] = sum1 * sPartials2[patIdx][state];
        dFirstDerivTmp[u] = sumFirstDeriv * sPartials2[patIdx][state];
        dSecondDerivTmp[u] = sumSecondDeriv * sPartials2[patIdx][state];
        
    }
}


/*
 * Find a scaling factor for each pattern
 */
__global__ void kernelPartialsDynamicScaling(REAL* allPartials,
                                             REAL* scalingFactors,
                                             int matrixCount) {
    int state = threadIdx.x;
    int matrix = threadIdx.y;
    int pattern = blockIdx.x;
    int patternCount = gridDim.x;

    int deltaPartialsByMatrix = __umul24(matrix, __umul24(PADDED_STATE_COUNT, patternCount));
    
    int offsetPartials = matrix * patternCount * PADDED_STATE_COUNT + pattern * PADDED_STATE_COUNT + state;

    // TODO: Currently assumes MATRIX_BLOCK_SIZE > matrixCount; FIX!!!
    __shared__ REAL partials[MATRIX_BLOCK_SIZE][PADDED_STATE_COUNT];
    __shared__ REAL storedPartials[MATRIX_BLOCK_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL max;

    if (matrix < matrixCount)
        partials[matrix][state] = allPartials[offsetPartials];
    else
        partials[matrix][state] = 0;
        
    storedPartials[matrix][state] = partials[matrix][state];

    __syncthreads();

#ifdef IS_POWER_OF_TWO
    // parallelized reduction *** only works for powers-of-2 ****
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
        if (state < i) {
#else
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {
        if (state < i && state + i < PADDED_STATE_COUNT ) {
#endif // IS_POWER_OF_TWO
    // parallelized reduction; assumes PADDED_STATE_COUNT is power of 2.
            REAL compare1 = partials[matrix][state];
            REAL compare2 = partials[matrix][state + i];
            if (compare2 > compare1)
            partials[matrix][state] = compare2;
        }
        __syncthreads();
    }

    if (state == 0 && matrix == 0) {
        max = 0;
        int m;
        for(m = 0; m < matrixCount; m++) {
            if (partials[m][0] > max)
                max = partials[m][0];
        }
        
        if (max == 0)
        	max = 1.0;

        scalingFactors[pattern] = max; // TODO: These are incoherent memory writes!!!
    }

    __syncthreads();

    if (matrix < matrixCount)
        allPartials[offsetPartials] ///= max;
                    = storedPartials[matrix][state] / max;

    __syncthreads();
}


/*
 * Find a scaling factor for each pattern
 */
__global__ void kernelPartialsDynamicScalingScalersLog(REAL* allPartials,
                                                      REAL* scalingFactors,
                                                      int matrixCount) {
    int state = threadIdx.x;
    int matrix = threadIdx.y;
    int pattern = blockIdx.x;
    int patternCount = gridDim.x;

    int deltaPartialsByMatrix = __umul24(matrix, __umul24(PADDED_STATE_COUNT, patternCount));
    
    int offsetPartials = matrix * patternCount * PADDED_STATE_COUNT + pattern * PADDED_STATE_COUNT + state;

    // TODO: Currently assumes MATRIX_BLOCK_SIZE > matrixCount; FIX!!!
    __shared__ REAL partials[MATRIX_BLOCK_SIZE][PADDED_STATE_COUNT];
    __shared__ REAL storedPartials[MATRIX_BLOCK_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL max;

    if (matrix < matrixCount)
        partials[matrix][state] = allPartials[offsetPartials];
    else
        partials[matrix][state] = 0;
        
    storedPartials[matrix][state] = partials[matrix][state];

    __syncthreads();

#ifdef IS_POWER_OF_TWO
    // parallelized reduction *** only works for powers-of-2 ****
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
        if (state < i) {
#else
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {
        if (state < i && state + i < PADDED_STATE_COUNT ) {
#endif // IS_POWER_OF_TWO
    // parallelized reduction; assumes PADDED_STATE_COUNT is power of 2.
            REAL compare1 = partials[matrix][state];
            REAL compare2 = partials[matrix][state + i];
            if (compare2 > compare1)
            partials[matrix][state] = compare2;
        }
        __syncthreads();
    }

    if (state == 0 && matrix == 0) {
        max = 0;
        int m;
        for(m = 0; m < matrixCount; m++) {
            if (partials[m][0] > max)
                max = partials[m][0];
        }
        
        if (max == 0) {
        	max = 1.0;
            scalingFactors[pattern] = 0.0;
        } else {
            scalingFactors[pattern] = log(max);
        }
    }

    __syncthreads();

    if (matrix < matrixCount)
        allPartials[offsetPartials] ///= max;
                    = storedPartials[matrix][state] / max;

    __syncthreads();
}



/*
 * Find a scaling factor for each pattern and accumulate into buffer
 */
__global__ void kernelPartialsDynamicScalingAccumulate(REAL* allPartials,
                                                       REAL* scalingFactors,
                                                       REAL* cumulativeScaling,
                                                       int matrixCount) {
    int state = threadIdx.x;
    int matrix = threadIdx.y;
    int pattern = blockIdx.x;
    int patternCount = gridDim.x;

    int deltaPartialsByMatrix = __umul24(matrix, __umul24(PADDED_STATE_COUNT, patternCount));

    // TODO: Currently assumes MATRIX_BLOCK_SIZE > matrixCount; FIX!!!
    __shared__ REAL partials[MATRIX_BLOCK_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL max;

    if (matrix < matrixCount)
        partials[matrix][state] = allPartials[matrix * patternCount * PADDED_STATE_COUNT + pattern *
                                              PADDED_STATE_COUNT + state];
    else
        partials[matrix][state] = 0;

    __syncthreads();
  
#ifdef IS_POWER_OF_TWO
    // parallelized reduction *** only works for powers-of-2 ****
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
        if (state < i) {
#else
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {
        if (state < i && state + i < PADDED_STATE_COUNT ) {
#endif // IS_POWER_OF_TWO        
            REAL compare1 = partials[matrix][state];
            REAL compare2 = partials[matrix][state + i];
            if (compare2 > compare1)
            partials[matrix][state] = compare2;
        }
        __syncthreads();
    }

    if (state == 0 && matrix == 0) {
        max = 0;
        int m;
        for(m = 0; m < matrixCount; m++) {
            if (partials[m][0] > max)
                max = partials[m][0];
        }
        
        if (max == 0)
        	max = 1.0;

        scalingFactors[pattern] = max; // TODO: These are incoherent memory writes!!!
        cumulativeScaling[pattern] += log(max);

    }

    __syncthreads();

    if (matrix < matrixCount)
        allPartials[matrix * patternCount * PADDED_STATE_COUNT + pattern * PADDED_STATE_COUNT +
                    state] /= max;

    __syncthreads();
}

/*
 * Find a scaling factor for each pattern and accumulate into buffer
 */
__global__ void kernelPartialsDynamicScalingAccumulateScalersLog(REAL* allPartials,
                                                                REAL* scalingFactors,
                                                                REAL* cumulativeScaling,
                                                                int matrixCount) {
    int state = threadIdx.x;
    int matrix = threadIdx.y;
    int pattern = blockIdx.x;
    int patternCount = gridDim.x;

    int deltaPartialsByMatrix = __umul24(matrix, __umul24(PADDED_STATE_COUNT, patternCount));

    // TODO: Currently assumes MATRIX_BLOCK_SIZE > matrixCount; FIX!!!
    __shared__ REAL partials[MATRIX_BLOCK_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL max;

    if (matrix < matrixCount)
        partials[matrix][state] = allPartials[matrix * patternCount * PADDED_STATE_COUNT + pattern *
                                              PADDED_STATE_COUNT + state];
    else
        partials[matrix][state] = 0;

    __syncthreads();
  
#ifdef IS_POWER_OF_TWO
    // parallelized reduction *** only works for powers-of-2 ****
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
        if (state < i) {
#else
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {
        if (state < i && state + i < PADDED_STATE_COUNT ) {
#endif // IS_POWER_OF_TWO        
            REAL compare1 = partials[matrix][state];
            REAL compare2 = partials[matrix][state + i];
            if (compare2 > compare1)
            partials[matrix][state] = compare2;
        }
        __syncthreads();
    }

    if (state == 0 && matrix == 0) {
        max = 0;
        int m;
        for(m = 0; m < matrixCount; m++) {
            if (partials[m][0] > max)
                max = partials[m][0];
        }
        
        if (max == 0) {
        	max = 1.0;
            scalingFactors[pattern] = 0.0;
        } else {
            REAL logMax = log(max);
            scalingFactors[pattern] = logMax;
            cumulativeScaling[pattern] += logMax;
        }

    }

    __syncthreads();

    if (matrix < matrixCount)
        allPartials[matrix * patternCount * PADDED_STATE_COUNT + pattern * PADDED_STATE_COUNT +
                    state] /= max;

    __syncthreads();
}


__global__ void kernelIntegrateLikelihoodsFixedScale(REAL* dResult,
                                                            REAL* dRootPartials,
                                                            REAL *dWeights,
                                                            REAL *dFrequencies,
                                                            REAL *dRootScalingFactors,
                                                            REAL* dPatternWeights,
                                                            int matrixCount,
                                                            int patternCount) {
    int state   = threadIdx.x;
    int pattern = blockIdx.x;
//    int patternCount = gridDim.x;

    __shared__ REAL stateFreq[PADDED_STATE_COUNT];
    // TODO: Currently assumes MATRIX_BLOCK_SIZE >> matrixCount
    __shared__ REAL matrixProp[MATRIX_BLOCK_SIZE];
    __shared__ REAL sum[PADDED_STATE_COUNT];

    // Load shared memory

    stateFreq[state] = dFrequencies[state];
    sum[state] = 0;

    for(int matrixEdge = 0; matrixEdge < matrixCount; matrixEdge += PADDED_STATE_COUNT) {
        int x = matrixEdge + state;
        if (x < matrixCount)
            matrixProp[x] = dWeights[x];
    }

    __syncthreads();

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;;

    for(int r = 0; r < matrixCount; r++) {
        sum[state] += dRootPartials[u + delta * r] * matrixProp[r];
    }

    sum[state] *= stateFreq[state];
    __syncthreads();

#ifdef IS_POWER_OF_TWO
    // parallelized reduction *** only works for powers-of-2 ****
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
        if (state < i) {
#else
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {
        if (state < i && state + i < PADDED_STATE_COUNT ) {
#endif // IS_POWER_OF_TWO
            sum[state] += sum[state + i];
        }
        __syncthreads();
    }

    if (state == 0)
        dResult[pattern] = (log(sum[state]) + dRootScalingFactors[pattern]) * dPatternWeights[pattern];
}

__global__ void kernelIntegrateLikelihoodsAutoScaling(REAL* dResult,
                                                     REAL* dRootPartials,
                                                     REAL* dWeights,
                                                     REAL* dFrequencies,
                                                     int* dRootScalingFactors,
                                                     REAL* dPatternWeights,
                                                     int matrixCount,
                                                     int patternCount) {
    int state   = threadIdx.x;
    int pattern = blockIdx.x;
//    int patternCount = gridDim.x;

    __shared__ REAL stateFreq[PADDED_STATE_COUNT];
    // TODO: Currently assumes MATRIX_BLOCK_SIZE >> matrixCount
    __shared__ REAL matrixProp[MATRIX_BLOCK_SIZE];
    __shared__ REAL matrixScalers[MATRIX_BLOCK_SIZE];
    __shared__ REAL sum[PADDED_STATE_COUNT];

    // Load shared memory

    stateFreq[state] = dFrequencies[state];
    sum[state] = 0;

    for(int matrixEdge = 0; matrixEdge < matrixCount; matrixEdge += PADDED_STATE_COUNT) {
        int x = matrixEdge + state;
        if (x < matrixCount) {
            matrixProp[x] = dWeights[x];
            matrixScalers[x] = dRootScalingFactors[pattern + (x * patternCount)];
        }
    }

    __syncthreads();

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;
    
    short maxScaleFactor = matrixScalers[0];
    for(int r = 1; r < matrixCount; r++) {
        int tmpFactor = matrixScalers[r];
        if (tmpFactor > maxScaleFactor)
            maxScaleFactor = tmpFactor;
    }
    
    for(int r = 0; r < matrixCount; r++) {
        int tmpFactor = matrixScalers[r];
        if (tmpFactor != maxScaleFactor) {
            int expTmp;
            sum[state] += ldexp(frexp(dRootPartials[u + delta * r], &expTmp), expTmp + (tmpFactor - maxScaleFactor)) * matrixProp[r];
        } else {
            sum[state] += dRootPartials[u + delta * r] * matrixProp[r];
        }
    }

    sum[state] *= stateFreq[state];
    __syncthreads();

#ifdef IS_POWER_OF_TWO
    // parallelized reduction *** only works for powers-of-2 ****
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
        if (state < i) {
#else
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {
        if (state < i && state + i < PADDED_STATE_COUNT ) {
#endif // IS_POWER_OF_TWO
            sum[state] += sum[state + i];
        }
        __syncthreads();
    }

    if (state == 0)
        dResult[pattern] = (log(sum[state]) + (M_LN2 * maxScaleFactor)) * dPatternWeights[pattern];
}


__global__ void kernelIntegrateLikelihoodsFixedScaleSecondDeriv(REAL* dResult,
                                              REAL* dFirstDerivResult,
                                              REAL* dSecondDerivResult,
                                              REAL* dRootPartials,
                                              REAL* dRootFirstDeriv,
                                              REAL* dRootSecondDeriv,
                                              REAL* dWeights,
                                              REAL* dFrequencies,
                                              REAL *dRootScalingFactors,
                                              REAL* dPatternWeights,
                                              int matrixCount,
                                              int patternCount) {
    int state   = threadIdx.x;
    int pattern = blockIdx.x;
//    int patternCount = gridDim.x;

    REAL tmpLogLike = 0.0;
    REAL tmpFirstDeriv = 0.0;

    __shared__ REAL stateFreq[PADDED_STATE_COUNT];
    // TODO: Currently assumes MATRIX_BLOCK_SIZE >> matrixCount
    __shared__ REAL matrixProp[MATRIX_BLOCK_SIZE];
    __shared__ REAL sum[PADDED_STATE_COUNT];
    __shared__ REAL sumD1[PADDED_STATE_COUNT];
    __shared__ REAL sumD2[PADDED_STATE_COUNT];

    // Load shared memory

    stateFreq[state] = dFrequencies[state];
    sum[state] = 0;
    sumD1[state] = 0;
    sumD2[state] = 0;

    for(int matrixEdge = 0; matrixEdge < matrixCount; matrixEdge += PADDED_STATE_COUNT) {
        int x = matrixEdge + state;
        if (x < matrixCount)
            matrixProp[x] = dWeights[x];
    }

    __syncthreads();

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;;

    for(int r = 0; r < matrixCount; r++) {
        sum[state] += dRootPartials[u + delta * r] * matrixProp[r];
        sumD1[state] += dRootFirstDeriv[u + delta * r] * matrixProp[r];
        sumD2[state] += dRootSecondDeriv[u + delta * r] * matrixProp[r];
    }

    sum[state] *= stateFreq[state];
    sumD1[state] *= stateFreq[state];
    sumD2[state] *= stateFreq[state];    
    __syncthreads();

#ifdef IS_POWER_OF_TWO
    // parallelized reduction *** only works for powers-of-2 ****
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
        if (state < i) {
#else
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {
        if (state < i && state + i < PADDED_STATE_COUNT ) {
#endif // IS_POWER_OF_TWO
            sum[state] += sum[state + i];
            sumD1[state] += sumD1[state + i];
            sumD2[state] += sumD2[state + i];
        }
        __syncthreads();
    }

    if (state == 0) {
        tmpLogLike = sum[state];
        dResult[pattern] = (log(tmpLogLike) + dRootScalingFactors[pattern]) * dPatternWeights[pattern];
        
        tmpFirstDeriv = sumD1[state] / tmpLogLike;
        dFirstDerivResult[pattern] = tmpFirstDeriv * dPatternWeights[pattern];
        
        dSecondDerivResult[pattern] = (sumD2[state] / tmpLogLike - tmpFirstDeriv * tmpFirstDeriv) * dPatternWeights[pattern];
    }
}


__global__ void kernelIntegrateLikelihoods(REAL* dResult,
                                              REAL* dRootPartials,
                                              REAL* dWeights,
                                              REAL* dFrequencies,
                                              REAL* dPatternWeights,
                                              int matrixCount,
                                              int patternCount) {
    int state   = threadIdx.x;
    int pattern = blockIdx.x;
//    int patternCount = gridDim.x;

    __shared__ REAL stateFreq[PADDED_STATE_COUNT];
    // TODO: Currently assumes MATRIX_BLOCK_SIZE >> matrixCount
    __shared__ REAL matrixProp[MATRIX_BLOCK_SIZE];
    __shared__ REAL sum[PADDED_STATE_COUNT];

    // Load shared memory

    stateFreq[state] = dFrequencies[state];
    sum[state] = 0;

    for(int matrixEdge = 0; matrixEdge < matrixCount; matrixEdge += PADDED_STATE_COUNT) {
        int x = matrixEdge + state;
        if (x < matrixCount)
            matrixProp[x] = dWeights[x];
    }

    __syncthreads();

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;

    for(int r = 0; r < matrixCount; r++) {
        sum[state] += dRootPartials[u + delta * r] * matrixProp[r];
    }

    sum[state] *= stateFreq[state];
    __syncthreads();

#ifdef IS_POWER_OF_TWO
    // parallelized reduction *** only works for powers-of-2 ****
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
        if (state < i) {
#else
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {
        if (state < i && state + i < PADDED_STATE_COUNT ) {
#endif // IS_POWER_OF_TWO
            sum[state] += sum[state + i];
        }
        __syncthreads();
    }

    if (state == 0)
        dResult[pattern] = log(sum[state]) * dPatternWeights[pattern];
}

__global__ void kernelIntegrateLikelihoodsSecondDeriv(REAL* dResult,
                                              REAL* dFirstDerivResult,
                                              REAL* dSecondDerivResult,
                                              REAL* dRootPartials,
                                              REAL* dRootFirstDeriv,
                                              REAL* dRootSecondDeriv,
                                              REAL* dWeights,
                                              REAL* dFrequencies,
                                              REAL* dPatternWeights,
                                              int matrixCount,
                                              int patternCount) {
    int state   = threadIdx.x;
    int pattern = blockIdx.x;
//    int patternCount = gridDim.x;

    REAL tmpLogLike = 0.0;
    REAL tmpFirstDeriv = 0.0;

    __shared__ REAL stateFreq[PADDED_STATE_COUNT];
    // TODO: Currently assumes MATRIX_BLOCK_SIZE >> matrixCount
    __shared__ REAL matrixProp[MATRIX_BLOCK_SIZE];
    __shared__ REAL sum[PADDED_STATE_COUNT];
    __shared__ REAL sumD1[PADDED_STATE_COUNT];
    __shared__ REAL sumD2[PADDED_STATE_COUNT];

    // Load shared memory

    stateFreq[state] = dFrequencies[state];
    sum[state] = 0;
    sumD1[state] = 0;
    sumD2[state] = 0;

    for(int matrixEdge = 0; matrixEdge < matrixCount; matrixEdge += PADDED_STATE_COUNT) {
        int x = matrixEdge + state;
        if (x < matrixCount)
            matrixProp[x] = dWeights[x];
    }

    __syncthreads();

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;

    for(int r = 0; r < matrixCount; r++) {
        sum[state] += dRootPartials[u + delta * r] * matrixProp[r];
        sumD1[state] += dRootFirstDeriv[u + delta * r] * matrixProp[r];
        sumD2[state] += dRootSecondDeriv[u + delta * r] * matrixProp[r];
    }

    sum[state] *= stateFreq[state];
    sumD1[state] *= stateFreq[state];
    sumD2[state] *= stateFreq[state];
    __syncthreads();

#ifdef IS_POWER_OF_TWO
    // parallelized reduction *** only works for powers-of-2 ****
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
        if (state < i) {
#else
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {
        if (state < i && state + i < PADDED_STATE_COUNT ) {
#endif // IS_POWER_OF_TWO
            sum[state] += sum[state + i];
            sumD1[state] += sumD1[state + i];
            sumD2[state] += sumD2[state + i];
        }
        __syncthreads();
    }

    if (state == 0) {
        tmpLogLike = sum[state];
        dResult[pattern] = log(tmpLogLike) * dPatternWeights[pattern];
        
        tmpFirstDeriv = sumD1[state] / tmpLogLike;
        dFirstDerivResult[pattern] = tmpFirstDeriv * dPatternWeights[pattern];
        
        dSecondDerivResult[pattern] = (sumD2[state] / tmpLogLike - tmpFirstDeriv * tmpFirstDeriv) * dPatternWeights[pattern];
    }
}


__global__ void kernelIntegrateLikelihoodsMulti(REAL* dResult,
                                              REAL* dRootPartials,
                                              REAL* dWeights,
                                              REAL* dFrequencies,
                                              REAL* dPatternWeights,
                                              int matrixCount,
                                              int patternCount,
											  int takeLog) {
    int state   = threadIdx.x;
    int pattern = blockIdx.x;
//    int patternCount = gridDim.x;

    __shared__ REAL stateFreq[PADDED_STATE_COUNT];
    // TODO: Currently assumes MATRIX_BLOCK_SIZE >> matrixCount
    __shared__ REAL matrixProp[MATRIX_BLOCK_SIZE];
    __shared__ REAL sum[PADDED_STATE_COUNT];

    // Load shared memory

    stateFreq[state] = dFrequencies[state];
    sum[state] = 0;

    for(int matrixEdge = 0; matrixEdge < matrixCount; matrixEdge += PADDED_STATE_COUNT) {
        int x = matrixEdge + state;
        if (x < matrixCount)
            matrixProp[x] = dWeights[x];
    }

    __syncthreads();

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;

    for(int r = 0; r < matrixCount; r++) {
        sum[state] += dRootPartials[u + delta * r] * matrixProp[r];
    }

    sum[state] *= stateFreq[state];
    __syncthreads();

#ifdef IS_POWER_OF_TWO
    // parallelized reduction *** only works for powers-of-2 ****
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
        if (state < i) {
#else
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {
        if (state < i && state + i < PADDED_STATE_COUNT ) {
#endif // IS_POWER_OF_TWO
            sum[state] += sum[state + i];
        }
        __syncthreads();
    }

    if (state == 0) {
		if (takeLog == 0)
			dResult[pattern] = sum[state]; 
		else if (takeLog == 1)
			dResult[pattern] = log(dResult[pattern] + sum[state]) * dPatternWeights[pattern];
		else
			dResult[pattern] += sum[state]; 
	}

}

__global__ void kernelIntegrateLikelihoodsFixedScaleMulti(REAL* dResult,
											  REAL* dRootPartials,
                                              REAL* dWeights,
                                              REAL* dFrequencies,
											  REAL** dPtrQueue,
											  REAL* dMaxScalingFactors,
											  REAL* dIndexMaxScalingFactors,
                                              REAL* dPatternWeights,
                                              int matrixCount,
                                              int patternCount,
											  int subsetCount,
											  int subsetIndex) {
    int state   = threadIdx.x;
    int pattern = blockIdx.x;
//    int patternCount = gridDim.x;

    __shared__ REAL stateFreq[PADDED_STATE_COUNT];
    // TODO: Currently assumes MATRIX_BLOCK_SIZE >> matrixCount
    __shared__ REAL matrixProp[MATRIX_BLOCK_SIZE];
    __shared__ REAL sum[PADDED_STATE_COUNT];

    // Load shared memory

    stateFreq[state] = dFrequencies[state];
    sum[state] = 0;

    for(int matrixEdge = 0; matrixEdge < matrixCount; matrixEdge += PADDED_STATE_COUNT) {
        int x = matrixEdge + state;
        if (x < matrixCount)
            matrixProp[x] = dWeights[x];
    }

    __syncthreads();

    int u = state + pattern * PADDED_STATE_COUNT;
    int delta = patternCount * PADDED_STATE_COUNT;

    for(int r = 0; r < matrixCount; r++) {
        sum[state] += dRootPartials[u + delta * r] * matrixProp[r];
    }

    sum[state] *= stateFreq[state];
    __syncthreads();

#ifdef IS_POWER_OF_TWO
    // parallelized reduction *** only works for powers-of-2 ****
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
        if (state < i) {
#else
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {
        if (state < i && state + i < PADDED_STATE_COUNT ) {
#endif // IS_POWER_OF_TWO
            sum[state] += sum[state + i];
        }
        __syncthreads();
    }
	
	REAL cumulativeScalingFactor = ((REAL*) *((int*)dPtrQueue + subsetIndex))[pattern];
	
	if (subsetIndex == 0) {
		int indexMaxScalingFactor = 0;
		REAL maxScalingFactor = cumulativeScalingFactor;
		for (int j = 1; j < subsetCount; j++) {
			REAL tmpScalingFactor = ((REAL*) *((int*)dPtrQueue + j))[pattern];
			if (tmpScalingFactor > maxScalingFactor) {
				indexMaxScalingFactor = j;
				maxScalingFactor = tmpScalingFactor;
			}
		}
		
		dIndexMaxScalingFactors[pattern] = indexMaxScalingFactor;
		dMaxScalingFactors[pattern] = maxScalingFactor;	
		
		if (indexMaxScalingFactor != 0)
			sum[state] *= exp((REAL)(cumulativeScalingFactor - maxScalingFactor));
			
		if (state == 0)
			dResult[pattern] = sum[state];
	} else {
		if (subsetIndex != dIndexMaxScalingFactors[pattern])
			sum[state] *= exp((REAL)(cumulativeScalingFactor - dMaxScalingFactors[pattern]));
	
		if (state == 0) {
			if (subsetIndex == subsetCount - 1)
				dResult[pattern] = (log(dResult[pattern] + sum[state]) + dMaxScalingFactors[pattern]) * dPatternWeights[pattern];
			else
				dResult[pattern] += sum[state];
		}
	}        
}


} // extern "C"

