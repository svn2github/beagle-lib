#include "hip/hip_runtime.h"
/*
 *
 * Copyright 2009 Phylogenetic Likelihood Working Group
 *
 * This file is part of BEAGLE.
 *
 * BEAGLE is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * BEAGLE is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with BEAGLE.  If not, see
 * <http://www.gnu.org/licenses/>.
 *
 * @author Marc Suchard
 * @author Daniel Ayres
 */

#include "libhmsbeagle/GPU/GPUImplDefs.h"
#include "libhmsbeagle/GPU/kernels/kernelsAll.cu" // This file includes the non-state-count specific kernels

#define DETERMINE_INDICES() \
    int state = threadIdx.x; \
    int patIdx = threadIdx.y; \
    int pattern = __umul24(blockIdx.x,PATTERN_BLOCK_SIZE) + patIdx; \
    int matrix = blockIdx.y; \
    int patternCount = totalPatterns; \
    int deltaPartialsByState = pattern * PADDED_STATE_COUNT; \
    int deltaPartialsByMatrix = matrix * PADDED_STATE_COUNT * patternCount; \
    int deltaMatrix = matrix * PADDED_STATE_COUNT * PADDED_STATE_COUNT; \
    int u = state + deltaPartialsByState + deltaPartialsByMatrix;

extern "C" {

__global__ void kernelPartialsPartialsNoScale(REAL* partials1,
                                                             REAL* partials2,
                                                             REAL* partials3,
                                                             REAL* matrices1,
                                                             REAL* matrices2,
                                                             int totalPatterns) {
    REAL sum1 = 0;
    REAL sum2 = 0;
    int i;

    DETERMINE_INDICES();

    REAL* matrix1 = matrices1 + deltaMatrix; // Points to *this* matrix
    REAL* matrix2 = matrices2 + deltaMatrix;

    int y = deltaPartialsByState + deltaPartialsByMatrix;

    // Load values into shared memory
    __shared__ REAL sMatrix1[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];
    __shared__ REAL sMatrix2[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL sPartials1[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];
    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        // These are all coherent global memory reads; checked in Profiler
        sPartials1[patIdx][state] = partials1[y + state];
        sPartials2[patIdx][state] = partials2[y + state];
    } else {
        sPartials1[patIdx][state] = 0;
        sPartials2[patIdx][state] = 0;
    }

    for (i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE) {
        // load one row of matrices
        if (patIdx < BLOCK_PEELING_SIZE) {
            // These are all coherent global memory reads.
            sMatrix1[patIdx][state] = matrix1[patIdx * PADDED_STATE_COUNT + state];
            sMatrix2[patIdx][state] = matrix2[patIdx * PADDED_STATE_COUNT + state];

            // sMatrix now filled with starting in state and ending in i
            matrix1 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
            matrix2 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
        }
        __syncthreads();

        int j;
        for(j = 0; j < BLOCK_PEELING_SIZE; j++) {
            sum1 += sMatrix1[j][state] * sPartials1[patIdx][i + j];
            sum2 += sMatrix2[j][state] * sPartials2[patIdx][i + j];
        }

        __syncthreads(); // GTX280 FIX HERE

    }

    if (pattern < totalPatterns)
        partials3[u] = sum1 * sum2;
}

__global__ void kernelPartialsPartialsFixedScale(REAL* partials1,
                                                                 REAL* partials2,
                                                                 REAL* partials3,
                                                                 REAL* matrices1,
                                                                 REAL* matrices2,
                                                                 REAL* scalingFactors,
                                                                 int totalPatterns) {
    REAL sum1 = 0;
    REAL sum2 = 0;
    int i;

    DETERMINE_INDICES();

    REAL* matrix1 = matrices1 + deltaMatrix; // Points to *this* matrix
    REAL* matrix2 = matrices2 + deltaMatrix;

    int y = deltaPartialsByState + deltaPartialsByMatrix;

    // Load values into shared memory
    __shared__ REAL sMatrix1[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];
    __shared__ REAL sMatrix2[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL sPartials1[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];
    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL fixedScalingFactors[PATTERN_BLOCK_SIZE];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        // These are all coherent global memory reads; checked in Profiler
        sPartials1[patIdx][state] = partials1[y + state];
        sPartials2[patIdx][state] = partials2[y + state];
    } else {
        sPartials1[patIdx][state] = 0;
        sPartials2[patIdx][state] = 0;
    }

    if (patIdx == 0 && state < PATTERN_BLOCK_SIZE )
        // TODO: If PATTERN_BLOCK_SIZE > PADDED_STATE_COUNT, there is a bug here
        fixedScalingFactors[state] = scalingFactors[blockIdx.x * PATTERN_BLOCK_SIZE + state];

    for (i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE) {
        // load one row of matrices
        if (patIdx < BLOCK_PEELING_SIZE) {
            // These are all coherent global memory reads.
            sMatrix1[patIdx][state] = matrix1[patIdx * PADDED_STATE_COUNT + state];
            sMatrix2[patIdx][state] = matrix2[patIdx * PADDED_STATE_COUNT + state];

            // sMatrix now filled with starting in state and ending in i
            matrix1 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
            matrix2 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
        }
        __syncthreads();

        int j;
        for(j = 0; j < BLOCK_PEELING_SIZE; j++) {
            sum1 += sMatrix1[j][state] * sPartials1[patIdx][i + j];
            sum2 += sMatrix2[j][state] * sPartials2[patIdx][i + j];
        }

        __syncthreads(); // GTX280 FIX HERE

    }

    if (pattern < totalPatterns)
        partials3[u] = sum1 * sum2 / fixedScalingFactors[patIdx];

}

__global__ void kernelStatesPartialsNoScale(int* states1,
                                                           REAL* partials2,
                                                           REAL* partials3,
                                                           REAL* matrices1,
                                                           REAL* matrices2,
                                                           int totalPatterns) {
    REAL sum1 = 0;
    REAL sum2 = 0;
    int i;

    DETERMINE_INDICES();

    int y = deltaPartialsByState + deltaPartialsByMatrix;

    // Load values into shared memory
    __shared__ REAL sMatrix2[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        sPartials2[patIdx][state] = partials2[y + state];
    } else {
        sPartials2[patIdx][state] = 0;
    }

    REAL* matrix2 = matrices2 + deltaMatrix;

    if (pattern < totalPatterns) {
        int state1 = states1[pattern]; // Coalesced; no need to share

        REAL* matrix1 = matrices1 + deltaMatrix + state1 * PADDED_STATE_COUNT;

        if (state1 < PADDED_STATE_COUNT)
            sum1 = matrix1[state];
        else
            sum1 = 1.0;
    }

    for (i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE) {
        // load one row of matrices
        if (patIdx < BLOCK_PEELING_SIZE) {
            sMatrix2[patIdx][state] = matrix2[patIdx * PADDED_STATE_COUNT + state];

            // sMatrix now filled with starting in state and ending in i
            matrix2 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
        }
        __syncthreads();

        int j;
        for(j = 0; j < BLOCK_PEELING_SIZE; j++) {
            sum2 += sMatrix2[j][state] * sPartials2[patIdx][i + j];
        }

        __syncthreads(); // GTX280 FIX HERE

    }

    if (pattern < totalPatterns)
        partials3[u] = sum1 * sum2;
}

__global__ void kernelStatesPartialsFixedScale(int* states1,
                                                               REAL* partials2,
                                                               REAL* partials3,
                                                               REAL* matrices1,
                                                               REAL* matrices2,
                                                               REAL* scalingFactors,
                                                               int totalPatterns) {
    REAL sum1 = 0;
    REAL sum2 = 0;
    int i;

    DETERMINE_INDICES();

    int y = deltaPartialsByState + deltaPartialsByMatrix;

    // Load values into shared memory
    __shared__ REAL sMatrix2[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL fixedScalingFactors[PATTERN_BLOCK_SIZE];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        sPartials2[patIdx][state] = partials2[y + state];
    } else {
        sPartials2[patIdx][state] = 0;
    }

    REAL* matrix2 = matrices2 + deltaMatrix;

    if (pattern < totalPatterns) {
        int state1 = states1[pattern]; // Coalesced; no need to share

        REAL* matrix1 = matrices1 + deltaMatrix + state1 * PADDED_STATE_COUNT;

        if (state1 < PADDED_STATE_COUNT)
            sum1 = matrix1[state];
        else
            sum1 = 1.0;
    }

    if (patIdx == 0 && state < PATTERN_BLOCK_SIZE )
        // TODO: If PATTERN_BLOCK_SIZE > PADDED_STATE_COUNT, there is a bug here
        fixedScalingFactors[state] = scalingFactors[blockIdx.x * PATTERN_BLOCK_SIZE + state];

    for (i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE) {
        // load one row of matrices
        if (patIdx < BLOCK_PEELING_SIZE) {
            sMatrix2[patIdx][state] = matrix2[patIdx * PADDED_STATE_COUNT + state];

            // sMatrix now filled with starting in state and ending in i
            matrix2 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
        }
        __syncthreads();

        int j;
        for(j = 0; j < BLOCK_PEELING_SIZE; j++) {
            sum2 += sMatrix2[j][state] * sPartials2[patIdx][i + j];
        }

        __syncthreads(); // GTX280 FIX HERE

    }

    if (pattern < totalPatterns)
        partials3[u] = sum1 * sum2 / fixedScalingFactors[patIdx];
}

__global__ void kernelStatesStatesNoScale(int* states1,
                                                         int* states2,
                                                         REAL* partials3,
                                                         REAL* matrices1,
                                                         REAL* matrices2,
                                                         int totalPatterns) {
    DETERMINE_INDICES();

    // Load values into shared memory
//  __shared__ REAL sMatrix1[PADDED_STATE_COUNT];
//  __shared__ REAL sMatrix2[PADDED_STATE_COUNT];

    int state1 = states1[pattern];
    int state2 = states2[pattern];

    // Points to *this* matrix
    REAL* matrix1 = matrices1 + deltaMatrix + state1 * PADDED_STATE_COUNT;
    REAL* matrix2 = matrices2 + deltaMatrix + state2 * PADDED_STATE_COUNT;

//  if (patIdx == 0) {
//      sMatrix1[state] = matrix1[state];
//      sMatrix2[state] = matrix2[state];
//  }

    __syncthreads();

    if (pattern < totalPatterns) {

        if ( state1 < PADDED_STATE_COUNT && state2 < PADDED_STATE_COUNT) {
//          partials3[u] = sMatrix1[state] * sMatrix2[state];
            partials3[u] = matrix1[state] * matrix2[state];
        } else if (state1 < PADDED_STATE_COUNT) {
//          partials3[u] = sMatrix1[state];
            partials3[u] = matrix1[state];
        } else if (state2 < PADDED_STATE_COUNT) {
//          partials3[u] = sMatrix2[state];
            partials3[u] = matrix2[state];
        } else {
            partials3[u] = 1.0;
        }
    }
}

__global__ void kernelStatesStatesFixedScale(int* states1,
                                                             int* states2,
                                                             REAL* partials3,
                                                             REAL* matrices1,
                                                             REAL* matrices2,
                                                             REAL* scalingFactors,
                                                             int totalPatterns) {
    DETERMINE_INDICES();

    // Load values into shared memory
    // Prefetching into shared memory gives no performance gain
    // TODO: Double-check.
//  __shared__ REAL sMatrix1[PADDED_STATE_COUNT];
//  __shared__ REAL sMatrix2[PADDED_STATE_COUNT];

    __shared__ REAL fixedScalingFactors[PATTERN_BLOCK_SIZE];

    int state1 = states1[pattern];
    int state2 = states2[pattern];

    // Points to *this* matrix
    REAL* matrix1 = matrices1 + deltaMatrix + state1 * PADDED_STATE_COUNT;
    REAL* matrix2 = matrices2 + deltaMatrix + state2 * PADDED_STATE_COUNT;

//  if (patIdx == 0) {
//      sMatrix1[state] = matrix1[state];
//      sMatrix2[state] = matrix2[state];
//  }

    // TODO: If PATTERN_BLOCK_SIZE > PADDED_STATE_COUNT, there is a bug here
    if (patIdx == 0 && state < PATTERN_BLOCK_SIZE )
        fixedScalingFactors[state] = scalingFactors[blockIdx.x * PATTERN_BLOCK_SIZE + state];

    __syncthreads();

    if (pattern < totalPatterns) {
        if (state1 < PADDED_STATE_COUNT && state2 < PADDED_STATE_COUNT) {
//          partials3[u] = sMatrix1[state] * sMatrix2[state];
            partials3[u] = matrix1[state] * matrix2[state] / fixedScalingFactors[patIdx];
        } else if (state1 < PADDED_STATE_COUNT) {
//          partials3[u] = sMatrix1[state];
            partials3[u] = matrix1[state] / fixedScalingFactors[patIdx];
        } else if (state2 < PADDED_STATE_COUNT) {
//          partials3[u] = sMatrix2[state];
            partials3[u] = matrix2[state] / fixedScalingFactors[patIdx];
        } else {
            partials3[u] = 1.0 / fixedScalingFactors[patIdx];
        }
    }
}

__global__ void kernelPartialsPartialsEdgeLikelihoods(REAL* dPartialsTmp,
                                                         REAL* dParentPartials,
                                                         REAL* dChildParials,
                                                         REAL* dTransMatrix,
                                                         int patternCount) {
    REAL sum1 = 0;

    int i;

    int state = threadIdx.x;
    int patIdx = threadIdx.y;
    int pattern = __umul24(blockIdx.x,PATTERN_BLOCK_SIZE) + patIdx;
    int matrix = blockIdx.y;
    int totalPatterns = patternCount;
    int deltaPartialsByState = pattern * PADDED_STATE_COUNT;
    int deltaPartialsByMatrix = matrix * PADDED_STATE_COUNT * totalPatterns;
    int deltaMatrix = matrix * PADDED_STATE_COUNT * PADDED_STATE_COUNT;
    int u = state + deltaPartialsByState + deltaPartialsByMatrix;

    REAL* matrix1 = dTransMatrix + deltaMatrix; // Points to *this* matrix

    int y = deltaPartialsByState + deltaPartialsByMatrix;

    // Load values into shared memory
    __shared__ REAL sMatrix1[BLOCK_PEELING_SIZE][PADDED_STATE_COUNT];

    __shared__ REAL sPartials1[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];
    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        // These are all coherent global memory reads; checked in Profiler
        sPartials1[patIdx][state] = dParentPartials[y + state];
        sPartials2[patIdx][state] = dChildParials[y + state];
    } else {
        sPartials1[patIdx][state] = 0;
        sPartials2[patIdx][state] = 0;
    }

    for (i = 0; i < PADDED_STATE_COUNT; i += BLOCK_PEELING_SIZE) {
        // load one row of matrices
        if (patIdx < BLOCK_PEELING_SIZE) {
            // These are all coherent global memory reads.
            sMatrix1[patIdx][state] = matrix1[patIdx * PADDED_STATE_COUNT + state];

            // sMatrix now filled with starting in state and ending in i
            matrix1 += BLOCK_PEELING_SIZE * PADDED_STATE_COUNT;
        }
        __syncthreads();

        int j;
        for(j = 0; j < BLOCK_PEELING_SIZE; j++) {
            sum1 += sMatrix1[j][state] * sPartials1[patIdx][i + j];
        }

        __syncthreads(); // GTX280 FIX HERE

    }

    if (pattern < totalPatterns)
        dPartialsTmp[u] = sum1 * sPartials2[patIdx][state];
}

__global__ void kernelStatesPartialsEdgeLikelihoods(REAL* dPartialsTmp,
                                                    REAL* dParentPartials,
                                                    int* dChildStates,
                                                    REAL* dTransMatrix,
                                                    int patternCount) {
    REAL sum1 = 0;

    int state = threadIdx.x;
    int patIdx = threadIdx.y;
    int pattern = __umul24(blockIdx.x,PATTERN_BLOCK_SIZE) + patIdx;
    int matrix = blockIdx.y;
    int totalPatterns = patternCount;
    int deltaPartialsByState = pattern * PADDED_STATE_COUNT;
    int deltaPartialsByMatrix = matrix * PADDED_STATE_COUNT * patternCount;
    int deltaMatrix = matrix * PADDED_STATE_COUNT * PADDED_STATE_COUNT;
    int u = state + deltaPartialsByState + deltaPartialsByMatrix;

    int y = deltaPartialsByState + deltaPartialsByMatrix;

    // Load values into shared memory
    __shared__ REAL sPartials2[PATTERN_BLOCK_SIZE][PADDED_STATE_COUNT];

    // copy PADDED_STATE_COUNT*PATTERN_BLOCK_SIZE lengthed partials
    if (pattern < totalPatterns) {
        sPartials2[patIdx][state] = dParentPartials[y + state];
    } else {
        sPartials2[patIdx][state] = 0;
    }

    if (pattern < totalPatterns) {
        int state1 = dChildStates[pattern]; // Coalesced; no need to share

        REAL* matrix1 = dTransMatrix + deltaMatrix + state1 * PADDED_STATE_COUNT;

        if (state1 < PADDED_STATE_COUNT)
            sum1 = matrix1[state];
        else
            sum1 = 1.0;
    }

    if (pattern < totalPatterns)
        dPartialsTmp[u] = sum1 * sPartials2[patIdx][state];                         
}

} // extern "C"

