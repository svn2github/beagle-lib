#include "hip/hip_runtime.h"
/*
 *
 * Copyright 2009 Phylogenetic Likelihood Working Group
 *
 * This file is part of BEAGLE.
 *
 * BEAGLE is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of
 * the License, or (at your option) any later version.
 *
 * BEAGLE is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with BEAGLE.  If not, see
 * <http://www.gnu.org/licenses/>.
 *
 * @author Marc Suchard
 * @author Daniel Ayres
 */

#include "libhmsbeagle/GPU/GPUImplDefs.h"

#define DETERMINE_INDICES() \
    int state = threadIdx.x; \
    int patIdx = threadIdx.y; \
    int pattern = __umul24(blockIdx.x,PATTERN_BLOCK_SIZE) + patIdx; \
    int matrix = blockIdx.y; \
    int patternCount = totalPatterns; \
    int deltaPartialsByState = pattern * PADDED_STATE_COUNT; \
    int deltaPartialsByMatrix = matrix * PADDED_STATE_COUNT * patternCount; \
    int deltaMatrix = matrix * PADDED_STATE_COUNT * PADDED_STATE_COUNT; \
    int u = state + deltaPartialsByState + deltaPartialsByMatrix;

extern "C" {

__global__ void kernelMatrixMulADB(REAL** listC,
                                   REAL* A,
                                   REAL* D,
                                   REAL* B,
                                   REAL* distanceQueue,
                                   int length,
                                   int wB,
                                   int totalMatrix) {

    __shared__ REAL* C;
    __shared__ REAL distance;

    int wMatrix = blockIdx.x % totalMatrix;

    // Block index
    int bx = blockIdx.x / totalMatrix;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int BLOCKS = gridDim.y;

    if (tx == 0 && ty == 0) {
        C = (REAL*) *((int*)listC + wMatrix); // Non-coalescent read
        distance = distanceQueue[wMatrix]; // Non-coalescent read
    }

    __syncthreads();

    const int EDGE = PADDED_STATE_COUNT - (BLOCKS - 1) * MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of A
    int aStep = MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of B
    int bStep = MULTIPLY_BLOCK_SIZE * PADDED_STATE_COUNT;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    REAL Csub = 0;

    int a = PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE * by;
    int b = MULTIPLY_BLOCK_SIZE * bx;
    int d = 0; //MULTIPLY_BLOCK_SIZE * bx;

    __shared__ REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    __shared__ REAL Bs[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    __shared__ REAL Ds[MULTIPLY_BLOCK_SIZE];

    for (int i = 0; i < BLOCKS - 1; i++) {

        if (ty == 0)
            Ds[tx] = exp(D[d + tx] * distance);

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

        __syncthreads();

        for (int k = 0; k < MULTIPLY_BLOCK_SIZE; ++k)
            Csub += As[ty][k] * Ds[k] * Bs[k][tx];

        __syncthreads();

        a += aStep;
        b += bStep;
        d += MULTIPLY_BLOCK_SIZE;
    }

    // Last block is too long
    if (tx < EDGE && ty < EDGE) {
        if (ty == 0)
            Ds[tx] = exp(D[d + tx] * distance);

#ifndef KERNEL_PRINT_ENABLED
        __syncthreads();
#endif

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

    } else {

        if (ty == 0)
            Ds[tx] = 0;

        As[ty][tx] = 0;
        Bs[ty][tx] = 0;
    }

    __syncthreads();

    for (int k = 0; k < EDGE; k++)
        Csub += As[ty][k] * Ds[k] * Bs[k][tx];

    __syncthreads();

    // Write the block sub-matrix to device memory;
    // each thread writes one element

    if ((tx < EDGE || bx < BLOCKS - 1) && (ty < EDGE || by < BLOCKS - 1)) { // It's OK to write
        if (Csub < 0)
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = 0;
        else
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = Csub;
    }
}

__global__ void kernelMatrixMulADBFirstDeriv(REAL** listC,
                                           REAL* A,
                                           REAL* D,
                                           REAL* B,
                                           REAL* distanceQueue,
                                           int length,
                                           int wB,
                                           int totalMatrix) {

    __shared__ REAL* C;
    __shared__ REAL* CFirstDeriv;
    __shared__ REAL distanceLength;
    __shared__ REAL distanceRate;

    int wMatrix = blockIdx.x % totalMatrix;

    // Block index
    int bx = blockIdx.x / totalMatrix;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int BLOCKS = gridDim.y;

    if (tx == 0 && ty == 0) {
        C = (REAL*) *((int*)listC + wMatrix); // Non-coalescent read
        CFirstDeriv = (REAL*) *((int*)listC + wMatrix + totalMatrix); // Non-coalescent read
        distanceLength = distanceQueue[wMatrix]; // Non-coalescent read
        distanceRate = distanceQueue[wMatrix + totalMatrix]; // Non-coalescent read
    }

    __syncthreads();

    const int EDGE = PADDED_STATE_COUNT - (BLOCKS - 1) * MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of A
    int aStep = MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of B
    int bStep = MULTIPLY_BLOCK_SIZE * PADDED_STATE_COUNT;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    REAL Csub = 0;
    REAL CFirstDerivSub = 0;

    int a = PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE * by;
    int b = MULTIPLY_BLOCK_SIZE * bx;
    int d = 0; //MULTIPLY_BLOCK_SIZE * bx;

    __shared__ REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    __shared__ REAL Bs[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    __shared__ REAL Ds[MULTIPLY_BLOCK_SIZE][2];

    for (int i = 0; i < BLOCKS - 1; i++) {

        if (ty == 0) {
            REAL scaledEigenTmp = D[d + tx] * distanceRate;
            Ds[tx][0] = exp(scaledEigenTmp * distanceLength);
            Ds[tx][1] = scaledEigenTmp * Ds[tx][0];
        }

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

        __syncthreads();

        for (int k = 0; k < MULTIPLY_BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Ds[k][0] * Bs[k][tx];
            CFirstDerivSub += As[ty][k] * Ds[k][1] * Bs[k][tx];
        }

        __syncthreads();

        a += aStep;
        b += bStep;
        d += MULTIPLY_BLOCK_SIZE;
    }

    // Last block is too long
    if (tx < EDGE && ty < EDGE) {
        if (ty == 0) {
            REAL scaledEigenTmp = D[d + tx] * distanceRate;
            Ds[tx][0] = exp(scaledEigenTmp * distanceLength);
            Ds[tx][1] = scaledEigenTmp * Ds[tx][0];
                }

#ifndef KERNEL_PRINT_ENABLED
        __syncthreads();
#endif

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

    } else {

        if (ty == 0) {
            Ds[tx][0] = 0;
            Ds[tx][1] = 0;
        }

        As[ty][tx] = 0;
        Bs[ty][tx] = 0;
    }

    __syncthreads();

    for (int k = 0; k < EDGE; k++) {
        Csub += As[ty][k] * Ds[k][0] * Bs[k][tx];
        CFirstDerivSub += As[ty][k] * Ds[k][1] * Bs[k][tx];
    }

    __syncthreads();

    // Write the block sub-matrix to device memory;
    // each thread writes one element

    if ((tx < EDGE || bx < BLOCKS - 1) && (ty < EDGE || by < BLOCKS - 1)) { // It's OK to write
        if (Csub < 0)
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = 0;
        else
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = Csub;

        CFirstDeriv[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
          PADDED_STATE_COUNT * ty + tx] = CFirstDerivSub;
    }
}

__global__ void kernelMatrixMulADBSecondDeriv(REAL** listC,
                                           REAL* A,
                                           REAL* D,
                                           REAL* B,
                                           REAL* distanceQueue,
                                           int length,
                                           int wB,
                                           int totalMatrix) {

    __shared__ REAL* C;
    __shared__ REAL* CFirstDeriv;
    __shared__ REAL* CSecondDeriv;
    __shared__ REAL distanceLength;
    __shared__ REAL distanceRate;

    int wMatrix = blockIdx.x % totalMatrix;

    // Block index
    int bx = blockIdx.x / totalMatrix;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int BLOCKS = gridDim.y;

    if (tx == 0 && ty == 0) {
        C = (REAL*) *((int*)listC + wMatrix); // Non-coalescent read
        CFirstDeriv = (REAL*) *((int*)listC + wMatrix + totalMatrix); // Non-coalescent read
        CSecondDeriv = (REAL*) *((int*)listC + wMatrix + totalMatrix * 2); // Non-coalescent read
        distanceLength = distanceQueue[wMatrix]; // Non-coalescent read
        distanceRate = distanceQueue[wMatrix + totalMatrix]; // Non-coalescent read
    }

    __syncthreads();

    const int EDGE = PADDED_STATE_COUNT - (BLOCKS - 1) * MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of A
    int aStep = MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of B
    int bStep = MULTIPLY_BLOCK_SIZE * PADDED_STATE_COUNT;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    REAL Csub = 0;
    REAL CFirstDerivSub = 0;
    REAL CSecondDerivSub = 0;

    int a = PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE * by;
    int b = MULTIPLY_BLOCK_SIZE * bx;
    int d = 0; //MULTIPLY_BLOCK_SIZE * bx;

    __shared__ REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    __shared__ REAL Bs[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    __shared__ REAL Ds[MULTIPLY_BLOCK_SIZE][3];

    for (int i = 0; i < BLOCKS - 1; i++) {

        if (ty == 0) {
            REAL scaledEigenTmp = D[d + tx] * distanceRate;
            Ds[tx][0] = exp(scaledEigenTmp * distanceLength);
            Ds[tx][1] = scaledEigenTmp * Ds[tx][0];
            Ds[tx][2] = scaledEigenTmp * Ds[tx][1];
        }

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

        __syncthreads();

        for (int k = 0; k < MULTIPLY_BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Ds[k][0] * Bs[k][tx];
            CFirstDerivSub += As[ty][k] * Ds[k][1] * Bs[k][tx];
            CSecondDerivSub += As[ty][k] * Ds[k][2] * Bs[k][tx];
        }

        __syncthreads();

        a += aStep;
        b += bStep;
        d += MULTIPLY_BLOCK_SIZE;
    }

    // Last block is too long
    if (tx < EDGE && ty < EDGE) {
        if (ty == 0) {
            REAL scaledEigenTmp = D[d + tx] * distanceRate;
            Ds[tx][0] = exp(scaledEigenTmp * distanceLength);
            Ds[tx][1] = scaledEigenTmp * Ds[tx][0];
            Ds[tx][2] = scaledEigenTmp * Ds[tx][1];
                }

#ifndef KERNEL_PRINT_ENABLED
        __syncthreads();
#endif

        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        Bs[ty][tx] = B[b + PADDED_STATE_COUNT * ty + tx];

    } else {

        if (ty == 0) {
            Ds[tx][0] = 0;
            Ds[tx][1] = 0;
            Ds[tx][2] = 0;
        }

        As[ty][tx] = 0;
        Bs[ty][tx] = 0;
    }

    __syncthreads();

    for (int k = 0; k < EDGE; k++) {
        Csub += As[ty][k] * Ds[k][0] * Bs[k][tx];
        CFirstDerivSub += As[ty][k] * Ds[k][1] * Bs[k][tx];
        CSecondDerivSub += As[ty][k] * Ds[k][2] * Bs[k][tx];
    }

    __syncthreads();

    // Write the block sub-matrix to device memory;
    // each thread writes one element

    if ((tx < EDGE || bx < BLOCKS - 1) && (ty < EDGE || by < BLOCKS - 1)) { // It's OK to write
        if (Csub < 0)
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = 0;
        else
            C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = Csub;

        CFirstDeriv[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
          PADDED_STATE_COUNT * ty + tx] = CFirstDerivSub;
          
        CSecondDeriv[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
          PADDED_STATE_COUNT * ty + tx] = CSecondDerivSub;
    }
}

#define READ_SCHUR_VALUES() \
		if (ty == 0) { \
			Ds[tx] = exp(D[d + tx] * distance); \
			Cs[tx] = D[d + PADDED_STATE_COUNT + tx] * distance; \
			if (Cs[tx]) { \
            	REAL expat = Ds[tx]; \
            	REAL cosbt = cos(Cs[tx]); \
            	Cs[tx] = -expat * sin(Cs[tx]); \
            	Ds[tx] *= cosbt; \
            } \
        }
// end READ_SCHUR_VALUES

#define POPULATE_SCHUR_BAND(limit) \
		if (ty == 0 && tx == 0) { \
			for(int k=0; k<limit; k++) { \
				if (Cs[k] && !Es[k]) { \
					E0[k] = Cs[k]; \
				} else { \
					E0[k] = 0; \
				} \
			} \
		}
// end POPULATE_SCHUR_BAND(limit)

#define DO_MULTIPLICATION(limit) \
		for (int k = 0; k < limit; k++) { \
			Csub += As[ty][k] * ( \
					Ds[k] * B0 [k * MULTIPLY_BLOCK_SIZE + tx] \
				  + E0[k] * Bp1[k * MULTIPLY_BLOCK_SIZE + tx] \
				  - Es[k] * Bm1[k * MULTIPLY_BLOCK_SIZE + tx] \
			); \
		}
// end DO_MULTIPLICATION(limit)

__global__ void kernelMatrixMulADBComplex(REAL** listC,
                                   REAL* A,
                                   REAL* D,
                                   REAL* B,
                                   REAL* distanceQueue,
                                   int length,
                                   int wB,
                                   int totalMatrix) {

    __shared__ REAL* C;
    __shared__ REAL distance;

    int wMatrix = blockIdx.x % totalMatrix;

    // Block index
    int bx = blockIdx.x / totalMatrix;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int BLOCKS = gridDim.y;

    if (tx == 0 && ty == 0) {
        C = (REAL*) *((int*)listC + wMatrix); // Non-coalescent read
        distance = distanceQueue[wMatrix]; // Non-coalescent read
    }

    __syncthreads();

    const int EDGE = PADDED_STATE_COUNT - (BLOCKS - 1) * MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of A
    int aStep = MULTIPLY_BLOCK_SIZE;

    // Step size used to iterate through the sub-matrices of B
    int bStep = MULTIPLY_BLOCK_SIZE * PADDED_STATE_COUNT;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    REAL Csub = 0;

    int a = PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE * by;
    int b = MULTIPLY_BLOCK_SIZE * bx;
    int d = 0; //MULTIPLY_BLOCK_SIZE * bx;

    __shared__ REAL As[MULTIPLY_BLOCK_SIZE][MULTIPLY_BLOCK_SIZE];
    __shared__ REAL Bs[MULTIPLY_BLOCK_SIZE + 2][MULTIPLY_BLOCK_SIZE];
    __shared__ REAL Cs[MULTIPLY_BLOCK_SIZE];
    __shared__ REAL Ds[MULTIPLY_BLOCK_SIZE];
    __shared__ REAL Es[MULTIPLY_BLOCK_SIZE + 2];
    
   	REAL* B0  = &Bs[1][0];
   	REAL* Bm1 = &Bs[0][0];
   	REAL* Bp1 = &Bs[2][0];    	
   	REAL* E0  = &Es[1];
   	
   	// Zero first row of Bs and Es
   	if (ty == 0) {
   		Bs[0][tx] = 0;
   		if (tx == 0) {
   			Es[0] = 0;	
   		}
   	}

    for (int i = 0; i < BLOCKS - 1; i++) {

        READ_SCHUR_VALUES();

        // Block read A and B sub-matrices
        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        B0[ty * MULTIPLY_BLOCK_SIZE + tx] = B[b + PADDED_STATE_COUNT * ty + tx];

        // Read extra row of B for Bp1
        if (ty == 0) {
        	B0[MULTIPLY_BLOCK_SIZE * MULTIPLY_BLOCK_SIZE + tx] =
        			B[b + PADDED_STATE_COUNT * MULTIPLY_BLOCK_SIZE + tx];
        }

        // All necessary values loaded
    	__syncthreads();

    	POPULATE_SCHUR_BAND(MULTIPLY_BLOCK_SIZE);

    	__syncthreads();

        DO_MULTIPLICATION(MULTIPLY_BLOCK_SIZE);

        // Move last entries in B0 and E0 to first entries in Bs and Es
        if (ty == 0) {
        	Bm1[tx] = Bm1[MULTIPLY_BLOCK_SIZE*MULTIPLY_BLOCK_SIZE + tx];
        	if (tx == 0) {
        		Es[0] = Es[MULTIPLY_BLOCK_SIZE];
        	}
        }

        __syncthreads();

        // Increment sub-matrices
        a += aStep;
        b += bStep;
        d += MULTIPLY_BLOCK_SIZE;

    }
	
    if (tx < EDGE && ty < EDGE) { // Last block is too long

        READ_SCHUR_VALUES();
        
        As[ty][tx] = A[a + PADDED_STATE_COUNT * ty + tx];
        B0[ty * MULTIPLY_BLOCK_SIZE + tx] = B[b + PADDED_STATE_COUNT * ty + tx];

    } else {
    	if (ty == 0) {
    		Ds[tx] = 0;
    		Cs[tx] = 0;
    	}
    	As[ty][tx] = 0;
    	B0[ty * MULTIPLY_BLOCK_SIZE + tx] = 0;    
    }

	// Zero last row of Bs and Es (only for unrolled iteration at end)
    if (ty == 0) {
    	Bs[MULTIPLY_BLOCK_SIZE+1][tx] = 0;
    }

    // All necessary values loaded
	__syncthreads();
	
	POPULATE_SCHUR_BAND(EDGE);

	__syncthreads();

	// Do matrix multiplication
	DO_MULTIPLICATION(EDGE);

    __syncthreads();

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    
    if (Csub < 0)
    	Csub = 0;

    if ((tx < EDGE || bx < BLOCKS - 1) && (ty < EDGE || by < BLOCKS - 1)) { // It's OK to write
        C[PADDED_STATE_COUNT* MULTIPLY_BLOCK_SIZE * by + MULTIPLY_BLOCK_SIZE * bx +
              PADDED_STATE_COUNT * ty + tx] = Csub;
    }
}

__global__ void kernelAccumulateFactors(REAL** dNodePtrQueue,
                                                   REAL* rootScaling,
                                                   int nodeCount,
                                                   int patternCount) {
    int pattern = threadIdx.x + blockIdx.x * PATTERN_BLOCK_SIZE;

    REAL total = 0;
    REAL* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
//      if (threadIdx.x == 0) // TODO Why does this not work???
            nodeScales = (REAL*) *((int*)dNodePtrQueue + n);
//      __syncthreads();

#ifdef KERNEL_PRINT_ENABLED
        if (pattern == 1)
            printf("added %1.2e\n", nodeScales[pattern]);
#endif
        REAL factor = nodeScales[pattern];
        if (factor != 1.0) {
#ifdef LSCALER
            total += factor;
#else
            total += log(factor);
#endif
        }
    }

    if (pattern < patternCount)
        rootScaling[pattern] += total;
}

__global__ void kernelRemoveFactors(REAL** dNodePtrQueue,
                                                   REAL* rootScaling,
                                                   int nodeCount,
                                                   int patternCount) {
    int pattern = threadIdx.x + blockIdx.x * PATTERN_BLOCK_SIZE;

    REAL total = 0;
    REAL* nodeScales;

    int n;
    for(n = 0; n < nodeCount; n++) {
//      if (threadIdx.x == 0) // TODO Why does this not work???
            nodeScales = (REAL*) *((int*)dNodePtrQueue + n);
//      __syncthreads();

#ifdef KERNEL_PRINT_ENABLED
        if (pattern == 1)
            printf("added %1.2e\n", nodeScales[pattern]);
#endif
        REAL factor = nodeScales[pattern];
        if (factor != 1.0) {
#ifdef LSCALER
            total += factor;
#else
            total += log(factor);
#endif
        }
    }

    if (pattern < patternCount)
        rootScaling[pattern] -= total;
} 

__global__ void kernelPartialsDynamicScalingSlow(REAL* allPartials,
                                                 REAL* scalingFactors,
                                                 int matrixCount) {
    int state = threadIdx.x;
    int matrix = threadIdx.y;
    int pattern = blockIdx.x;
    int patternCount = gridDim.x;

    int deltaPartialsByMatrix = __umul24(matrix, __umul24( PADDED_STATE_COUNT, patternCount));

    __shared__ REAL partials[PADDED_STATE_COUNT];

    __shared__ REAL max;

    if (state == 0)
        max = 0.0;

    int m;
    for(m = 0; m < matrixCount; m++) {
        partials[state] = allPartials[m * patternCount * PADDED_STATE_COUNT + pattern *
                                      PADDED_STATE_COUNT + state];
        __syncthreads();

#ifdef IS_POWER_OF_TWO
    // parallelized reduction *** only works for powers-of-2 ****
    for (int i = PADDED_STATE_COUNT / 2; i > 0; i >>= 1) {
        if (state < i) {
#else
    for (int i = SMALLEST_POWER_OF_TWO / 2; i > 0; i >>= 1) {
        if (state < i && state + i < PADDED_STATE_COUNT ) {
#endif // IS_POWER_OF_TWO
                REAL compare1 = partials[state];
                REAL compare2 = partials[state + i];
                if(compare2 > compare1)
                    partials[state] = compare2;
            }
            __syncthreads();
        }
        if(state == 0) {
            if( partials[0] > max)
                max = partials[0];
        }
    }

    if(state == 0) {
        if (max == 0)
        	max = 1.0;
#ifdef LSCALER
        scalingFactors[pattern] = log(max);
#else
        scalingFactors[pattern] = max;
#endif
    }


    __syncthreads();

    for(m = 0; m < matrixCount; m++)
        allPartials[m * patternCount * PADDED_STATE_COUNT + pattern * PADDED_STATE_COUNT +
                    state] /= max;

}

} // extern "C"

